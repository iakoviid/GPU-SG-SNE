#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "add_wrapper.hpp"

__global__ void addKernel(float *c, const float *a, const float *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// C = A + B
void addWithCUDA(float *cpuC, const float *cpuA, const float *cpuB, const size_t sz)
{    
    //TODO: add error checking
    
    // choose which GPU to run on
    hipSetDevice(0);
    
    // allocate GPU buffers
    float *gpuA, *gpuB, *gpuC;
    hipMalloc((void**)&gpuA, sz*sizeof(float));
    hipMalloc((void**)&gpuB, sz*sizeof(float));
    hipMalloc((void**)&gpuC, sz*sizeof(float));
    
    // copy input vectors from host memory to GPU buffers
    hipMemcpy(gpuA, cpuA, sz*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpuB, cpuB, sz*sizeof(float), hipMemcpyHostToDevice);
    
    // launch kernel on the GPU with one thread per element
    addKernel<<<1,sz>>>(gpuC, gpuA, gpuB);
    
    // wait for the kernel to finish
    hipDeviceSynchronize();
    
    // copy output vector from GPU buffer to host memory
    hipMemcpy(cpuC, gpuC, sz*sizeof(float), hipMemcpyDeviceToHost);
    
    // cleanup
    hipFree(gpuA);
    hipFree(gpuB);
    hipFree(gpuC);
}

void resetDevice()
{
    hipDeviceReset();
}
