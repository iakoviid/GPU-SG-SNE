#include "hip/hip_runtime.h"

#include "nuconv.cuh"
#include "gpu_timer.h"
#include "utils_cuda.cuh"
#include "complexF.cuh"
#include "complexD.cuh"
#include <fstream>
#include <float.h>
extern hipStream_t streamRep;
#define Blocks 64
#define Threads 1024

__global__ void Normalize(volatile float *__restrict__ y,
                          const uint32_t nPts, const uint32_t ng,
                          const uint32_t d, const float maxy) {
  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {
    for (register int dim = 0; dim < d; dim++) {
      y[TID + dim * nPts] /= maxy;
      if (y[TID + dim * nPts] >= 1) {
        y[TID + dim * nPts] =1 - FLT_EPSILON;
      }
      y[TID + dim * nPts] *= (ng - 3);
    }
  }
}
__global__ void Normalize(volatile double *__restrict__ y,
                          const uint32_t nPts, const uint32_t ng,
                          const uint32_t d, const double maxy) {
  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {
    for (register int dim = 0; dim < d; dim++) {
      y[TID + dim * nPts] /= maxy;
      if (y[TID + dim * nPts] == 1) {
        y[TID + dim * nPts] = 1 - DBL_EPSILON;
      }
      y[TID + dim * nPts] *= (ng - 3);
    }
  }
}
template <class dataPoint,class Complext>
void nuconv(dataPoint *PhiScat, dataPoint *y, dataPoint *VScat,  int n,
            int d, int m, int nGridDim, double *timeInfo, hipfftHandle &plan,
            hipfftHandle &plan_rhs, dataPoint *VGrid,
            dataPoint *PhiGrid, Complext *Kc,
            Complext *Xc) {
  struct GpuTimer timer;
  int szV = pow(nGridDim + 2, d) * m;
  timer.Start(streamRep);

 // ~~~~~~~~~~ Scale coordinates (inside bins)
  thrust::device_ptr<dataPoint> yVec_ptr(y);
  dataPoint maxy =
      thrust::reduce(thrust::cuda::par.on(streamRep), yVec_ptr,
                     yVec_ptr + n * d, 0.0, thrust::maximum<dataPoint>());
 //hipDeviceSynchronize();


  dataPoint h =
      maxy / (nGridDim - 1 - std::numeric_limits<dataPoint>::epsilon());

  // ~~~~~~~~~~ scat2grid

  Normalize<<<Blocks, Threads, 0, streamRep>>>(y, n, nGridDim + 2, d, maxy);
  //hipDeviceSynchronize();

 timer.Stop(streamRep);

  timeInfo[5] += timer.Elapsed()/1000.0;

  timer.Start(streamRep);
  s2g(VGrid, y, VScat, nGridDim, n, d, m);

  timer.Stop(streamRep);
  timeInfo[0] += timer.Elapsed()/1000.0;

 //hipDeviceSynchronize();



 // ~~~~~~~~~~ grid2grid

  uint32_t *const nGridDims = new uint32_t[d]();
  for (int i = 0; i < d; i++) {
    nGridDims[i] = nGridDim + 2;
  }
  timer.Start(streamRep);

  switch (d) {

  case 1:
    conv1dnopadcuda(PhiGrid, VGrid, h, nGridDims, m, d, plan, plan_rhs,Kc,Xc);

    break;

  case 2:
    conv2dnopadcuda(PhiGrid, VGrid, h, nGridDims, m, d, plan, plan_rhs,Kc,Xc);

    break;

  case 3:
    conv3dnopadcuda(PhiGrid, VGrid, h, nGridDims, m, d, plan, plan_rhs,Kc,Xc);
    break;
  }
// hipDeviceSynchronize();
  timer.Stop(streamRep);
  timeInfo[1] = timer.Elapsed()/1000.0;

  // ~~~~~~~~~~ grid2scat
  timer.Start(streamRep);
  g2s(PhiScat, PhiGrid, y, nGridDim, n, d, m);

  timer.Stop(streamRep);
  timeInfo[2] = timer.Elapsed()/1000.0;
  // ~~~~~~~~~~ deallocate memory
 //hipDeviceSynchronize();
  delete nGridDims;
}
template void nuconv(float *PhiScat, float *y, float *VScat,  int n,
                     int d, int m, int nGridDim, double *timeInfo,
                     hipfftHandle &plan, hipfftHandle &plan_rhs, float *VGrid, float *PhiGrid, ComplexF *Kc, ComplexF *Xc);
template void nuconv(double *PhiScat, double *y, double *VScat,  int n,
                     int d, int m, int nGridDim, double *timeInfo,
                     hipfftHandle &plan, hipfftHandle &plan_rhs, double *VGrid, double *PhiGrid,
                      ComplexD *Kc, ComplexD *Xc);
