#include "hip/hip_runtime.h"
#include "Frep.cuh"
#include "complexD.cuh"
#include "complexF.cuh"
#include "gpu_timer.h"
#define N_GRID_SIZE 137
#define Blocks 64
#define Threads 1024
hipStream_t streamRep = 0;
template <class dataPoint>
__global__ void ComputeChargesKernel(volatile dataPoint *__restrict__ VScat,
                                     const dataPoint *const y, const int n,
                                     const int d) {
  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < n;
       TID += gridDim.x * blockDim.x) {

    switch (d) {
    case 1: {
      VScat[TID] = 1;
      VScat[TID + n] = y[TID];
      break;
    }
    case 2: {
      VScat[3 * TID] = 1;
      VScat[3 * TID + 1] = y[2 * TID];
      VScat[3 * TID + 2] = y[2 * TID + 1];
      break;
    }
    case 3: {
      VScat[4 * TID] = 1;
      VScat[4 * TID + 1] = y[3 * TID];
      VScat[4 * TID + 2] = y[3 * TID + 1];
      VScat[4 * TID + 3] = y[3 * TID + 2];
      break;
    }
    }
  }
}
template <class dataPoint>
void ComputeCharges(dataPoint *VScat, dataPoint *y_d, const int n,
                    const int d) {
  ComputeChargesKernel<<<Blocks, Threads>>>(VScat, y_d, n, d);
}
template <class dataPoint>
__global__ void
compute_repulsive_forces_kernel(volatile dataPoint *__restrict__ frep,
                                const dataPoint *const Y, const int num_points,
                                const int nDim, const dataPoint *const Phi,
                                volatile dataPoint *__restrict__ zetaVec) {

  register dataPoint Ysq = 0;
  register dataPoint z = 0;
  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x;
       TID < num_points; TID += gridDim.x * blockDim.x) {
    Ysq = 0;
    z = 0;
    for (uint32_t j = 0; j < nDim; j++) {
      Ysq += Y[nDim * TID + j] * Y[nDim * TID + j];
      z -= 2 * Y[nDim * TID + j] * Phi[TID + (num_points) * (j + 1)];
      frep[nDim * TID + j] =
          Y[nDim * TID + j] * Phi[TID] - Phi[TID + (j + 1) * num_points];
    }

    z += (1 + 2 * Ysq) * Phi[TID];
    zetaVec[TID] = z;
  }
}

template <class dataPoint>
dataPoint zetaAndForce(dataPoint *Ft_d, dataPoint *y_d, int n, int d,
                       dataPoint *Phi,
                       thrust::device_vector<dataPoint> &zetaVec) {
  // can posibly reduce amongs the threads and then divide

  compute_repulsive_forces_kernel<<<Blocks, Threads>>>(
      Ft_d, y_d, n, d, Phi, thrust::raw_pointer_cast(zetaVec.data()));
  // hipDeviceSynchronize();
  dataPoint z = thrust::reduce(zetaVec.begin(), zetaVec.end()) - n;

  normalize<<<Blocks, Threads, 0, streamRep>>>(Ft_d, z, n * d);
  return z;
}


int getBestGridSize1(int nGrid) {

  // list of FFT sizes that work "fast" with FFTW
  int listGridSize[N_GRID_SIZE] = {
      8,   9,   10,  11,  12,  13,  14,  15,  16,  20,  25,  26,  28,  32,
      33,  35,  36,  39,  40,  42,  44,  45,  48,  49,  50,  52,  54,  55,
      56,  60,  63,  64,  65,  66,  70,  72,  75,  77,  78,  80,  84,  88,
      90,  91,  96,  98,  99,  100, 104, 105, 108, 110, 112, 117, 120, 125,
      126, 130, 132, 135, 140, 144, 147, 150, 154, 156, 160, 165, 168, 175,
      176, 180, 182, 189, 192, 195, 196, 198, 200, 208, 210, 216, 220, 224,
      225, 231, 234, 240, 245, 250, 252, 260, 264, 270, 273, 275, 280, 288,
      294, 297, 300, 308, 312, 315, 320, 325, 330, 336, 343, 350, 351, 352,
      360, 364, 375, 378, 385, 390, 392, 396, 400, 416, 420, 432, 440, 441,
      448, 450, 455, 462, 468, 480, 490, 495, 500, 504, 512};

  // select closest (larger) size for given grid size
  for (int i = 0; i < N_GRID_SIZE; i++)
    if ((nGrid + 2) <= listGridSize[i])
      return listGridSize[i] - 2;

  return listGridSize[N_GRID_SIZE - 1] - 2;
}

double computeFrepulsive_gpu(double *Freph, double *yh, int n, int d, double h,
                             double *timeInfo) {

  struct GpuTimer timer;
  timer.Start(streamRep);

  double *y;
  int m = d + 1;
  int nVec = m;
  double miny[4];
  for (int i = 0; i < 4; i++)
    miny[i] = std::numeric_limits<double>::infinity();

  for (int i = 0; i < n; i++)
    for (int j = 0; j < d; j++)
      miny[j] = miny[j] > yh[i * d + j] ? yh[i * d + j] : miny[j];

  gpuErrchk(hipMallocManaged(&y, (d)*n * sizeof(double)));
  gpuErrchk(hipMemcpy(y, yh, n * d * sizeof(double), hipMemcpyHostToDevice));

  // ~~~~~~~~~~ move data to (0,0,...)

  // double miny[4];
  thrust::device_ptr<double> yVec_ptr(y);

  for (int j = 0; j < d; j++) {
    addScalarToCoord<<<Blocks, Threads>>>(y, -miny[j], n, j, d);
  }

  double maxy = thrust::reduce(yVec_ptr, yVec_ptr + n * d, 0.0,
                               thrust::maximum<double>());

  int ng = std::max((int)std::ceil(maxy / h), 14);

  int n1 = getBestGridSize1(ng);
  int nGrid = n1;
  double *yt;
  gpuErrchk(hipMallocManaged(&yt, (d)*n * sizeof(double)));
  double *VScat;
  gpuErrchk(hipMallocManaged(&VScat, (d + 1) * n * sizeof(double)));
  double *PhiScat;
  gpuErrchk(hipMallocManaged(&PhiScat, (d + 1) * n * sizeof(double)));
  int szV = pow(n1 + 2, d) * m;
  double *VGrid;
  gpuErrchk(hipMallocManaged(&VGrid, szV * sizeof(double)));
  double *PhiGrid;
  gpuErrchk(hipMallocManaged(&PhiGrid, szV * sizeof(double)));
  ComplexD *Kc, *Xc;
  gpuErrchk(hipMallocManaged(&Kc, szV * sizeof(ComplexD)));
  gpuErrchk(hipMallocManaged(&Xc, nVec * szV * sizeof(ComplexD)));
  thrust::device_vector<double> zetaVec(n);
  double *Frep;
  gpuErrchk(hipMallocManaged(&Frep, n * d * sizeof(double)));

  initKernel<<<Blocks, Threads>>>(VGrid, (double)0, szV);
  initKernel<<<Blocks, Threads>>>(PhiGrid, (double)0, szV);

  hipfftHandle plan, plan_rhs;

  int n2 = n1 + 2;
  switch (d) {
  case 1: {
    int ng[1] = {(int)n2};
    hipfftPlan1d(&plan, n2, HIPFFT_Z2Z, 1);
    hipfftPlanMany(&plan_rhs, 1, ng, NULL, 1, n2, NULL, 1, n2, HIPFFT_Z2Z, d + 1);
    break;
  }
  case 2: {
    int ng[2] = {(int)n2, (int)n2};
    hipfftPlanMany(&plan, 2, ng, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2Z, 1);
    hipfftPlanMany(&plan_rhs, 2, ng, NULL, 1, n2 * n2, NULL, 1, n2 * n2,
                  HIPFFT_Z2Z, d + 1);
    break;
  }
  case 3: {
    int ng[3] = {(int)n2, (int)n2, (int)n2};
    hipfftPlanMany(&plan, 3, ng, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2Z, 1);
    hipfftPlanMany(&plan_rhs, 3, ng, NULL, 1, n2 * n2 * n2, NULL, 1,
                  n2 * n2 * n2, HIPFFT_Z2Z, d + 1);
    break;
  }
  }

  ArrayCopy<<<Blocks, Threads>>>(y, yt, n * d);

  ComputeCharges(VScat, y, n, d);

  timer.Stop(streamRep);

  timeInfo[5] = timer.Elapsed() / 1000.0;
  timer.Start(streamRep);
  nuconv(PhiScat, yt, VScat, n, d, d + 1, nGrid, timeInfo, plan, plan_rhs,
         VGrid, PhiGrid, Kc, Xc);

  // hipDeviceSynchronize();

  timer.Stop(streamRep);
  timeInfo[4] = timer.Elapsed() / 1000;
  timer.Start(streamRep);
  double zeta = zetaAndForce(Frep, y, n, d, PhiScat, zetaVec);
  timer.Stop(streamRep);
  timeInfo[3] = timer.Elapsed() / 1000.0;

  gpuErrchk(
      hipMemcpy(Freph, Frep, n * d * sizeof(double), hipMemcpyDeviceToHost));
  hipfftDestroy(plan);
  hipfftDestroy(plan_rhs);
  gpuErrchk(hipFree(PhiGrid));
  gpuErrchk(hipFree(VGrid));
  gpuErrchk(hipFree(yt));
  gpuErrchk(hipFree(VScat));
  gpuErrchk(hipFree(PhiScat));
  gpuErrchk(hipFree(Kc));
  gpuErrchk(hipFree(Xc));
  gpuErrchk(hipFree(Frep));
  return zeta;
}
