#include "hip/hip_runtime.h"
#include "../matrix_indexing.hpp"
#include "non_periodic_convF.cuh"
#include "utils_cuda.cuh"
extern hipStream_t streamRep;

#define idx2(i, j, d) (SUB2IND2D(i, j, d))
#define idx3(i, j, k, d1, d2) (SUB2IND3D(i, j, k, d1, d2))
#define idx4(i, j, k, l, m, n, o) (SUB2IND4D(i, j, k, l, m, n, o))
#define HIP_PI_F acos(-1.0)
#define Blocks 64
#define Threads 512
#define Blocks2D 64
#define Threads2D 512
#define Blocks3D 64
#define Threads3D 512

// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(Complex *a, const Complex *b,
                                                   int size, uint32_t nVec) {
  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  for (int j = 0; j < nVec; j++) {

    for (int i = threadID; i < size; i += numThreads) {
      a[i + j * size] = ComplexScale(ComplexMul(a[i + j * size], b[i]), 1.0f);
    }
  }
}

__global__ void setDataFft1D(Complex *Kc, Complex *Xc, int ng, int nVec,
                             float *VGrid, float hsq, int sign) {

  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = threadID; i < ng; i += numThreads) {
    Kc[i].x = kernel1d(hsq, i);
    Kc[i].y = 0;
    if (i > 0) {
      Kc[i].x = Kc[i].x + sign * kernel1d(hsq, ng - i);
      if (sign == -1) {

        Complex arg;
        arg.x = 0;
        arg.y = -2 * HIP_PI_F * i / (2 * ng);
        Kc[i] = ComplexMul(Kc[i], my_cexpf(arg));
      }
    }
    for (int j = 0; j < nVec; j++) {
      Xc[i + j * ng].x = VGrid[i + j * ng];
      Xc[i + j * ng].y = 0;
      if (sign == -1) {
        Complex arg;
        arg.x = 0;
        arg.y = -2 * HIP_PI_F * i / (2 * ng);
        Xc[i + j * ng] = ComplexMul(Xc[i + j * ng], my_cexpf(arg));
      }
    }
  }
}

__global__ void setDataFft2D(Complex *Kc, Complex *Xc, int n1, int n2, int nVec,
                             const float *const VGrid, float hsq, int signx,
                             int signy) {

      register  int i,j;
  for (register uint32_t TID = blockIdx.x * blockDim.x + threadIdx.x; TID < n1*n2;TID += blockDim.x * gridDim.x) {
      i=TID%n1;
      j=(TID/n1);

      Kc[idx2(i, j, n1)].x = kernel2d(hsq, i, j);
      Kc[idx2(i, j, n1)].y = 0;
      if (i > 0) {
        Kc[idx2(i, j, n1)].x += signx * kernel2d(hsq, n1 - i, j);
      }
      if (j > 0) {
        Kc[idx2(i, j, n1)].x += signy * kernel2d(hsq, i, n2 - j);
      }
      if (i > 0 && j > 0) {
        Kc[idx2(i, j, n1)].x += signx * signy * kernel2d(hsq, n1 - i, n2 - j);
      }

      for (uint32_t iVec = 0; iVec < nVec; iVec++) {
        Xc[idx3(i, j, iVec, n1, n2)].x = VGrid[idx3(i, j, iVec, n1, n2)];
        Xc[idx3(i, j, iVec, n1, n2)].y = 0;
        if (signx == -1) {
          Complex arg;
          arg.x = 0;
          arg.y = -2 * HIP_PI_F * i / (2 * n1);
          Xc[idx3(i, j, iVec, n1, n2)] =
              ComplexMul(Xc[idx3(i, j, iVec, n1, n2)], my_cexpf(arg));
        }
        if (signy == -1) {
          Complex arg;
          arg.x = 0;
          arg.y = -2 * HIP_PI_F * j / (2 * n2);
          Xc[idx3(i, j, iVec, n1, n2)] =
              ComplexMul(Xc[idx3(i, j, iVec, n1, n2)], my_cexpf(arg));
        }
      }
      if (signx == -1) {
        Complex arg;
        arg.x = 0;
        arg.y = -2 * HIP_PI_F * i / (2 * n1);
        Kc[idx2(i, j, n1)] = ComplexMul(Kc[idx2(i, j, n1)], my_cexpf(arg));
      }

      if (signy == -1) {
        Complex arg;
        arg.x = 0;
        arg.y = -2 * HIP_PI_F * j / (2 * n2);
        Kc[idx2(i, j, n1)] = ComplexMul(Kc[idx2(i, j, n1)], my_cexpf(arg));
      }
    }
  }
__global__ void addToPhiGrid(Complex *Xc, float *PhiGrid, int ng, float scale) {

  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = threadID; i < ng; i += numThreads) {
    PhiGrid[i] += scale * Xc[i].x;
  }
}

__global__ void normalizeInverse(Complex *Xc, int ng, uint32_t nVec) {

  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = threadID; i < ng; i += numThreads) {
    for (uint32_t iVec = 0; iVec < nVec; iVec++) {
      Complex arg;
      arg.x = 0;
      arg.y = +2 * HIP_PI_F * i / (2 * ng);
      Xc[i + iVec * ng] = ComplexMul(Xc[i + iVec * ng], my_cexpf(arg));
    }
  }
}

__global__ void normalizeInverse2D(Complex *Xc, uint32_t n1, uint32_t n2,
                                   uint32_t nVec, int signx, int signy) {

    register  int i,j;
  for (register uint32_t TID = blockIdx.x * blockDim.x + threadIdx.x; TID < n1*n2;TID += blockDim.x * gridDim.x) {
      i=TID%n1;
      j=(TID/n1);

      for (uint32_t iVec = 0; iVec < nVec; iVec++) {
        if (signx == -1) {
          Complex arg;
          arg.x = 0;
          arg.y = +2 * HIP_PI_F * i / (2 * n1);
          Xc[idx3(i, j, iVec, n1, n2)] =
              ComplexMul(Xc[idx3(i, j, iVec, n1, n2)], my_cexpf(arg));
        }
        if (signy == -1) {
          Complex arg;
          arg.x = 0;
          arg.y = +2 * HIP_PI_F * j / (2 * n2);
          Xc[idx3(i, j, iVec, n1, n2)] =
              ComplexMul(Xc[idx3(i, j, iVec, n1, n2)], my_cexpf(arg));
        }
      }
    }
  
}

void conv1dnopadcuda(float *PhiGrid, float *VGrid, float h,
                     uint32_t *const nGridDims, uint32_t nVec, int nDim,
                     hipfftHandle &plan, hipfftHandle &plan_rhs, Complex *Kc,
                     Complex *Xc) {

  uint32_t n1 = nGridDims[0];
  float hsq = h * h;

  /*even*/
  setDataFft1D<<<Blocks, Threads, 0, streamRep>>>(Kc, Xc, n1, nVec, VGrid, hsq,
                                                  1);

  // hipDeviceSynchronize(streamRep); // why

  hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(Kc),
               reinterpret_cast<hipfftComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_FORWARD);

  ComplexPointwiseMulAndScale<<<Blocks, Threads, 0, streamRep>>>(Xc, Kc, n1,
                                                                 nVec);

  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_BACKWARD);
  addToPhiGrid<<<Blocks, Threads, 0, streamRep>>>(Xc, PhiGrid, n1 * nVec,
                                                  (0.5 / n1));

  // hipDeviceSynchronize(streamRep); // why

  setDataFft1D<<<Blocks, Threads, 0, streamRep>>>(Kc, Xc, n1, nVec, VGrid, hsq,
                                                  -1);

  hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(Kc),
               reinterpret_cast<hipfftComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_FORWARD);

  ComplexPointwiseMulAndScale<<<Blocks, Threads, 0, streamRep>>>(Xc, Kc, n1,
                                                                 nVec);

  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_BACKWARD);

  normalizeInverse<<<Blocks, Threads, 0, streamRep>>>(Xc, n1, nVec);

  addToPhiGrid<<<Blocks, Threads, 0, streamRep>>>(Xc, PhiGrid, n1 * nVec,
                                                  (0.5 / n1));

  return;
}
void conv2dnopadcuda(float *const PhiGrid, const float *const VGrid,
                     const float h, uint32_t *const nGridDims,
                     const uint32_t nVec, const uint32_t nDim,
                     hipfftHandle &plan, hipfftHandle &plan_rhs, Complex *Kc,
                     Complex *Xc) {
  float hsq = h * h;

  // find the size of the last dimension in FFTW (add padding)
  uint32_t n1 = nGridDims[0];
  uint32_t n2 = nGridDims[1];
  int grid=64;
  int block=1024;

  // ============================== EVEN-EVEN

  setDataFft2D<<<grid, block, 0, streamRep>>>(Kc, Xc, n1, n2, nVec, VGrid, hsq,
                                              1, 1);
//hipDeviceSynchronize();
  hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(Kc),
               reinterpret_cast<hipfftComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_FORWARD);
//hipDeviceSynchronize();

  ComplexPointwiseMulAndScale<<<Blocks2D, Threads2D, 0, streamRep>>>(
      Xc, Kc, n1 * n2, nVec);
//hipDeviceSynchronize();
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_BACKWARD);
//hipDeviceSynchronize();  
addToPhiGrid<<<Blocks2D, Threads2D, 0, streamRep>>>(
      Xc, PhiGrid, n1 * n2 * nVec, (0.25 / (n1 * n2)));
//hipDeviceSynchronize();

  // ============================== ODD-EVEN

  setDataFft2D<<<grid, block, 0, streamRep>>>(Kc, Xc, n1, n2, nVec, VGrid, hsq,
                                              -1, 1);
//hipDeviceSynchronize(); 

 hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(Kc),
               reinterpret_cast<hipfftComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_FORWARD);
//hipDeviceSynchronize();
  ComplexPointwiseMulAndScale<<<Blocks2D, Threads2D, 0, streamRep>>>(
      Xc, Kc, n1 * n2, nVec);
//hipDeviceSynchronize();
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_BACKWARD);
//hipDeviceSynchronize(); 
 normalizeInverse2D<<<grid, block, 0, streamRep>>>(Xc, n1, n2, nVec, -1, 1);
//hipDeviceSynchronize(); 
 addToPhiGrid<<<Blocks2D, Threads2D, 0, streamRep>>>(
      Xc, PhiGrid, n1 * n2 * nVec, (0.25 / (n1 * n2)));
//hipDeviceSynchronize();
  // ============================== EVEN-ODD

  setDataFft2D<<<grid, block, 0, streamRep>>>(Kc, Xc, n1, n2, nVec, VGrid, hsq,
                                              1, -1);
//hipDeviceSynchronize(); 
 hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(Kc),
               reinterpret_cast<hipfftComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_FORWARD);
//hipDeviceSynchronize(); 
 ComplexPointwiseMulAndScale<<<Blocks2D, Threads2D, 0, streamRep>>>(
      Xc, Kc, n1 * n2, nVec);
//hipDeviceSynchronize();
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_BACKWARD);
//hipDeviceSynchronize();
  normalizeInverse2D<<<grid, block, 0, streamRep>>>(Xc, n1, n2, nVec, 1, -1);
//hipDeviceSynchronize();
  addToPhiGrid<<<Blocks2D, Threads2D, 0, streamRep>>>(
      Xc, PhiGrid, n1 * n2 * nVec, (0.25 / (n1 * n2)));
//hipDeviceSynchronize();
  // ============================== ODD-ODD

  setDataFft2D<<<grid, block, 0, streamRep>>>(Kc, Xc, n1, n2, nVec, VGrid, hsq,
                                              -1, -1);
//hipDeviceSynchronize();  
hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(Kc),
               reinterpret_cast<hipfftComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_FORWARD);
//hipDeviceSynchronize();
  ComplexPointwiseMulAndScale<<<Blocks2D, Threads2D, 0, streamRep>>>(
      Xc, Kc, n1 * n2, nVec);
//hipDeviceSynchronize();
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_BACKWARD);
//hipDeviceSynchronize();
  normalizeInverse2D<<<grid, block, 0, streamRep>>>(Xc, n1, n2, nVec, -1, -1);
//hipDeviceSynchronize();
  addToPhiGrid<<<Blocks2D, Threads2D, 0, streamRep>>>(
      Xc, PhiGrid, n1 * n2 * nVec, (0.25 / (n1 * n2)));
//hipDeviceSynchronize();
  return;
}

__global__ void setDataFft3D(Complex *Kc, Complex *Xc,const int n1,const int n2,const int n3,
                             const int nVec, const float *const VGrid,const float hsq,
                            const int signx,const int signy,const int signz) {
      register int i,j,k;
      register Complex K,X;
	for (register uint32_t TID = blockIdx.x * blockDim.x + threadIdx.x; TID < n1*n2*n3;
           TID += blockDim.x * gridDim.x) {
	i=TID%n1;
	j=(TID/n1)%n2;
	k=(TID/n1)/n2;
        K.x = kernel3d(hsq, i, j, k);
        K.y = 0;
        if (i > 0) {
          K.x += signx * kernel3d(hsq, n1 - i, j, k);
        }
        if (j > 0) {
          K.x += signy * kernel3d(hsq, i, n2 - j, k);
        }
        if (i > 0 && j > 0) {
          K.x +=
              signx * signy * kernel3d(hsq, n1 - i, n2 - j, k);
        }
        if (k > 0) {
          K.x += signz * kernel3d(hsq, i, j, n3 - k);
        }
        if (k > 0 && i > 0) {
          K.x +=
              signx * signz * kernel3d(hsq, n1 - i, j, n3 - k);
        }
        if (k > 0 && j > 0) {
          K.x +=
              signy * signz * kernel3d(hsq, i, n2 - j, n3 - k);
        }
        if (k > 0 && i > 0 && j > 0) {
          K.x +=
              signx * signy * signz * kernel3d(hsq, n1 - i, n2 - j, n3 - k);
        }

        for (uint32_t iVec = 0; iVec < nVec; iVec++) {
          X.x =
              VGrid[idx4(i, j, k, iVec, n1, n2, n3)];
          X.y = 0;
          if (signx == -1) {
            Complex arg;
            arg.x = 0;
            arg.y = -2 * HIP_PI_F * i / (2 * n1);
            X =
                ComplexMul(X, my_cexpf(arg));
          }
          if (signy == -1) {
            Complex arg;
            arg.x = 0;
            arg.y = -2 * HIP_PI_F * j / (2 * n2);
            X = ComplexMul(X, my_cexpf(arg));
          }
          if (signz == -1) {
            Complex arg;
            arg.x = 0;
            arg.y = -2 * HIP_PI_F * k / (2 * n3);
            X= ComplexMul(X, my_cexpf(arg));
          }
	 Xc[idx4(i, j, k, iVec, n1, n2, n3)] =X;
        }
        if (signx == -1) {
          Complex arg;
          arg.x = 0;
          arg.y = -2 * HIP_PI_F * i / (2 * n1);
          K =   ComplexMul(K, my_cexpf(arg));
        }

        if (signy == -1) {
          Complex arg;
          arg.x = 0;
          arg.y = -2 * HIP_PI_F * j / (2 * n2);
          K =
              ComplexMul(K, my_cexpf(arg));
        }

        if (signz == -1) {
          Complex arg;
          arg.x = 0;
          arg.y = -2 * HIP_PI_F * k / (2 * n3);
          K =
              ComplexMul(K, my_cexpf(arg));
        }
	Kc[idx3(i, j, k, n1, n2)]=K;
      }

  
}

__global__ void normalizeInverse3D(Complex *Xc, uint32_t n1, uint32_t n2,
                                   uint32_t n3, uint32_t nVec, int signx,
                                   int signy, int signz) {

  
       register int i,j,k;
      for (register uint32_t TID = blockIdx.x * blockDim.x + threadIdx.x; TID < n1*n2*n3;
           TID += blockDim.x * gridDim.x) {
        i=TID%n1;
        j=(TID/n1)%n2;
        k=(TID/n1)/n2;

        for (uint32_t iVec = 0; iVec < nVec; iVec++) {
          if (signx == -1) {
            Complex arg;
            arg.x = 0;
            arg.y = +2 * HIP_PI_F * i / (2 * n1);
            Xc[idx4(i, j, k, iVec, n1, n2, n3)] =
                ComplexMul(Xc[idx4(i, j, k, iVec, n1, n2, n3)], my_cexpf(arg));
          }
          if (signy == -1) {
            Complex arg;
            arg.x = 0;
            arg.y = +2 * HIP_PI_F * j / (2 * n2);
            Xc[idx4(i, j, k, iVec, n1, n2, n3)] =
                ComplexMul(Xc[idx4(i, j, k, iVec, n1, n2, n3)], my_cexpf(arg));
          }
          if (signz == -1) {
            Complex arg;
            arg.x = 0;
            arg.y = +2 * HIP_PI_F * k / (2 * n3);
            Xc[idx4(i, j, k, iVec, n1, n2, n3)] =
                ComplexMul(Xc[idx4(i, j, k, iVec, n1, n2, n3)], my_cexpf(arg));
          }
        }
      }
    }
void term3D(Complex *Kc, Complex *Xc, uint32_t n1, uint32_t n2, uint32_t n3,
            uint32_t nVec, const float *const VGrid, float *PhiGrid, float hsq,
            hipfftHandle plan, hipfftHandle plan_rhs, int signx, int signy,
            int signz) {
  dim3 block(16, 14, 2);
  dim3 grid(iDivUp(n1, 16), iDivUp(n2, 14), iDivUp(n3, 2));

  setDataFft3D<<<64, 1024, 0, streamRep>>>(Kc, Xc, n1, n2, n3, nVec, VGrid,
                                              hsq, signx, signy, signz);

  hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(Kc),
               reinterpret_cast<hipfftComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_FORWARD);

  ComplexPointwiseMulAndScale<<<Blocks3D, Threads3D, 0, streamRep>>>(
      Xc, Kc, n1 * n2 * n3, nVec);

  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_BACKWARD);
  if (signx == -1 || signy == -1 || signz == -1) {
    normalizeInverse3D<<<64, 1024, 0, streamRep>>>(Xc, n1, n2, n3, nVec,
                                                      signx, signy, signz);
  }
  addToPhiGrid<<<Blocks3D, Threads3D, 0, streamRep>>>(
      Xc, PhiGrid, n1 * n2 * n3 * nVec, (0.125 / (n1 * n2 * n3)));
}

void conv3dnopadcuda(float *const PhiGrid, const float *const VGrid,
                     const float h, uint32_t *const nGridDims,
                     const uint32_t nVec, const uint32_t nDim,
                     hipfftHandle &plan, hipfftHandle &plan_rhs, Complex *Kc,
                     Complex *Xc) {

  float hsq = h * h;

  // find the size of the last dimension in FFTW (add padding)
  uint32_t n1 = nGridDims[0];
  uint32_t n2 = nGridDims[1];
  uint32_t n3 = nGridDims[2];
  // ============================== EVEN-EVEN-EVEN

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, 1, 1,
         1);

  // ============================== ODD-EVEN-EVEN

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, -1, 1,
         1);

  // ============================== EVEN-ODD-EVEN

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, 1, -1,
         1);

  // ============================== ODD-ODD-EVEN

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, -1, -1,
         1);

  // ============================== EVEN-EVEN-ODD

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, 1, 1,
         -1);

  // ============================== EVEN-ODD-EVEN

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, -1, 1,
         -1);

  // ============================== EVEN-ODD-ODD

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, 1, -1,
         -1);

  // ============================== ODD-ODD-ODD

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, -1, -1,
         -1);
}
