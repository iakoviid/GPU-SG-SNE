#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include "common.cuh"
#include "utils_cuda.cuh"
#include <cmath>

/*Not optimazed for testing purposes*/
template <class dataPoint>
__global__ void Frep_exactKernel(volatile dataPoint *__restrict__ frep,
                            const dataPoint *const pointsX,
                            volatile dataPoint *__restrict__ zetaVec,
                            const int num_points, const int d) {

  register dataPoint Yi[3] = {0};
  register dataPoint Yj[3] = {0};
  register dataPoint dist = 0.0;
  for (register int i = threadIdx.x + blockIdx.x * blockDim.x; i < num_points;
       i += gridDim.x * blockDim.x) {
    for (int dd = 0; dd < d; dd++) {
      Yi[dd] = pointsX[i + dd * num_points];
    }
    for (int j = 0; j < num_points; j++) {

      if (i != j) {

        dist = 0.0;
        for (int dd = 0; dd < d; dd++) {
          Yj[dd] = pointsX[j + dd * num_points];
          dist += (Yj[dd] - Yi[dd]) * (Yj[dd] - Yi[dd]);
        }

        for (int dd = 0; dd < d; dd++) {
          frep[i + dd * num_points] +=
              (Yi[dd] - Yj[dd]) / ((1 + dist) * (1 + dist));
        }

        zetaVec[i] += 1.0 / (1.0 + dist);
      }
    }
  }
}
template <class dataPoint>
dataPoint computeFrepulsive_exact(dataPoint *frep, dataPoint *pointsX, int n,
                                  int d) {

  thrust::device_vector<dataPoint> zetaVec(n);
  int threads = 1024;
  int Blocks = 64;
  Frep_exactKernel<<<Blocks, threads>>>(
      frep, pointsX, thrust::raw_pointer_cast(zetaVec.data()), n, d);
  dataPoint z = thrust::reduce(zetaVec.begin(), zetaVec.end());
  ArrayScale<<<Blocks, threads>>>(frep, 1 / z, n * d);
  return z;
}

template <class dataPoint>
dataPoint computeFrepulsive_exactCPU(dataPoint * frep,dataPoint * pointsX,int N,int d){

  dataPoint *zetaVec = (dataPoint *) calloc( N, sizeof( dataPoint ) );

  for (int i = 0; i < N; i++) {
    dataPoint Yi[10] = {0};
    for (int dd = 0; dd < d; dd++ )
      Yi[dd] = pointsX[i + dd*N];

    dataPoint Yj[10] = {0};

    for(int j = 0; j < N; j++) {

      if(i != j) {

        dataPoint dist = 0.0;
        for (int dd = 0; dd < d; dd++ ){
           Yj[dd] = pointsX[j + dd*N];
           dist += (Yj[dd] - Yi[dd]) * (Yj[dd] - Yi[dd]);
        }

        for (int dd = 0; dd < d; dd++ ){
	  frep[i + dd*N] += (Yi[dd] - Yj[dd]) /
	       ( (1 + dist)*(1 + dist) );
	}

        zetaVec[i] += 1.0 / (1.0 + dist);

      }
    }
  }
dataPoint zeta=0;
for(int i=0;i<N;i++){zeta+=zetaVec[i];}

  for (int i = 0; i < N; i++) {
for(int j=0;j<d;j++){
    frep[i + j*N] /= zeta;
}
  }

  free( zetaVec );

  return zeta;

}

template <class dataPoint>
dataPoint computeErrorCPU(dataPoint* frep,dataPoint* y,int n,int d){
        dataPoint* freph=(dataPoint *)malloc(sizeof(dataPoint)*n*d);
        hipMemcpy(freph,frep,n*d*sizeof(dataPoint),hipMemcpyDeviceToHost);
        dataPoint* yh=(dataPoint *)malloc(sizeof(dataPoint)*n*d);
        hipMemcpy(yh,y,n*d*sizeof(dataPoint),hipMemcpyDeviceToHost);
        dataPoint* frept=(dataPoint *)calloc(n*d,sizeof(dataPoint));
        computeFrepulsive_exactCPU(frept,yh,n,d);
         dataPoint maxErr = 0;
         int pos=0;
         for (int i = 0; i<n*d; i++)
         {
           if(maxErr< abs( freph[i] - frept[i] )){
             pos=i;
             maxErr=abs( freph[i] - frept[i] );
           }
       }
       std::cout<<"MaxError= "<<maxErr<<" pos= "<<pos<<" freph= "<<freph[pos]<<" frept="<<frept[pos]<< "\n";
       std::cout<<"y "<<yh[pos]<<" , "<<yh[pos+n]<<"\n";
       std::cout<<"y "<<yh[0]<<" , "<<yh[n]<<"\n";
       std::cout<<"Frept= "<<frept[0]<<" , "<<frept[0+n]<<"\n";
       std::cout<<"Freph= "<<freph[0]<<" , "<<freph[0+n]<<"\n";

        free(freph);
        free(frept);
        free(yh);
return maxErr;
}


template <class dataPoint>
dataPoint computeErrorCPUrmse(dataPoint* frep,dataPoint* y,int n,int d){
        dataPoint* freph=(dataPoint *)malloc(sizeof(dataPoint)*n*d);
        hipMemcpy(freph,frep,n*d*sizeof(dataPoint),hipMemcpyDeviceToHost);
        dataPoint* yh=(dataPoint *)malloc(sizeof(dataPoint)*n*d);
        hipMemcpy(yh,y,n*d*sizeof(dataPoint),hipMemcpyDeviceToHost);
        dataPoint* frept=(dataPoint *)calloc(n*d,sizeof(dataPoint));
        computeFrepulsive_exactCPU(frept,yh,n,d);
         dataPoint errordiff = 0;
         dataPoint norm = 0;
        for (int i = 0; i<n*d; i++)
         {
           errordiff+=(frept[i]-freph[i])*(frept[i]-freph[i]);
           norm+=frept[i]*frept[i];
         }
         dataPoint Err=std::sqrt(errordiff)/std::sqrt(norm);
        std::cout<<"Err= "<<Err<<"\n";
        free(freph);
        free(frept);
        free(yh);
return Err;
}
template <class dataPoint>
__global__ void distance(dataPoint* x,dataPoint *y,int n, int d){

  for (register int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
       i += gridDim.x * blockDim.x) {
         dataPoint dist=0;
         for (int dd = 0; dd < d; dd++) {
           dist+=(x[i+dd*n]-y[i+dd*n])*(x[i+dd*n]-y[i+dd*n]);
         }
         x[i]=dist;
       }
}
template <class dataPoint>
__global__ void vectornorm(dataPoint* x,dataPoint *y,int n, int d){

  for (register int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
       i += gridDim.x * blockDim.x) {
         dataPoint dist=0;
         for (int dd = 0; dd < d; dd++) {
           dist+=(x[i+dd*n])*(x[i+dd*n]);
         }
         y[i]=dist;
       }
}
#include "utils.cuh"
template <class dataPoint>
dataPoint computeError(dataPoint *frep, dataPoint *y, int n, int d) {
  int threads = 1024;
  int Blocks = 64;
  thrust::device_vector<dataPoint> frep_exact(n * d);
  thrust::device_vector<dataPoint> normas(n);
  computeFrepulsive_exact(thrust::raw_pointer_cast(frep_exact.data()), y, n, d);
  hipDeviceSynchronize();
  vectornorm<<<64,1024>>>(thrust::raw_pointer_cast(frep_exact.data()), thrust::raw_pointer_cast(normas.data()), n, d);
  hipDeviceSynchronize();
  distance<<<Blocks,threads>>>(thrust::raw_pointer_cast(frep_exact.data()),frep,n,d);
 hipDeviceSynchronize();
  dataPoint error = thrust::reduce(frep_exact.begin(), frep_exact.begin()+n,0.0);
  dataPoint norm =thrust::reduce(normas.begin(), normas.begin()+n,0.0);
  error=std::sqrt(error)/std::sqrt(norm);
  
std::cout<<"Exit_Compute_Error "<<error <<"\n";
  return error;

}

     
