#include "hip/hip_runtime.h"
#include "sparsematrix.cuh"

/*

void free_sparse_matrix(sparse_matrix * P){
  free(P->row);
  free(P->col);
  free(P->val);

}
*/
void free_sparse_matrixGPU(sparse_matrix *P) {

  CUDA_CALL(hipFree(P->row));
  CUDA_CALL(hipFree(P->col));
  CUDA_CALL(hipFree(P->val));
}

__global__ void makeStochasticKernel(matval *val, matidx *col, matidx *row,
                                     uint32_t n, uint32_t *stoch) {
  const uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t warp_id = thread_id / 32;
  const uint32_t lane = thread_id % 32;
  __shared__ coord sdata[256 / 32];
  const unsigned int warp_id_block = threadIdx.x / 32;

  const unsigned int n_warps=gridDim.x*blockDim.x/32;
  for (uint32_t j=warp_id;j < n;j=j+n_warps) {
    matval sum = 0;
    for (uint32_t t = row[j]+lane; t < row[j + 1]; t += 32) {
      sum += val[t];
    }
    sum=warp_reduce(sum);
    if (lane == 0) {
      sdata[warp_id_block] = sum;
    }
    __syncwarp(FULL_WARP_MASK);
    sum = sdata[warp_id_block];

    if (fabs(sum - 1) > 1e-12) {
      for (uint32_t t = row[j]+lane; t < row[j + 1]; t += 32) {
        val[t] /= sum;
      }
      if (lane == 0) {
        stoch[j] = 0;
      }
    } else {
      if (lane == 0) {
        stoch[j] = 1;
      }
    }
  }
}

uint32_t makeStochasticGPU(sparse_matrix *P) {

  uint32_t *stoch;
  CUDA_CALL(hipMallocManaged(&stoch, P->n * sizeof(uint32_t)));

  makeStochasticKernel<<<256, 32>>>(P->val, P->col, P->row, P->n, stoch);

  uint32_t nStoch = thrust::reduce(stoch, stoch + P->n);

  CUDA_CALL(hipFree(stoch));
  return nStoch;
}

void permuteMatrixGPU(sparse_matrix *P, int *perm, int *iperm) {
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);
  hipsparseMatDescr_t descrA;
  hipsparseCreateMatDescr(&descrA);
  // Get sparse matrix
  matidx *row_P = P->row;
  matidx *col_P = P->col;
  matval *val_P = P->val;

  int N = P->n;
  matidx nnz = P->nnz;

  // Allocate memory for permuted matrix
  matval *perm_val_P = (matval *)malloc(nnz * sizeof(matval));
  CUDA_CALL(hipMallocManaged(&perm_val_P, nnz * sizeof(matval)));

  size_t pBufferSizeInBytes = 0;
  void *pBuffer = NULL;

  // step 1: allocate buffer
  hipsparseXcsrsort_bufferSizeExt(handle, N, N, nnz, col_P, row_P,
                                 &pBufferSizeInBytes);
  hipMalloc(&pBuffer, sizeof(char) * pBufferSizeInBytes);

  // step 3: sort CSR format
  hipsparseXcsrsort(handle, N, N, nnz, descrA, col_P, row_P, perm, pBuffer);

  hipsparseDgthr(handle, nnz, val_P, perm_val_P, perm, HIPSPARSE_INDEX_BASE_ZERO);

  hipFree(P->val);
  P->val = perm_val_P;
}
void add_cusparse(sparse_matrix A, sparse_matrix B, sparse_matrix *C,
                  hipsparseHandle_t handle) {
  // --- Initialize matrix descriptors
  hipsparseMatDescr_t descrA, descrB, descrC;
  hipsparseCreateMatDescr(&descrA);
  hipsparseCreateMatDescr(&descrB);
  hipsparseCreateMatDescr(&descrC);
  int M = A.m;
  int N = A.n;
  // --- Summing the two matrices
  int baseC, nnz3;
  // --- nnzTotalDevHostPtr points to host memory
  int *nnzTotalDevHostPtr = &nnz3;
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
  int *d_csrRowPtrC;
  hipMalloc(&d_csrRowPtrC, (M + 1) * sizeof(int));
  hipsparseXcsrgeamNnz(handle, M, N, descrA, A.nnz, A.row, A.col, descrB, B.nnz,
                      B.row, B.col, descrC, d_csrRowPtrC, nnzTotalDevHostPtr);
  if (NULL != nnzTotalDevHostPtr) {
    nnz3 = *nnzTotalDevHostPtr;
  } else {
    hipMemcpy(&nnz3, d_csrRowPtrC + M, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&baseC, d_csrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
    nnz3 -= baseC;
  }
  int *d_csrColIndC;
  hipMalloc(&d_csrColIndC, nnz3 * sizeof(int));
  matval *d_csrValC;
  hipMalloc(&d_csrValC, nnz3 * sizeof(matval));
  matval alpha = (matval)1 / 2, beta = (matval)1 / 2;
  hipsparseDcsrgeam(handle, M, N, &alpha, descrA, A.nnz, A.val, A.row, A.col,
                   &beta, descrB, B.nnz, B.val, B.row, B.col, descrC, d_csrValC,
                   d_csrRowPtrC, d_csrColIndC);

  C->row = d_csrRowPtrC;
  C->nnz = nnz3;
  C->col = d_csrColIndC;
  C->val = d_csrValC;
  C->m = M;
  C->n = N;
}

void symmetrizeMatrixGPU(sparse_matrix *A, sparse_matrix *C) {
  int M = A->m;
  int N = A->n;
  int nnz = A->nnz;
  int *csc_row_ptr_at;
  hipMalloc(reinterpret_cast<void **>(&csc_row_ptr_at),
             (A->nnz) * sizeof(int));
  int *csc_column_ptr_at;
  hipMalloc(reinterpret_cast<void **>(&csc_column_ptr_at),
             (A->n + 1) * sizeof(int));
  matval *csc_values_at;
  hipMalloc(reinterpret_cast<void **>(&csc_values_at),
             (A->nnz) * sizeof(matval));
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);
  // Do the transpose operation
  printf("nnz=%d n=%d\n",A->nnz,A->n );
  hipsparseDcsr2csc(handle, A->m, A->n, A->nnz, A->val, A->row, A->col,
                   csc_values_at, csc_row_ptr_at, csc_column_ptr_at,
                   HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO);

  hipDeviceSynchronize();

  hipsparseMatDescr_t descrA, descrB, descrC;
  hipsparseCreateMatDescr(&descrA);
  hipsparseCreateMatDescr(&descrB);
  hipsparseCreateMatDescr(&descrC);
  // --- Summing the two matrices
  int baseC, nnz3;
  // --- nnzTotalDevHostPtr points to host memory
  int *nnzTotalDevHostPtr = &nnz3;
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
  hipMalloc(&C->row, (M + 1) * sizeof(int));
  hipsparseXcsrgeamNnz(handle, M, N, descrA, A->nnz, A->row, A->col, descrB, nnz,
                      csc_column_ptr_at, csc_row_ptr_at, descrC, C->row,
                      nnzTotalDevHostPtr);
  if (NULL != nnzTotalDevHostPtr) {
    nnz3 = *nnzTotalDevHostPtr;
  } else {
    hipMemcpy(&nnz3, C->row + M, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&baseC, C->row, sizeof(int), hipMemcpyDeviceToHost);
    nnz3 -= baseC;
  }
  hipMalloc(&C->col, nnz3 * sizeof(int));
  hipMalloc(&C->val, nnz3 * sizeof(matval));
  matval alpha = (matval)1 / 2, beta = (matval)1 / 2;
  hipsparseDcsrgeam(handle, M, N, &alpha, descrA, nnz, A->val, A->row, A->col,
                   &beta, descrB, nnz, csc_values_at, csc_column_ptr_at,
                   csc_row_ptr_at, descrC, C->val, C->row, C->col);

  C->nnz = nnz3;
  C->m = M;
  C->n = N;
  hipDeviceSynchronize();
  printf("nnz=%d n=%d \n",C->nnz,C->n );

  hipFree(csc_column_ptr_at);
  hipFree(csc_row_ptr_at);
  hipFree(csc_values_at);
}
void tsnecuda::util::SymmetrizeMatrix(hipsparseHandle_t &handle,
        thrust::device_vector<float> &d_symmetrized_values,
        thrust::device_vector<int32_t> &d_symmetrized_rowptr,
        thrust::device_vector<int32_t> &d_symmetrized_colind,
        thrust::device_vector<float> &d_values,
        thrust::device_vector<int32_t> &d_indices,
        const float magnitude_factor,
        const int num_points,
        const int num_neighbors)
{

    // Allocate memory
    int32_t *csr_row_ptr_a = nullptr;
    hipMalloc(reinterpret_cast<void**>(&csr_row_ptr_a),
               (num_points+1)*sizeof(int32_t));
    int32_t *csr_column_ptr_a = thrust::raw_pointer_cast(d_indices.data());
    float *csr_values_a = thrust::raw_pointer_cast(d_values.data());

    // Copy the data
    thrust::device_vector<int> d_vector_memory(csr_row_ptr_a,
            csr_row_ptr_a+num_points+1);
    thrust::sequence(d_vector_memory.begin(), d_vector_memory.end(),
                     0, static_cast<int32_t>(num_neighbors));
    thrust::copy(d_vector_memory.begin(), d_vector_memory.end(), csr_row_ptr_a);
    hipDeviceSynchronize();

    // Initialize the matrix descriptor
    hipsparseMatDescr_t matrix_descriptor;
    hipsparseCreateMatDescr(&matrix_descriptor);
    hipsparseSetMatType(matrix_descriptor, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(matrix_descriptor, HIPSPARSE_INDEX_BASE_ZERO);

    // Sort the matrix properly
    size_t permutation_buffer_byte_size = 0;
    void *permutation_buffer = NULL;
    int32_t *permutation = NULL;

    // step 1: Allocate memory buffer
    hipsparseXcsrsort_bufferSizeExt(handle, num_points, num_points,
            num_points*num_neighbors, csr_row_ptr_a,
            csr_column_ptr_a, &permutation_buffer_byte_size);
    hipDeviceSynchronize();
    hipMalloc(&permutation_buffer,
               sizeof(char)*permutation_buffer_byte_size);

    // step 2: Setup permutation vector permutation to be the identity
    hipMalloc(reinterpret_cast<void**>(&permutation),
            sizeof(int32_t)*num_points*num_neighbors);
    hipsparseCreateIdentityPermutation(handle, num_points*num_neighbors,
                                      permutation);
    hipDeviceSynchronize();

    // step 3: Sort CSR format
    hipsparseXcsrsort(handle, num_points, num_points,
            num_points*num_neighbors, matrix_descriptor, csr_row_ptr_a,
            csr_column_ptr_a, permutation, permutation_buffer);
    hipDeviceSynchronize();

    // step 4: Gather sorted csr_values
    float* csr_values_a_sorted = nullptr;
    hipMalloc(reinterpret_cast<void**>(&csr_values_a_sorted),
            (num_points*num_neighbors)*sizeof(float));
    hipsparseSgthr(handle, num_points*num_neighbors, csr_values_a,
            csr_values_a_sorted, permutation, HIPSPARSE_INDEX_BASE_ZERO);
    hipDeviceSynchronize();

    // Free some memory
    hipFree(permutation_buffer);
    hipFree(permutation);
    csr_values_a = csr_values_a_sorted;

    // We need A^T, so we do a csr2csc() call
    int32_t* csc_row_ptr_at = nullptr;
    hipMalloc(reinterpret_cast<void**>(&csc_row_ptr_at),
            (num_points*num_neighbors)*sizeof(int32_t));
    int32_t* csc_column_ptr_at = nullptr;
    hipMalloc(reinterpret_cast<void**>(&csc_column_ptr_at),
            (num_points+1)*sizeof(int32_t));
    float* csc_values_at = nullptr;
    hipMalloc(reinterpret_cast<void**>(&csc_values_at),
            (num_points*num_neighbors)*sizeof(float));

    // Do the transpose operation
    hipsparseScsr2csc(handle, num_points, num_points,
                     num_neighbors*num_points, csr_values_a, csr_row_ptr_a,
                     csr_column_ptr_a, csc_values_at, csc_row_ptr_at,
                     csc_column_ptr_at, HIPSPARSE_ACTION_NUMERIC,
                     HIPSPARSE_INDEX_BASE_ZERO);
    hipDeviceSynchronize();

    // Now compute the output size of the matrix
    int32_t base_C, num_nonzeros_C;
    int32_t symmetrized_num_nonzeros = -1;
    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
    d_symmetrized_rowptr.resize(num_points+1);
    hipsparseXcsrgeamNnz(handle, num_points, num_points,
            matrix_descriptor, num_points*num_neighbors, csr_row_ptr_a,
                csr_column_ptr_a,
            matrix_descriptor, num_points*num_neighbors, csc_column_ptr_at,
                csc_row_ptr_at,
            matrix_descriptor,
            thrust::raw_pointer_cast(d_symmetrized_rowptr.data()),
            &symmetrized_num_nonzeros);
    hipDeviceSynchronize();

    // Do some useful checking...
    if (-1 != symmetrized_num_nonzeros) {
        num_nonzeros_C = symmetrized_num_nonzeros;
    } else {
        hipMemcpy(&num_nonzeros_C,
                thrust::raw_pointer_cast(d_symmetrized_rowptr.data()) +
                num_points, sizeof(int32_t), hipMemcpyDeviceToHost);
        hipMemcpy(&base_C,
                thrust::raw_pointer_cast(d_symmetrized_rowptr.data()),
                sizeof(int), hipMemcpyDeviceToHost);
    }

    // Allocate memory for the new summed array
    d_symmetrized_colind.resize(num_nonzeros_C);
    d_symmetrized_values.resize(num_nonzeros_C);

    // Sum the arrays
    float kAlpha = 1.0f / (2.0f * num_points);
    float kBeta = 1.0f / (2.0f * num_points);

    hipsparseScsrgeam(handle, num_points, num_points,
            &kAlpha, matrix_descriptor, num_points*num_neighbors,
            csr_values_a, csr_row_ptr_a, csr_column_ptr_a,
            &kBeta, matrix_descriptor, num_points*num_neighbors,
            csc_values_at, csc_column_ptr_at, csc_row_ptr_at,
            matrix_descriptor,
            thrust::raw_pointer_cast(d_symmetrized_values.data()),
            thrust::raw_pointer_cast(d_symmetrized_rowptr.data()),
            thrust::raw_pointer_cast(d_symmetrized_colind.data()));
    hipDeviceSynchronize();

    // Free the memory we were using...
    hipFree(csr_values_a);
    hipFree(csc_values_at);
    hipFree(csr_row_ptr_a);
    hipFree(csc_column_ptr_at);
    hipFree(csc_row_ptr_at);
}
