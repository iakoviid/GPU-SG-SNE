
#include <hip/hip_runtime.h>
extern hipStream_t streamAttr;

template <typename data_type, int d>
__global__ void
ell_spmv_kernel(const unsigned int n, const unsigned int elements_in_rows,
                const unsigned int *__restrict__ col_ids, const data_type *__restrict__ data,
                const data_type *__restrict__ Y, data_type *__restrict__ Fatr) {
  register unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
  register unsigned int column;
  register data_type sum1, sum2, sum3;
  register unsigned int element;
  register data_type dx, dy, dz;
  register data_type value,dist;

  if (row < n) {
    sum1 = 0;
    sum2 = 0;
    sum3 = 0;

    for (element = 0; element < elements_in_rows; element++) {
      const unsigned int element_offset = row + element * n;
      column = col_ids[element_offset];
      switch (d) {
      case 1:
        dx = Y[row] - Y[column];
        dist = dx * dx;
        value = data[element_offset] / (1 + dist);
        sum1 += value * dx;
        break;
      case 2:
        dx = Y[row] - Y[column];
        dy = Y[row + n] - Y[column + n];
        dist = dx * dx + dy * dy;
        value = data[element_offset] / (1 + dist);
        sum1 += value * dx;
        sum2 += value * dy;
        break;
      case 3:
        dx = Y[row] - Y[column];
        dy = Y[row + n] - Y[column + n];
        dz = Y[row + 2 * n] - Y[column + 2 * n];
        dist = dx * dx + dy * dy + dz * dz;
        value = data[element_offset] / (1 + dist);
        sum1 += value * dx;
        sum2 += value * dy;
        sum3 += value * dz;
        break;
      }
    }

    switch (d) {
    case 1:
      Fatr[row] = sum1;
      break;
    case 2:
      Fatr[row] = sum1;
      Fatr[row + n] = sum2;
      break;
    case 3:
      Fatr[row] = sum1;
      Fatr[row + n] = sum2;
      Fatr[row + 2 * n] = sum3;
      break;
    }
  }
}
template <typename data_type, int d>
__global__ void coo_spmv_kernel(const int n_elements,
                                const unsigned int *__restrict__ col_ids,
                                const unsigned int *__restrict__ row_ids,
                                const data_type *__restrict__ data,
                                const data_type *__restrict__ Y,
                                data_type *__restrict__ Fattr, const unsigned int n) {
  register unsigned int row, column;
  register data_type dist, pq;
  register unsigned int element;
  register data_type dx, dy, dz;
  for (element = blockIdx.x * blockDim.x + threadIdx.x; element < n_elements;
       element += blockDim.x * gridDim.x) {
    row = row_ids[element];
    column = col_ids[element];
    switch (d) {
    case 1:
      dx = (Y[row] - Y[column]);
      dist = dx * dx;
      pq = data[element] / (1 + dist);
      atomicAdd(Fattr + row, pq * dx);
      break;
    case 2:
      dx = (Y[row] - Y[column]);
      dy = (Y[row + n] - Y[column + n]);
      dist = dx * dx + dy * dy;
      pq = data[element] / (1 + dist);
      atomicAdd(Fattr + row, pq * dx);
      atomicAdd(Fattr + row + n, pq * dy);
      break;
    case 3:
      dx = (Y[row] - Y[column]);
      dy = (Y[row + n] - Y[column + n]);
      dz = (Y[row + 2 * n] - Y[column + 2 * n]);
      dist = dx * dx + dy * dy + dz * dz;
      pq = data[element] / (1 + dist);
      atomicAdd(Fattr + row, pq * dx);
      atomicAdd(Fattr + row + n, pq * dy);
      atomicAdd(Fattr + row + 2 * n, pq * dz);
      break;
    }
  }
}

template <typename data_type>
void gpu_hybrid_spmv(int elements_in_rows, int coo_size, data_type *Y,
                     unsigned int rows_count, data_type *F,
                     unsigned int *ell_cols, data_type *ell_data,
                     data_type *coo_data, unsigned int *coo_row_ids,
                     unsigned int *coo_col_ids, int d) {

  /// ELL Part
  {
    dim3 block_size = dim3(512);
    dim3 grid_size{};

    grid_size.x = (rows_count + block_size.x - 1) / block_size.x;
    switch (d) {
    case 1:
      ell_spmv_kernel<data_type, 1><<<grid_size, block_size, 0, streamAttr>>>(
          rows_count, elements_in_rows, ell_cols, ell_data, Y, F);
      break;
    case 2:
      ell_spmv_kernel<data_type, 2><<<grid_size, block_size, 0, streamAttr>>>(
          rows_count, elements_in_rows, ell_cols, ell_data, Y, F);
      break;
    case 3:
      ell_spmv_kernel<data_type, 3><<<grid_size, block_size, 0, streamAttr>>>(
          rows_count, elements_in_rows, ell_cols, ell_data, Y, F);
      break;
    }
  }

  /// COO Part
  {
    dim3 block_size = dim3(512);
    dim3 grid_size{};

    const int n_elements = coo_size;
    grid_size.x = (n_elements + block_size.x - 1) / block_size.x;

    switch (d) {
    case 1:
      coo_spmv_kernel<data_type, 1><<<grid_size, block_size, 0, streamAttr>>>(
          n_elements, coo_col_ids, coo_row_ids, coo_data, Y, F, rows_count);
      break;
    case 2:
      coo_spmv_kernel<data_type, 2><<<grid_size, block_size, 0, streamAttr>>>(
          n_elements, coo_col_ids, coo_row_ids, coo_data, Y, F, rows_count);
      break;
    case 3:
      coo_spmv_kernel<data_type, 3><<<grid_size, block_size, 0, streamAttr>>>(
          n_elements, coo_col_ids, coo_row_ids, coo_data, Y, F, rows_count);
      break;
    }
  }
}
