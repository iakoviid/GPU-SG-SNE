#include "hip/hip_runtime.h"
#include "prepareMatrix.cuh"
#include "sparse_reorder.cuh"
__global__ void Csr2CooKernel(volatile int *__restrict__ coo_indices,
                              const int *__restrict__ pij_row_ptr,
                              const int *__restrict__ pij_col_ind,
                              const int num_points, const int num_nonzero) {
  register int TID, i, j, start, end;
  TID = threadIdx.x + blockIdx.x * blockDim.x;
  if (TID >= num_nonzero)
    return;
  start = 0;
  end = num_points + 1;
  i = (num_points + 1) >> 1;
  while (end - start > 1) {
    j = pij_row_ptr[i];
    end = (j > TID) ? i : end;
    start = (j <= TID) ? i : start;
    i = (start + end) >> 1;
  }
  coo_indices[TID] = i;
}
void Csr2Coo(int nnz, int n, int *row, int *col, int *coo_indices) {
  const int num_threads = 1024;
  const int num_blocks = iDivUp(nnz, num_threads);

  Csr2CooKernel<<<num_blocks, num_threads>>>(coo_indices, row, col, n, nnz);
  hipDeviceSynchronize();
}

void Csr2Coo(sparse_matrix<float> *P) {
  const int num_threads = 1024;
  const int num_blocks = iDivUp(P->nnz, num_threads);
  int *coo_indices;
  hipMalloc((void **)&coo_indices, sizeof(int) * (P->nnz));

  Csr2CooKernel<<<num_blocks, num_threads>>>(coo_indices, P->row, P->col, P->n,
                                             P->nnz);
  hipDeviceSynchronize();
  hipFree(P->row);
  P->row = coo_indices;
}
void Csr2Coo(sparse_matrix<float> **P) {
  const int num_threads = 1024;
  const int num_blocks = iDivUp((*P)->nnz, num_threads);
  int *coo_indices;
  hipMalloc((void **)&coo_indices, sizeof(int) * ((*P)->nnz));

  Csr2CooKernel<<<num_blocks, num_threads>>>(coo_indices, (*P)->row, (*P)->col,
                                             (*P)->n, (*P)->nnz);
  hipDeviceSynchronize();
  hipFree((*P)->row);
  (*P)->row = coo_indices;
}
void csr2bsr(int blockDim, int n, int m, int nnz, int *csrRowptr,
             int *csrColInd, float *csrVal, int **bsrRowPtr, int **bsrColInd,
             float **bsrVal, int *nnzblocks, int *n_block_rows,
             hipsparseHandle_t handle) {

  int *csrRowPtrA, *csrColIndA;
  float *csrValA;
  hipMalloc((void **)&csrRowPtrA, sizeof(int) * (m + 1));
  hipMalloc((void **)&csrColIndA, sizeof(int) * nnz);
  hipMalloc((void **)&csrValA, sizeof(float) * nnz);
  hipMemcpy(csrValA, csrVal, nnz * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(csrColIndA, csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(csrRowPtrA, csrRowptr, (m + 1) * sizeof(int),
             hipMemcpyHostToDevice);

  hipsparseMatDescr_t descr = 0;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;
  int base, nnzb;
  int mb = (m + blockDim - 1) / blockDim;
  hipMalloc(bsrRowPtr, sizeof(int) * (mb + 1));
  // nnzTotalDevHostPtr points to host memory
  int *nnzTotalDevHostPtr = &nnzb;
  hipsparseXcsr2bsrNnz(handle, dir, m, n, descr, csrRowPtrA, csrColIndA,
                      blockDim, descr, *bsrRowPtr, nnzTotalDevHostPtr);
  if (NULL != nnzTotalDevHostPtr) {
    nnzb = *nnzTotalDevHostPtr;
  } else {
    hipMemcpy(&nnzb, *bsrRowPtr + mb, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&base, *bsrRowPtr, sizeof(int), hipMemcpyDeviceToHost);
    nnzb -= base;
  }
  hipMalloc(bsrColInd, sizeof(int) * nnzb);
  hipMalloc(bsrVal, sizeof(float) * (blockDim * blockDim) * nnzb);
  hipsparseScsr2bsr(handle, dir, m, n, descr, csrValA, csrRowPtrA, csrColIndA,
                   blockDim, descr, *bsrVal, *bsrRowPtr, *bsrColInd);
  *nnzblocks = nnzb;
  *n_block_rows = mb;
  hipFree(csrRowPtrA);
  hipFree(csrColIndA);
  hipFree(csrValA);
}
void csr2bsr(int blockDim, int n, int m, int nnz, int *csrRowptr,
             int *csrColInd, double *csrVal, int **bsrRowPtr, int **bsrColInd,
             double **bsrVal, int *nnzblocks, int *n_block_rows,
             hipsparseHandle_t handle) {

  int *csrRowPtrA, *csrColIndA;
  double *csrValA;
  hipMalloc((void **)&csrRowPtrA, sizeof(int) * (m + 1));
  hipMalloc((void **)&csrColIndA, sizeof(int) * nnz);
  hipMalloc((void **)&csrValA, sizeof(double) * nnz);
  hipMemcpy(csrValA, csrVal, nnz * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(csrColIndA, csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(csrRowPtrA, csrRowptr, (m + 1) * sizeof(int),
             hipMemcpyHostToDevice);

  hipsparseMatDescr_t descr = 0;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;
  int base, nnzb;
  int mb = (m + blockDim - 1) / blockDim;
  hipMalloc(bsrRowPtr, sizeof(int) * (mb + 1));
  // nnzTotalDevHostPtr points to host memory
  int *nnzTotalDevHostPtr = &nnzb;
  hipsparseXcsr2bsrNnz(handle, dir, m, n, descr, csrRowPtrA, csrColIndA,
                      blockDim, descr, *bsrRowPtr, nnzTotalDevHostPtr);
  if (NULL != nnzTotalDevHostPtr) {
    nnzb = *nnzTotalDevHostPtr;
  } else {
    hipMemcpy(&nnzb, *bsrRowPtr + mb, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&base, *bsrRowPtr, sizeof(int), hipMemcpyDeviceToHost);
    nnzb -= base;
  }
  hipMalloc(bsrColInd, sizeof(int) * nnzb);
  hipMalloc(bsrVal, sizeof(double) * (blockDim * blockDim) * nnzb);
  hipsparseDcsr2bsr(handle, dir, m, n, descr, csrValA, csrRowPtrA, csrColIndA,
                   blockDim, descr, *bsrVal, *bsrRowPtr, *bsrColInd);
  *nnzblocks = nnzb;
  *n_block_rows = mb;
  hipFree(csrRowPtrA);
  hipFree(csrColIndA);
  hipFree(csrValA);
}
#include "matrix_converter.h"

/*
template <typename data_type>
__global__ void csr2ell(volatile unsigned int *__restrict__ data,
                        volatile data_type *__restrict__ columns,
                        const int *const row_ptr, const int *const col,
                        const data_type *const val, const int n,
                        const int elements_per_ell) {
  register int start;
  register int end;

  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < n;
       TID += gridDim.x * blockDim.x) {
    start = row_ptr[TID];
    end = row_ptr[TID + 1];
    for (register int element = start;
         element < min(start + elements_per_ell, end); element++) {
      data[TID + (element - start) * n] = val[element];
      columns[TID + (element - start) * n] = col[element];
    }
  }
}
template <typename data_type>
__global__ void coorowsize(volatile int *__restrict__ cooElementsRow,
                           const int *const row_ptr, const int *const col,
                           const data_type *const val, const int n,
                           const int elements_per_ell) {
  register int start;
  register int end;
  register int row_size for (register int TID =
                                 threadIdx.x + blockIdx.x * blockDim.x;
                             TID < n; TID += gridDim.x * blockDim.x) {
    start = row_ptr[TID];
    end = row_ptr[TID + 1];
    row_size = end - start;
    if (row_size > elements_per_ell) {
      cooElementsRow[TID] = row_size - elements_per_ell;
    } else {
      cooElementsRow[TID] = 0;
    }
  }
}
template <typename data_type>
__global__ void Csr2Coo(data_type *coo_data, int *coo_col_ids, int *coo_row_ids,
                        const int elements_per_ell, int *row_ptr, int *col,
                        data_type *val, const int n) {
  register int start;
  register int end;

  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < n;
       TID += gridDim.x * blockDim.x) {
    start = row_ptr[TID];
    end = row_ptr[TID + 1];

    for (register int element = elements_per_ell; element < end - start;
         element++) {
    }
  }
}

template <typename data_type> void PrepareHybrid(sparse_matrix<data_type> *P) {
  int elements_per_ell = P->nnz / P->n;
  CUDA_CALL(
      hipMalloc(&P->ell_data, P->n * elements_per_ell * sizeof(data_type)));
  CUDA_CALL(
      hipMalloc(&P->ell_cols, P->n * elements_per_ell * sizeof(unsigned int)));
  initKernel<<<64, 1024>>>(P->ell_data, (data_type)0, P->n * elements_per_ell);
  initKernel<<<64, 1024>>>(P->ell_cols, (int)0, P->n * elements_per_ell);
  hipDeviceSynchronize();
  csr2ell(P->ell_data, P->ell_cols, P->row, P->col, P->val, P->n,
          elements_per_ell);
  thrust::device_vector<int> cooElementsRow(n);
  coorowsize(thrust::raw_pointer_cast(cooElementsRow.data()), P->row, P->col,
             P->val, P->n, elements_per_ell);
  hipDeviceSynchronize();
  int coo_size =
      thrust::transform_reduce(cooElementsRow.begin(), cooElementsRow.end());

  CUDA_CALL(hipMalloc(&P->coo_data, coo_size * sizeof(data_type)));
  CUDA_CALL(hipMalloc(&P->coo_col_ids, coo_size * sizeof(unsigned int)));
  CUDA_CALL(hipMalloc(&P->coo_row_ids, coo_size * sizeof(unsigned int)));
  Csr2Coo(P->coo_data, P->coo_col_ids, P->coo_row_ids, elements_per_ell, P->row,
          P->col, P->val, n);
  P->coo_size = coo_size;
  P->elements_in_rows = elements_per_ell;
}
*/
template <typename data_type>
void PrepareHybrid(sparse_matrix<data_type> *P) {
  int nnz, rows;
  nnz = P->nnz;
  rows = P->n;
  csr_matrix_class<data_type> A;
  A.nnz = nnz;
  A.n = rows;
  A.data.reset(new data_type[nnz]);
  A.columns.reset(new unsigned int[nnz]);
  A.row_ptr.reset(new unsigned int[rows + 1]);
  hipMemcpy(A.data.get(), P->val, nnz * sizeof(data_type),
             hipMemcpyDeviceToHost);
  hipMemcpy(A.row_ptr.get(), P->row, (rows + 1) * sizeof(int),
             hipMemcpyDeviceToHost);
  hipMemcpy(A.columns.get(), P->col, nnz * sizeof(int),
             hipMemcpyDeviceToHost);
  hybrid_matrix_class<data_type> D(A);
  D.allocate(A, 0.001);
  const size_t A_size = D.ell_matrix->get_matrix_size();
  const size_t col_ids_size = A_size;
  CUDA_CALL(hipMalloc(&P->ell_data, A_size * sizeof(data_type)));
  CUDA_CALL(hipMalloc(&P->ell_cols, A_size * sizeof(unsigned int)));
  hipMemcpy(P->ell_data, D.ell_matrix->data.get(), A_size * sizeof(data_type),
             hipMemcpyHostToDevice);
  hipMemcpy(P->ell_cols, D.ell_matrix->columns.get(),
             col_ids_size * sizeof(unsigned int), hipMemcpyHostToDevice);

  const size_t coo_size = D.coo_matrix->get_matrix_size();
  CUDA_CALL(hipMalloc(&P->coo_data, coo_size * sizeof(data_type)));
  CUDA_CALL(hipMalloc(&P->coo_col_ids, coo_size * sizeof(unsigned int)));
  CUDA_CALL(hipMalloc(&P->coo_row_ids, coo_size * sizeof(unsigned int)));

  hipMemcpy(P->coo_data, D.coo_matrix->data.get(),
             coo_size * sizeof(data_type), hipMemcpyHostToDevice);
  hipMemcpy(P->coo_col_ids, D.coo_matrix->cols.get(),
             coo_size * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(P->coo_row_ids, D.coo_matrix->rows.get(),
             coo_size * sizeof(unsigned int), hipMemcpyHostToDevice);
  P->coo_size = coo_size;
  P->elements_in_rows = D.ell_matrix->elements_in_rows;

}
template <typename data_type>
sparse_matrix<data_type> *PrepareHybrid(int nnz, int rows, int *csrCidx,
                                        int *csrRptr, data_type *csrVal) {
  sparse_matrix<data_type> *Pd =
      (sparse_matrix<data_type> *)malloc(sizeof(sparse_matrix<data_type>));
  csr_matrix_class<data_type> A;
  A.nnz = nnz;
  A.n = rows;
  A.data.reset(new data_type[nnz]);
  A.columns.reset(new unsigned int[nnz]);
  A.row_ptr.reset(new unsigned int[rows + 1]);
  for (int i = 0; i < nnz; i++) {
    A.columns[i] = (unsigned int)csrCidx[i];
    A.data[i] = csrVal[i];
  }
  for (int i = 0; i < rows + 1; i++) {
    A.row_ptr[i] = csrRptr[i];
  }
  hybrid_matrix_class<data_type> D(A);
  D.allocate(A, 0.001);
  const size_t A_size = D.ell_matrix->get_matrix_size();
  const size_t col_ids_size = A_size;
  CUDA_CALL(hipMalloc(&Pd->ell_data, A_size * sizeof(data_type)));
  CUDA_CALL(hipMalloc(&Pd->ell_cols, A_size * sizeof(unsigned int)));
  hipMemcpy(Pd->ell_data, D.ell_matrix->data.get(), A_size * sizeof(data_type),
             hipMemcpyHostToDevice);
  hipMemcpy(Pd->ell_cols, D.ell_matrix->columns.get(),
             col_ids_size * sizeof(unsigned int), hipMemcpyHostToDevice);

  const size_t coo_size = D.coo_matrix->get_matrix_size();
  CUDA_CALL(hipMalloc(&Pd->coo_data, coo_size * sizeof(data_type)));
  CUDA_CALL(hipMalloc(&Pd->coo_col_ids, coo_size * sizeof(unsigned int)));
  CUDA_CALL(hipMalloc(&Pd->coo_row_ids, coo_size * sizeof(unsigned int)));

  hipMemcpy(Pd->coo_data, D.coo_matrix->data.get(),
             coo_size * sizeof(data_type), hipMemcpyHostToDevice);
  hipMemcpy(Pd->coo_col_ids, D.coo_matrix->cols.get(),
             coo_size * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(Pd->coo_row_ids, D.coo_matrix->rows.get(),
             coo_size * sizeof(unsigned int), hipMemcpyHostToDevice);
  Pd->n = rows;
  Pd->nnz = nnz;
  Pd->coo_size = coo_size;
  Pd->elements_in_rows = D.ell_matrix->elements_in_rows;

  return Pd;
}
template <typename data_type>
sparse_matrix<data_type> *PrepareSparseMatrix(sparse_matrix<data_type> *P,
                                              int *perm, int format,
                                              const char *method, int bs) {
  sparse_matrix<data_type> *Pd =
      (sparse_matrix<data_type> *)malloc(sizeof(sparse_matrix<data_type>));
  int N = P->n;
  int M = P->m;
  int n = P->n;
  int nnz = P->nnz;
  int mb = 0;
  int nnzb = 0;
  for (int i = 0; i < n; i++) {
    perm[i] = i;
  }
  if (format == 0) {
    CUDA_CALL(hipMallocManaged(&Pd->col, nnz * sizeof(matidx)));
    CUDA_CALL(hipMallocManaged(&Pd->val, nnz * sizeof(data_type)));
    CUDA_CALL(hipMallocManaged(&Pd->row, (n + 1) * sizeof(matidx)));

    hipMemcpy(Pd->col, P->row, nnz * sizeof(matidx), hipMemcpyHostToDevice);
    hipMemcpy(Pd->val, P->val, nnz * sizeof(data_type),
               hipMemcpyHostToDevice);
    hipMemcpy(Pd->row, P->col, (n + 1) * sizeof(matidx),
               hipMemcpyHostToDevice);
  } else if (format == 2) {
    // csr to coo
    CUDA_CALL(hipMallocManaged(&Pd->col, nnz * sizeof(matidx)));
    CUDA_CALL(hipMallocManaged(&Pd->val, nnz * sizeof(data_type)));
    CUDA_CALL(hipMallocManaged(&Pd->row, nnz * sizeof(matidx)));
    matidx *coorow = (matidx *)malloc(sizeof(matidx) * nnz);
    for (int i = 0; i < n; i++) {
      for (int j = P->col[i]; j < P->col[i + 1]; j++) {
        coorow[j] = i;
      }
    }
    hipMemcpy(Pd->col, P->row, nnz * sizeof(matidx), hipMemcpyHostToDevice);
    hipMemcpy(Pd->val, P->val, nnz * sizeof(data_type),
               hipMemcpyHostToDevice);
    hipMemcpy(Pd->row, coorow, nnz * sizeof(matidx), hipMemcpyHostToDevice);
    free(coorow);
  } else if (format == 1) {
    data_type *csr_val_permuted =
        (data_type *)calloc(P->nnz, sizeof(data_type));
    int *csr_col_permuted = (int *)calloc(P->nnz, sizeof(int));
    int *csr_row_permuted = (int *)calloc(P->n + 1, sizeof(int));
    hipsolverSpHandle_t handle = NULL;
    hipsparseHandle_t cusparseHandle = NULL; /* used in residual evaluation */

    hipStream_t stream = NULL;
    hipsparseMatDescr_t descrA = NULL;
    checkCudaErrors(hipsolverSpCreate(&handle));
    checkCudaErrors(hipsparseCreate(&cusparseHandle));

    checkCudaErrors(hipStreamCreate(&stream));
    /* bind stream to cusparse and cusolver*/
    checkCudaErrors(hipsolverSpSetStream(handle, stream));
    checkCudaErrors(hipsparseSetStream(cusparseHandle, stream));

    /* configure matrix descriptor*/
    checkCudaErrors(hipsparseCreateMatDescr(&descrA));
    checkCudaErrors(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    int baseA = 0; /* base index in CSR format */
    if (baseA) {
      checkCudaErrors(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));
    } else {
      checkCudaErrors(
          hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));
    }
    SparseReorder<data_type>(method, handle, descrA, P->m, P->n, P->nnz, P->col,
                             P->row, P->val, csr_row_permuted, csr_col_permuted,
                             csr_val_permuted, perm);
    // csr2bsr(bs, N, M, nnz,  P->col,  P->row, P->val, &Pd->row, &Pd->col,
    // &Pd->val,&nnzb, &mb, cusparseHandle);
    csr2bsr(bs, N, M, nnz, csr_row_permuted, csr_col_permuted, csr_val_permuted,
            &Pd->row, &Pd->col, &Pd->val, &nnzb, &mb, cusparseHandle);
    free(csr_val_permuted);
    free(csr_col_permuted);
    free(csr_row_permuted);
  } else if (format == 3) {
    Pd = PrepareHybrid<data_type>(P->nnz, P->n, P->row, P->col, P->val);
  }
  Pd->n = n;
  Pd->nnz = nnz;
  Pd->blockSize = bs;
  Pd->blockRows = mb;
  Pd->nnzb = nnzb;
  Pd->format = format;
  return Pd;
}
template void PrepareHybrid(sparse_matrix<float> *P);
template void PrepareHybrid(sparse_matrix<double> *P);

template sparse_matrix<float> *PrepareSparseMatrix(sparse_matrix<float> *P,
                                                   int *perm, int format,
                                                   const char *method, int bs);
template sparse_matrix<double> *PrepareSparseMatrix(sparse_matrix<double> *P,
                                                    int *perm, int format,
                                                    const char *method, int bs);
