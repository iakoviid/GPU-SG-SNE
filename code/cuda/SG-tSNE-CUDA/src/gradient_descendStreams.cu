#include "hip/hip_runtime.h"
#include "compute_error.cu"
#include "gradient_descend.cuh"
#include <fstream>
#include <math.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
hipStream_t streamAttr = 0;
hipStream_t streamRep = 0;
using namespace std;
template <class dataPoint>
__global__ void compute_dy(volatile dataPoint *__restrict__ dy,const dataPoint * const Fattr,const dataPoint * const Frep,
                          const int n,const int d,const dataPoint alpha) {
  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < n * d;
       TID += gridDim.x * blockDim.x) {
    dy[TID] = (alpha * Fattr[TID]) - Frep[TID];
  }
}

template <class dataPoint>
__device__ __host__ static inline dataPoint sign(dataPoint x) {

  return (x == .0 ? .0 : (x < .0 ? -1.0 : 1.0));
}

template <class dataPoint>
__global__ void gradient_update(const dataPoint *const dY,volatile dataPoint * __restrict__ uY,const int N,
                                const int no_dims,volatile dataPoint *__restrict__ Y, volatile dataPoint *__restrict__ gains,
                                const dataPoint momentum,const dataPoint eta) {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N * no_dims;
       i += gridDim.x * blockDim.x) {
    gains[i] = (sign(dY[i]) != sign(uY[i])) ? (gains[i] + .2) : (gains[i] * .8);
    if (gains[i] < .01)
      gains[i] = .01;
    uY[i] = momentum * uY[i] - eta * gains[i] * dY[i];
    Y[i] = Y[i] + uY[i];
  }
}

template <class dataPoint>
void update_positions(dataPoint *dY, dataPoint *uY, int n, int d, dataPoint* Y,
                      dataPoint *gains, dataPoint momentum, dataPoint eta) {

  gradient_update<<<64, 1024>>>(dY, uY, n, d, Y, gains, momentum, eta);
}

template <class dataPoint, class Complext>
dataPoint compute_gradient(dataPoint *Fattr, dataPoint *Frep,
    dataPoint *dy, double *timeFrep, double *timeFattr, tsneparams params,
    dataPoint *y, sparse_matrix<dataPoint> P, double *TotaltimeInfo,
    hipfftHandle &plan, hipfftHandle &plan_rhs, int nGrid, dataPoint *yt,
    dataPoint *VScat, dataPoint *PhiScat, dataPoint *VGrid, dataPoint *PhiGrid,
    Complext *Kc, Complext *Xc, thrust::device_vector<dataPoint> &zetaVec,
     dataPoint *errorRep = nullptr) {
   double timeInfo[7] = {0};
  // ----- parse input parameters
  int d = params.d;
  int n = params.n;
  // ----- timing
  struct GpuTimer timer;

  // ------ Compute PQ (fattr)
  timer.Start();
  AttractiveEstimation(P, d, y, Fattr);
  timer.Stop();
  *timeFattr = timer.Elapsed();

  hipDeviceSynchronize();

  // ------ Compute QQ (frep)
  timer.Start();
//  dataPoint* yh=(dataPoint*)malloc(sizeof(dataPoint)*n*d);
//CUDA_CALL(hipMemcpy(yh, y, n * d * sizeof(dataPoint),hipMemcpyDeviceToHost));
 dataPoint zeta = computeFrepulsive_interp(
      Frep, y, n, d, (dataPoint)params.h, timeInfo, nGrid, plan, plan_rhs, yt,
      VScat, PhiScat, VGrid, PhiGrid, Kc, Xc, zetaVec);
//  dataPoint* Freph=(dataPoint *)malloc(sizeof(dataPoint)*n*d);

//CUDA_CALL(hipMemcpy(Freph, Frep, n * d * sizeof(dataPoint),hipMemcpyDeviceToHost));
 // free(yh);
  //free(Freph);

  timer.Stop();
  *timeFrep += timer.Elapsed();
  timeInfo[0]=*timeFattr;
  for (int i = 0; i < 7; i++) {TotaltimeInfo[i] = timeInfo[i];}

  // ----- Compute gradient (dY)
  hipDeviceSynchronize();
  compute_dy<<<64, 1024>>>(dy, Fattr, Frep, n, d, (dataPoint)params.alpha);
hipDeviceSynchronize();
  if (params.ComputeError > 0 && errorRep != nullptr) {
    *errorRep = computeError(Frep, y, n, d);
  }



  return zeta;
}
void kl_minimization(float *y, tsneparams params, sparse_matrix<float> P,double* timeInfo) {

  // ----- t-SNE hard coded parameters - Same as in vdM's code
  int stop_lying_iter = params.earlyIter, mom_switch_iter = 250;
  float momentum = .5, final_momentum = .8;
  float eta = 200.0;
  // int iterPrint = 50;
  // hipStreamCreate(&streamAttr);
  // hipStreamCreate(&streamRep);
  double timeFattr = 0.0;
  double timeFrep = 0.0;
  double timeUpdate=0.0;
  int n = params.n;
  int d = params.d;
  int max_iter = params.maxIter;

  float zeta = 0;
  float *errorRep = nullptr;
  int errorCalcs = (int)max_iter / 10;
  if (params.ComputeError > 0) {
    errorRep = (float *)malloc(sizeof(float) * errorCalcs);
  }
  // ----- Allocate memory
  float *dy;
  float *uy;
  float *gains;
  CUDA_CALL(hipMallocManaged(&dy, d * n * sizeof(float)));
  CUDA_CALL(hipMallocManaged(&uy, d * n * sizeof(float)));
  CUDA_CALL(hipMallocManaged(&gains, d * n * sizeof(float)));

  /*-------Initialize-----*/
  initKernel<<<64, 1024>>>(uy, (float)0.0, n * d);
  initKernel<<<64, 1024>>>(gains, (float)1.0, n * d);

  //double timeInfo[1000 * 7] = {0};

  hipfftHandle plan, plan_rhs;

  int n1 = getBestGridSize(params.ng);
  int n2 = n1 + 2;
  switch (d) {
  case 1: {
    int ng[1] = {(int)n2};
    hipfftPlan1d(&plan, n2, HIPFFT_C2C, 1);
    hipfftPlanMany(&plan_rhs, 1, ng, NULL, 1, n2, NULL, 1, n2, HIPFFT_C2C, d + 1);
    break;
  }
  case 2: {
        int ng[2] = {(int)n2, (int)n2};
    int *inembed = NULL, *onembed = NULL;
      hipfftPlan2d(&plan, n2, n2, HIPFFT_C2C);
//hipfftPlanMany(&plan, 2, ng, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, 1);
    hipfftPlanMany(&plan_rhs, 2, ng, inembed, 1, n2 * n2, onembed, 1, n2 * n2,
                  HIPFFT_C2C, d + 1);
    break;
  }
  case 3: {
    int ng[3] = {(int)n2, (int)n2, (int)n2};
    hipfftPlanMany(&plan, 3, ng, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, 1);
    hipfftPlanMany(&plan_rhs, 3, ng, NULL, 1, n2 * n2 * n2, NULL, 1,
                  n2 * n2 * n2, HIPFFT_C2C, d + 1);
    break;
  }
  }
  // hipfftSetStream(plan, streamRep);
  // hipfftSetStream(plan_rhs, streamRep);
  int m = d + 1;
  int nVec = d + 1;
  /*Allocate memory*/
  float *yt;
  CUDA_CALL(hipMallocManaged(&yt, (d)*n * sizeof(float)));
  float *VScat;
  CUDA_CALL(hipMallocManaged(&VScat, (d + 1) * n * sizeof(float)));
  float *PhiScat;
  CUDA_CALL(hipMallocManaged(&PhiScat, (d + 1) * n * sizeof(float)));
  int szV = pow(n1 + 2, d) * m;
  float *VGrid;
  CUDA_CALL(hipMallocManaged(&VGrid, szV * sizeof(float)));
  float *PhiGrid;
  CUDA_CALL(hipMallocManaged(&PhiGrid, szV * sizeof(float)));
  Complex *Kc, *Xc;
  CUDA_CALL(hipMallocManaged(&Kc, szV * sizeof(Complex)));
  CUDA_CALL(hipMallocManaged(&Xc, nVec * szV * sizeof(Complex)));
  thrust::device_vector<float> zetaVec(n);
  float *Fattr;
  float *Frep;
  CUDA_CALL(hipMallocManaged(&Fattr, n * d * sizeof(float)));
  CUDA_CALL(hipMallocManaged(&Frep, n * d * sizeof(float)));
  struct timeval t1, t2;
  gettimeofday(&t1, NULL);
  struct GpuTimer timer;
  float KL[100]={0};
  for (int iter = 0; iter < max_iter; iter++) {

    initKernel<<<64, 1024>>>(Fattr, (float)0.0, n * d);
    initKernel<<<64, 1024, 0, streamRep>>>(VGrid, (float)0, szV);
    initKernel<<<64, 1024, 0, streamRep>>>(PhiGrid, (float)0, szV);
hipDeviceSynchronize();
    if (iter % 100 == 0) {
      printf("---------------------------%d---------------------------\n",
             iter);
//      appendProgressGPU(y, n,d,"sg_dump.txt");
	std::cout<<"Zeta= "<<zeta<<"\n";

    }

    if (params.ComputeError > 0 && iter % 10 == 0 ) {
      zeta = compute_gradient(Fattr,Frep,dy, &timeFrep, &timeFattr, params, y, P,
                              &timeInfo[7*iter], plan, plan_rhs, n1, yt,
                              VScat, PhiScat, VGrid, PhiGrid, Kc, Xc, zetaVec,
                              &errorRep[(int)iter / 10]);
    } else {
      zeta = compute_gradient(Fattr,Frep,dy, &timeFrep, &timeFattr, params, y, P,
                              &timeInfo[7*iter], plan, plan_rhs, n1, yt,
                              VScat, PhiScat, VGrid, PhiGrid, Kc, Xc, zetaVec);
    }
/*
   if(iter%10==0){ KL[iter/10]=tsneCost(P,y,  n,d,params.alpha, zeta );
 std::cout<<"Error : "<<KL[iter/10]<<"\n";
}*/
    timer.Start();
    update_positions(dy, uy, n, d, y, gains, momentum, eta);
    timer.Stop();
    timeUpdate += timer.Elapsed();

    // Stop lying about the P-values after a while, and switch momentum
    if (iter == stop_lying_iter) {
      params.alpha = 1;
    }

    // Change momentum after a while
    if (iter == mom_switch_iter) {
      momentum = final_momentum;
    }
  }
thrust::device_ptr<float> yVec_ptr(y);
float meany[4];
  for (int i = 0; i < d; i++) {
    meany[i] = thrust::reduce(yVec_ptr + (i)*n, yVec_ptr + (i + 1) * n) / n;
hipDeviceSynchronize();
  addScalar<<<32, 256>>>(y+i*n, -meany[i], n);
  }
hipDeviceSynchronize();
  gettimeofday(&t2, NULL);
  double elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;    // sec to ms
  elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
  printf("time=%lf\n",elapsedTime );
  timeInfo[7*1000]=elapsedTime;
/*
  cout << "Time for computing Attractive Estimation " << timeFattr << "ms\n";
  cout << "Time for computing Repulsive Estimation " << timeFrep << " ms\n";
  cout << "Time for computing point update " << timeUpdate << " ms\n";
  cout << "Detailed " << timeInfo[1] << " ms in s2g, " << timeInfo[2]
       << " ms in g2g, " << timeInfo[3] << " ms in g2s\n";
  cout << timeInfo[4] << " ms in zetaAndForce, " << timeInfo[5]
       << " ms  in nuconv, " << timeInfo[6] << " ms in preprocessing\n";
  cout << "and " << timeInfo[0] << " in permutations\n";
*/
//  timeInfo[0]=timeFattr;
/*
  ofstream fout_cost;
  fout_cost.open ("sg_cost.txt");
  for(int i=0;i<100;i++){fout_cost<<KL[i]<<"\n"; }
  fout_cost.close();
*/
  if (params.ComputeError > 0) {
    ofstream errorf;
    ifstream errorfin;
    errorfin.open("errorInfo.txt");
    errorf.open("errorInfo.txt",std::ios::app);
     if(errorfin.is_open())
   {
    for (int i = 0; i < errorCalcs; i++) {
      errorf << errorRep[i] << " ";
    }
	}
    errorf<<"\n";

    errorf.close();
    errorfin.close();
    free(errorRep);
  }
  CUDA_CALL(hipFree(PhiGrid));
  CUDA_CALL(hipFree(VGrid));
  CUDA_CALL(hipFree(yt));
  CUDA_CALL(hipFree(VScat));
  CUDA_CALL(hipFree(PhiScat));
  CUDA_CALL(hipFree(Kc));
  CUDA_CALL(hipFree(Xc));
  CUDA_CALL(hipFree(dy));
  CUDA_CALL(hipFree(uy));
  CUDA_CALL(hipFree(gains));
  CUDA_CALL(hipFree(Fattr));
  CUDA_CALL(hipFree(Frep));

}

void kl_minimization(double *y, tsneparams params, sparse_matrix<double> P) {
  // ----- t-SNE hard coded parameters - Same as in vdM's code
  int stop_lying_iter = params.earlyIter, mom_switch_iter = 250;
  double momentum = .5, final_momentum = .8;
  double eta = 200.0;
  // int iterPrint = 50;
  hipStreamCreate(&streamAttr);
  hipStreamCreate(&streamRep);
  double timeFattr = 0.0;
  double timeFrep = 0.0;
  double timeUpdate=0.0;

  int n = params.n;
  int d = params.d;
  int max_iter = params.maxIter;

  double zeta = 0;
  double *errorRep = nullptr;
  int errorCalcs = (int)max_iter / 10;
  if (params.ComputeError > 0) {
    errorRep = (double *)malloc(sizeof(double) * errorCalcs);
  }
  // ----- Allocate memory
  double *dy;
  double *uy;
  double *gains;
  CUDA_CALL(hipMallocManaged(&dy, d * n * sizeof(double)));
  CUDA_CALL(hipMallocManaged(&uy, d * n * sizeof(double)));
  CUDA_CALL(hipMallocManaged(&gains, d * n * sizeof(double)));

  /*-------Initialize-----*/
  initKernel<<<64, 1024>>>(uy, (double)0.0, n * d);
  initKernel<<<64, 1024>>>(gains, (double)1.0, n * d);
  // double timeInfo[7] = {0};
  double timeInfo[1000 * 7] = {0};

  hipfftHandle plan, plan_rhs;

  int n1 = getBestGridSize(params.ng);
  int n2 = n1 + 2;
  switch (d) {
  case 1: {
    int ng[1] = {(int)n2};
    hipfftPlan1d(&plan, n2, HIPFFT_Z2Z, 1);
    hipfftPlanMany(&plan_rhs, 1, ng, NULL, 1, n2, NULL, 1, n2, HIPFFT_Z2Z, d + 1);
    break;
  }
  case 2: {
    int ng[2] = {(int)n2, (int)n2};
    hipfftPlanMany(&plan, 2, ng, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2Z, 1);
    hipfftPlanMany(&plan_rhs, 2, ng, NULL, 1, n2 * n2, NULL, 1, n2 * n2,
                  HIPFFT_Z2Z, d + 1);
    break;
  }
  case 3: {
    int ng[3] = {(int)n2, (int)n2, (int)n2};
    hipfftPlanMany(&plan, 3, ng, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2Z, 1);
    hipfftPlanMany(&plan_rhs, 3, ng, NULL, 1, n2 * n2 * n2, NULL, 1,
                  n2 * n2 * n2, HIPFFT_Z2Z, d + 1);
    break;
  }
  }
  hipfftSetStream(plan, streamRep);
  hipfftSetStream(plan_rhs, streamRep);
  int m = d + 1;
  int nVec = d + 1;
  /*Allocate memory*/
  double *yt;
  CUDA_CALL(hipMallocManaged(&yt, (d)*n * sizeof(double)));
  double *VScat;
  CUDA_CALL(hipMallocManaged(&VScat, (d + 1) * n * sizeof(double)));
  double *PhiScat;
  CUDA_CALL(hipMallocManaged(&PhiScat, (d + 1) * n * sizeof(double)));
  int szV = pow(n1 + 2, d) * m;
  double *VGrid;
  CUDA_CALL(hipMallocManaged(&VGrid, szV * sizeof(double)));
  double *PhiGrid;
  CUDA_CALL(hipMallocManaged(&PhiGrid, szV * sizeof(double)));
  ComplexD *Kc, *Xc;
  CUDA_CALL(hipMallocManaged(&Kc, szV * sizeof(ComplexD)));
  CUDA_CALL(hipMallocManaged(&Xc, nVec * szV * sizeof(ComplexD)));
  thrust::device_vector<double> zetaVec(n);
  double *Fattr;
  double *Frep;
  CUDA_CALL(hipMallocManaged(&Fattr, n * d * sizeof(double)));
  CUDA_CALL(hipMallocManaged(&Frep, n * d * sizeof(double)));
  struct GpuTimer timer;

  for (int iter = 0; iter < max_iter; iter++) {
    initKernel<<<64, 1024>>>(Fattr, (double)0.0, n * d);
    initKernel<<<64, 1024, 0, streamRep>>>(VGrid, (double)0, szV);
    initKernel<<<64, 1024, 0, streamRep>>>(PhiGrid, (double)0, szV);
    if (iter % 100 == 0) {
      printf("---------------------------%d---------------------------\n",
             iter);
    }

    if (params.ComputeError > 0 && iter % 10 == 0) {
      zeta = compute_gradient(Fattr,Frep,dy, &timeFrep, &timeFattr, params, y, P,
                              &timeInfo[iter * 7], plan, plan_rhs, n1, yt,
                              VScat, PhiScat, VGrid, PhiGrid, Kc, Xc, zetaVec,
                              &errorRep[(int)iter / 10]);
    } else {
      zeta = compute_gradient(Fattr,Frep,dy, &timeFrep, &timeFattr, params, y, P,
                              &timeInfo[iter * 7], plan, plan_rhs, n1, yt,
                              VScat, PhiScat, VGrid, PhiGrid, Kc, Xc, zetaVec);
    }

    timer.Start();
    update_positions(dy, uy, n, d, y, gains, momentum, eta);
    timer.Stop();
    timeUpdate+= timer.Elapsed();
    // Stop lying about the P-values after a while, and switch momentum
    if (iter == stop_lying_iter) {
      params.alpha = 1;
    }

    // Change momentum after a while
    if (iter == mom_switch_iter) {
      momentum = final_momentum;
    }
  }
  cout << "Time for computing Attractive Estimation " << timeFattr << "ms\n";
  cout << "Time for computing Repulsive Estimation " << timeFrep << " ms\n";
  cout << "Time for computing point update " << timeUpdate << " ms\n";
  ofstream fout;  // Create Object of Ofstream
  ifstream fin;
  fin.open("Attractive.txt");
  fout.open ("Attractive.txt",ios::app); // Append mode
  if(fin.is_open())
    {fout<<P.n<<" "<<timeFattr<<" "<<P.nnz <<"\n";}
    fin.close();
    fout.close(); // Closing the file
  cout << "Detailed " << timeInfo[1] << " ms in s2g, " << timeInfo[2]
       << " ms in g2g, " << timeInfo[3] << " ms in g2s\n";
  cout << timeInfo[4] << " ms in zetaAndForce, " << timeInfo[5]
       << " ms  in nuconv, " << timeInfo[6] << " ms in preprocessing\n";
  cout << "and " << timeInfo[0] << " in permutations\n";

  ofstream myfile;
  myfile.open("timeInfo.txt");
  for (int i = 0; i < max_iter; i++) {
    for (int j = 0; j < 7; j++) {
      myfile << timeInfo[7 * i + j] << " ";
    }
    myfile << "\n";
  }
  myfile.close();

  if (params.ComputeError > 0) {
    ofstream errorf;
    errorf.open("errorInfo.txt");
    for (int i = 0; i < errorCalcs; i++) {
      errorf << errorRep[i] << "\n";
    }
    errorf.close();
    free(errorRep);
  }
  CUDA_CALL(hipFree(PhiGrid));
  CUDA_CALL(hipFree(VGrid));
  CUDA_CALL(hipFree(yt));
  CUDA_CALL(hipFree(VScat));
  CUDA_CALL(hipFree(PhiScat));
  CUDA_CALL(hipFree(Kc));
  CUDA_CALL(hipFree(Xc));
  CUDA_CALL(hipFree(dy));
  CUDA_CALL(hipFree(uy));
  CUDA_CALL(hipFree(gains));
  CUDA_CALL(hipFree(Fattr));
  CUDA_CALL(hipFree(Frep));
}
