#include <stdint.h>
#include "utils_cuda.cuh"
#include "timer.h"
#include <fstream>
#include <iostream>
#include "hipsolver.h"
#include "sparsematrix.hpp"
#include "types.hpp"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "pq.cuh"
#include "helper/hip/hip_runtime_api.h"
#include "sparse_reorder.cuh"
using namespace std;
#define CUDA_CALL(x)                                                           \
  {                                                                            \
    if ((x) != hipSuccess) {                                                  \
      printf("CUDA error at %s:%d\n", __FILE__, __LINE__);                     \
      printf("  %s\n", hipGetErrorString(hipGetLastError()));                \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

void csr2bsr(int blockDim, int n, int m, int nnz, int *csrRowptr,
             int *csrColInd, coord *csrVal, int **bsrRowPtr, int **bsrColInd,
             coord **bsrVal, int *nnzblocks, int *n_block_rows,
             hipsparseHandle_t handle) {

  int *csrRowPtrA, *csrColIndA;
  coord *csrValA;
  hipMalloc((void **)&csrRowPtrA, sizeof(int) * (m + 1));
  hipMalloc((void **)&csrColIndA, sizeof(int) * nnz);
  hipMalloc((void **)&csrValA, sizeof(coord) * nnz);
  hipMemcpy(csrValA, csrVal, nnz * sizeof(coord), hipMemcpyHostToDevice);
  hipMemcpy(csrColIndA, csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(csrRowPtrA, csrRowptr, (m + 1) * sizeof(int),
             hipMemcpyHostToDevice);

  hipsparseMatDescr_t descr = 0;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseDirection_t dir = HIPSPARSE_DIRECTION_ROW;
  int base, nnzb;
  int mb = (m + blockDim - 1) / blockDim;
  hipMalloc(bsrRowPtr, sizeof(int) * (mb + 1));
  // nnzTotalDevHostPtr points to host memory
  int *nnzTotalDevHostPtr = &nnzb;
  hipsparseXcsr2bsrNnz(handle, dir, m, n, descr, csrRowPtrA, csrColIndA,
                      blockDim, descr, *bsrRowPtr, nnzTotalDevHostPtr);
  if (NULL != nnzTotalDevHostPtr) {
    nnzb = *nnzTotalDevHostPtr;
  } else {
    hipMemcpy(&nnzb, *bsrRowPtr + mb, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&base, *bsrRowPtr, sizeof(int), hipMemcpyDeviceToHost);
    nnzb -= base;
  }
  hipMalloc(bsrColInd, sizeof(int) * nnzb);
  hipMalloc(bsrVal, sizeof(coord) * (blockDim * blockDim) * nnzb);
  hipsparseDcsr2bsr(handle, dir, m, n, descr, csrValA, csrRowPtrA, csrColIndA,
                   blockDim, descr, *bsrVal, *bsrRowPtr, *bsrColInd);
  *nnzblocks = nnzb;
  *n_block_rows = mb;
  hipFree(csrRowPtrA);
  hipFree(csrColIndA);
  hipFree(csrValA);
}

template <class dataPoint>
dataPoint maxerror(dataPoint * dw, dataPoint *dv, int n, int d) {
  dataPoint *w=(dataPoint* )malloc(n*d*sizeof(dataPoint));
  dataPoint *v=(dataPoint* )malloc(n*d*sizeof(dataPoint));
  CUDA_CALL(hipMemcpy(v, dv, n * d * sizeof(coord),hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(w, dw, n * d * sizeof(coord),hipMemcpyDeviceToHost));

  dataPoint maxError = 0;
  dataPoint avgError = 0;

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < d; j++) {
      if ((v[i + j * n] - w[i + j * n]) * (v[i + j * n] - w[i + j * n]) >
          maxError) {
        maxError =
            (v[i + j * n] - w[i + j * n]) * (v[i + j * n] - w[i + j * n]);
          }
      avgError += (v[i + j * n] - w[i + j * n]) * (v[i + j * n] - w[i + j * n]);
    }
  }
  free(w);
  free(v);
  return maxError;
}

#define FLAG_BSDB_PERM
int main(int argc, char *argv[]) {
  hipsolverSpHandle_t handle = NULL;
  hipsparseHandle_t cusparseHandle = NULL; /* used in residual evaluation */
  hipStream_t stream = NULL;
  hipsparseMatDescr_t descrA = NULL;
  checkCudaErrors(hipsolverSpCreate(&handle));
  checkCudaErrors(hipsparseCreate(&cusparseHandle));

  checkCudaErrors(hipStreamCreate(&stream));
  /* bind stream to cusparse and cusolver*/
  checkCudaErrors(hipsolverSpSetStream(handle, stream));
  checkCudaErrors(hipsparseSetStream(cusparseHandle, stream));

  /* configure matrix descriptor*/
  checkCudaErrors(hipsparseCreateMatDescr(&descrA));
  checkCudaErrors(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
  int baseA = 0; /* base index in CSR format */
  if (baseA) {
    checkCudaErrors(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));
  } else {
    checkCudaErrors(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));
  }
  struct GpuTimer timer;

  int M, N, nz;
  int *I, *J;
  double *val;
  N = atoi(argv[1]);
  M = atoi(argv[2]);
  nz = atoi(argv[3]);
  int bs = atoi(argv[4]);
  int d=atoi(argv[6]);
  I = (int *)malloc(sizeof(int) * nz);
  J = (int *)malloc(sizeof(int) * nz);
  val = (coord *)malloc(sizeof(coord) * nz);

  for (int i = 0; i < nz; i++) {
    scanf("%d %d %lf\n", &J[i], &I[i], &val[i]);
    I[i]--;
    J[i]--;
  }

  double *csr_val = (double *)calloc(nz, sizeof(double));
  int *csr_col = (int *)calloc(nz, sizeof(int));
  int *csr_row = (int *)calloc(M + 1, sizeof(int));

  for (int i = 0; i < nz; i++) {
    csr_val[i] = val[i];
    csr_col[i] = J[i];
    csr_row[I[i] + 1]++;
  }
  for (int i = 0; i < M; i++) {
    csr_row[i + 1] += csr_row[i];
  }
  coord *bsrValC;
  int *bsrRowPtrC, *bsrColIndC;
  int mb, nnzb;
  sparse_matrix P;
  P.n = N;
  P.m = M;
  P.nnz = nz;
  P.val = csr_val;
  P.col = csr_row;
  P.row = csr_col;

  symmetrizeMatrix(&P);
  N = P.n;
  M = P.m;
  nz = P.nnz;
  printf("nnz= %d\n", P.nnz);

  int *perm = static_cast<int *>(malloc(N * sizeof(int)));
  double *csr_val_permuted = (double *)calloc(nz, sizeof(double));
  int *csr_col_permuted = (int *)calloc(nz, sizeof(int));
  int *csr_row_permuted = (int *)calloc(M + 1, sizeof(int));

  SparseReorder(argv[5], handle, descrA, M, N, nz, P.col, P.row, P.val,
               csr_row_permuted, csr_col_permuted, csr_val_permuted, perm);
  csr2bsr(bs, N, M, nz, csr_row_permuted, csr_col_permuted, csr_val_permuted, &bsrRowPtrC, &bsrColIndC, &bsrValC,&nnzb, &mb, cusparseHandle);
  int n=N;

  printf("nnzb=%d mb=%d\n", nnzb, mb);
  coord  *x = (coord   *)malloc(n * d * sizeof(coord ));
  coord  *Fattr;
  coord  *y;
  for(int i=0;i<M*d;i++){
    x[i]=100*(double)rand()/RAND_MAX;
  }
  CUDA_CALL(hipMallocManaged(&Fattr, n * d * sizeof(coord)));
  CUDA_CALL(hipMallocManaged(&y, n * d * sizeof(coord)));
  initKernel<<<64, 1024>>>(Fattr, 0.0, n * d);
  CUDA_CALL(hipMemcpy(y, x, n * d * sizeof(coord),hipMemcpyHostToDevice));

  for(int i=0; i<10;i++){
  timer.Start();
  AttractiveEstimation(bsrRowPtrC, bsrColIndC, bsrValC,Fattr, y, n, d, bs,mb,nnzb,nz,1);
  timer.Stop();
  printf("time bsr %f \n",timer.Elapsed() );
  }

  coord  *Fattr2;
  CUDA_CALL(hipMallocManaged(&Fattr2, n * d * sizeof(coord)));
  initKernel<<<64, 1024>>>(Fattr2, 0.0, n * d);
  int*csrRow,*csrCol;
  coord *csrVal;
  CUDA_CALL(hipMallocManaged(&csrVal, nz * sizeof(coord)));
  CUDA_CALL(hipMallocManaged(&csrCol,nz * sizeof(int)));
  CUDA_CALL(hipMallocManaged(&csrRow,(M + 1)* sizeof(int)));
  CUDA_CALL(hipMemcpy(csrVal, csr_val_permuted, nz * sizeof(coord),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(csrRow, csr_row_permuted, (M+1) * sizeof(int),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(csrCol, csr_col_permuted, nz * sizeof(int),hipMemcpyHostToDevice));

  for(int i=0; i<10;i++){
    timer.Start();
  AttractiveEstimation(csrRow, csrCol, csrVal,Fattr2, y, n, d, bs,mb,nnzb,nz,0);
  timer.Stop();
  printf("time csr %f \n",timer.Elapsed() );
}

  printf("maxError %lf\n",maxerror(Fattr,Fattr2,n,d) );
  free(csr_val_permuted);
  free(csr_row_permuted);
  free(csr_col_permuted);

  /*
  int *bsrRow=(int *)malloc(sizeof(int)*(mb+1));
  int *bsrCol=(int *)malloc(sizeof(int)*(nnzb));
  coord *bsrVal=(coord *)malloc(sizeof(coord)*(nnzb*bs*bs));
  CUDA_CALL(hipMemcpy(bsrVal, bsrValC, nnzb*bs*bs * sizeof(coord),hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(bsrRow, bsrRowPtrC, (mb+1) * sizeof(int),hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(bsrCol, bsrColIndC, (nnzb)* sizeof(int),hipMemcpyDeviceToHost));
  ofstream myfile;

  myfile.open("bsr.txt");
  for(int i=0;i<mb+1;i++){
    myfile <<bsrRow[i] <<"\n";
  }
  for(int i=0;i<nnzb*bs*bs;i++){
    myfile <<bsrVal[i] <<"\n";
  }
  for(int i=0;i<nnzb;i++){
    myfile <<bsrCol[i] <<"\n";
  }
  myfile.close();
*/
  if (handle) {
    checkCudaErrors(hipsolverSpDestroy(handle));
  }
  if (cusparseHandle) {
    checkCudaErrors(hipsparseDestroy(cusparseHandle));
  }
  if (stream) {
    checkCudaErrors(hipStreamDestroy(stream));
  }
  if (descrA) {
    checkCudaErrors(hipsparseDestroyMatDescr(descrA));
  }
  return 0;
}
