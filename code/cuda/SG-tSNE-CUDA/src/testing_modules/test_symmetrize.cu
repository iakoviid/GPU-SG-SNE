
#include <stdio.h>
#include <hipsparse.h>
#include "../types.hpp"
#include "../sparsematrix.hpp"
#include <random>

sparse_matrix *generateRandomCSC(int n){

  sparse_matrix *P = (sparse_matrix *) malloc(sizeof(sparse_matrix));

  P->n = n; P->m = n;

  P->col = (matidx *) malloc( (n+1)*sizeof(matidx) );

  for (int j=0 ; j<n ; j++)
    P->col[j] = rand() % 10 + 2;

  int cumsum = 0;
  for(int i = 0; i < P->n; i++){
    int temp = P->col[i];
    P->col[i] = cumsum;
    cumsum += temp;
  }
  P->col[P->n] = cumsum;
  P->nnz = cumsum;

  P->row = (matidx *) malloc( (P->nnz)*sizeof(matidx) );
  P->val = (matval *) malloc( (P->nnz)*sizeof(matval) );

  std::uniform_real_distribution<double> unif(0,1);
  std::default_random_engine re;

  for (int l = 0; l < P->nnz; l++){
    P->row[l] = rand() % n;
    P->val[l] = unif(re);
  }

  return P;

}
/*
void symmetrizeMatrixGPU(sparse_matrix *A, sparse_matrix *C,hipsparseHandle_t &handle) {
  int M = A->m;
  int N = A->n;
  int nnz = A->nnz;
  int *csc_row_ptr_at;
  hipMalloc(reinterpret_cast<void **>(&csc_row_ptr_at),
             (A->nnz) * sizeof(int));
  int *csc_column_ptr_at;
  hipMalloc(reinterpret_cast<void **>(&csc_column_ptr_at),
             (A->n + 1) * sizeof(int));
  matval *csc_values_at;
  hipMalloc(reinterpret_cast<void **>(&csc_values_at),
             (A->nnz) * sizeof(matval));

  // Do the transpose operation
  hipsparseDcsr2csc(handle, A->m, A->n, A->nnz, A->val, A->row, A->col,
                   csc_values_at, csc_row_ptr_at, csc_column_ptr_at,
                   HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO);

  hipDeviceSynchronize();

  hipsparseMatDescr_t descrA;
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  // --- Summing the two matrices
  int baseC, nnz3;
  // --- nnzTotalDevHostPtr points to host memory
  int32_t symmetrized_num_nonzeros = -1;
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
  hipMalloc(&C->row, (M + 1) * sizeof(int));

  hipsparseXcsrgeamNnz(handle, M, N, descrA, A->nnz, A->row, A->col, descrA, nnz,
                      csc_column_ptr_at, csc_row_ptr_at, descrA, C->row,
                      &symmetrized_num_nonzeros);
  if (-1 != symmetrized_num_nonzeros) {
    nnz3 = symmetrized_num_nonzeros;
  } else {
    hipMemcpy(&nnz3, C->row + M, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&baseC, C->row, sizeof(int), hipMemcpyDeviceToHost);
    nnz3 -= baseC;
  }
  hipMalloc(&(C->col), nnz3 * sizeof(int));
  hipMalloc(&(C->val), nnz3 * sizeof(matval));
  matval alpha = (matval)1 / 2, beta = (matval)1 / 2;
  hipsparseDcsrgeam(handle, M, N, &alpha, descrA, nnz, A->val, A->row, A->col,
                   &beta, descrA, nnz, csc_values_at, csc_column_ptr_at,
                   csc_row_ptr_at, descrA, C->val, C->row, C->col);

  C->nnz = nnz3;
  C->m = M;
  C->n = N;
  hipDeviceSynchronize();

  hipFree(csc_column_ptr_at);
  hipFree(csc_row_ptr_at);
  hipFree(csc_values_at);
}*/
void symmetrizeMatrixGPU(sparse_matrix *A, sparse_matrix *C,hipsparseHandle_t handle) {

  double *d_csrValB;       hipMalloc(&d_csrValB, A->nnz * sizeof(double));
  int *d_csrRowPtrB;      hipMalloc(&d_csrRowPtrB, (A->m + 1) * sizeof(int));
  int *d_csrColIndB;      hipMalloc(&d_csrColIndB, A->nnz * sizeof(int));

  hipsparseDcsr2csc(handle, A->m, A->n, A->nnz, A->val, A->row,A->col,
                   d_csrValB, d_csrColIndB, d_csrRowPtrB,
                   HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO);
  hipDeviceSynchronize();
  // --- Summing the two matrices
  int baseC, nnz3;
  hipsparseMatDescr_t descrA, descrB, descrC;
  hipsparseCreateMatDescr(&descrA);
  hipsparseCreateMatDescr(&descrB);
  hipsparseCreateMatDescr(&descrC);
  // --- nnzTotalDevHostPtr points to host memory
  int *nnzTotalDevHostPtr = &nnz3;
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
  hipMalloc(&C->row, (A->m + 1) * sizeof(int));
  hipsparseXcsrgeamNnz(handle, A->m, A->n, descrA, A->nnz, A->row, A->col, descrB, A->nnz, d_csrRowPtrB, d_csrColIndB, descrC, C->row, nnzTotalDevHostPtr);
  if (NULL != nnzTotalDevHostPtr) {
      nnz3 = *nnzTotalDevHostPtr;
  }
  else{
      hipMemcpy(&nnz3, C->row + A->m, sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(&baseC, C->row, sizeof(int), hipMemcpyDeviceToHost);
      nnz3 -= baseC;
  }

  hipMalloc(&C->col, nnz3 * sizeof(int));
  hipMalloc(&C->val, nnz3 * sizeof(double));
  double alpha = 0.5, beta = 0.5;
  hipsparseDcsrgeam(handle, A->m, A->n, &alpha, descrA, A->nnz, A->val, A->row, A->col, &beta, descrB, A->nnz, d_csrValB, d_csrRowPtrB, d_csrColIndB, descrC, C->val, C->row, C->col);
  hipDeviceSynchronize();
  C->nnz=nnz3;
  C->m=A->m;
  C->n=A->n;
  hipFree(d_csrValB);
  hipFree(d_csrRowPtrB);
  hipFree(d_csrColIndB);

}
int main(int argc, char **argv)
 {
   int n=atoi(argv[1]);
   hipsparseHandle_t handle;    hipsparseCreate(&handle);
   sparse_matrix *Ah = generateRandomCSC(n);
   int nnz=Ah->nnz;

   printf("Ah->nnz=%d  \n",Ah->nnz);

   const int M = n;                                    // --- Number of rows
   const int N = n;                                    // --- Number of columns
   sparse_matrix A;
   A.m=M;
   A.n=N;
   A.nnz=nnz;
   hipMalloc(&A.val, nnz * sizeof(matval));
   hipMalloc(&A.row, (M + 1) * sizeof(matidx));
   hipMalloc(&A.col, nnz * sizeof(matidx));
   hipMemcpy(A.val, Ah->val, nnz * sizeof(matval), hipMemcpyHostToDevice);
   hipMemcpy(A.row, Ah->col, (M + 1) * sizeof(matidx), hipMemcpyHostToDevice);
   hipMemcpy(A.col, Ah->row, nnz * sizeof(matidx), hipMemcpyHostToDevice);
   sparse_matrix C;

   symmetrizeMatrixGPU(&A,&C,handle);
   double* val=(double*)malloc(C.nnz*sizeof(double));
   int* row=(int*)malloc(C.nnz*sizeof(int));
   int* col=(int*)malloc(C.nnz*sizeof(int));
   hipMemcpy(val, C.val, C.nnz * sizeof(matval), hipMemcpyDeviceToHost);
   hipMemcpy(row, C.row, (C.n + 1) * sizeof(matidx), hipMemcpyDeviceToHost);
   hipMemcpy(col, C.col, C.nnz * sizeof(matidx), hipMemcpyDeviceToHost);
   symmetrizeMatrix(Ah);
   printf("Ah->nnz=%d vs C.nnz=%d \n",Ah->nnz,C.nnz );
   for(int i=0;i<C.nnz;i++){
     printf("%lf vs %lf\n",Ah->val[i],val[i] );
   }

       return 0;
   }
