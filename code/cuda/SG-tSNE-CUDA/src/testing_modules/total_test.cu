#include "../gridding.cuh"
#include "../gridding.hpp"
#include "../relocateData.cuh"
#include "../relocateData.hpp"
#include "../utils_cuda.cuh"
#include "../Frep.cuh"
#include "../Frep.hpp"

#include <iostream>
#include <stdio.h>
#include <sys/time.h>
using namespace std;
#include "../types.hpp"

template <class dataPoint>
dataPoint maxerror(dataPoint *const w, dataPoint *dv, int n, int d) {
  printf("Enter Comparison_-----------------------------\n" );

  dataPoint *v = (dataPoint *)malloc(n * d * sizeof(dataPoint));
  hipMemcpy(v, dv, d * n * sizeof(dataPoint), hipMemcpyDeviceToHost);
  dataPoint maxError = 0;
  dataPoint avgError = 0;
  int pos = 0;

  for (int i = 0; i < n ; i++) {
    for(int j=0;j<d;j++){
      if((v[i+j*n] - w[i*d+j]) * (v[i+j*n] - w[i*d+j]) >
          0.001 || i<2){
      cout<<"v= "<<v[i+j*n]<<" "<< "w= "<<w[i*d+j]<<"\n";
      }
    if ((v[i+j*n] - w[i*d+j]) * (v[i+j*n] - w[i*d+j]) > maxError) {
      maxError = (v[i+j*n] - w[i*d+j]) * (v[i+j*n] - w[i*d+j]);
      pos = i;
    }
    avgError += (v[i+j*n] - w[i*d+j]) * (v[i+j*n] - w[i*d+j]);
  }}

  printf("maxError=%lf pos=%d v[i]=%lf vs w[i]=%lf avgError=%lf n=%d size=%d\n",
         maxError, pos, v[pos], w[pos*d], avgError / (n * d), n, n * d);
  free(v);
  return maxError;
}

coord *generateRandomCoord(int n, int d,coord scale) {

  coord *y = (coord *)malloc(n * d * sizeof(coord));
  for (int i = 0; i < n * d; i++)
    y[i] = ((coord)rand() / (RAND_MAX)) * scale;

  return y;
}

template <class dataPoint>
void copydata(dataPoint *const w, dataPoint *dw, int n, int d) {
  dataPoint *v = (dataPoint *)malloc(sizeof(dataPoint) * n * d);
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < d; j++) {

      v[i + n * j] = w[i * d + j];
    }
  }
  hipMemcpy(dw, v, d * n * sizeof(dataPoint), hipMemcpyHostToDevice);
  free(v);
  return;
}

int main(int argc, char **argv) {
  srand(time(NULL));

  int n = atoi(argv[1]);
  int d = atoi(argv[2]);
  //int ng = atoi(argv[3]);
  int iterations=atoi(argv[4]);
  coord scale=(coord)atoi(argv[5]);
  coord *y, *y_d;
  struct timeval t1, t2;
  double elapsedTime;
  CUDA_CALL(hipMallocManaged(&y_d, (d)*n * sizeof(coord)));
  coord *Frep, *Frep_d;
  CUDA_CALL(hipMallocManaged(&Frep_d, (d)*n * sizeof(coord)));
  Frep=(coord *)malloc(n*d*sizeof(coord));
  for(int i=0;i<iterations;i++){
  y = generateRandomCoord(n, d,scale);
  copydata(y, y_d, n, d);
/*
  coord maxy = 0;
  for (int i = 0; i < n * d; i++)
    maxy = maxy < y[i] ? y[i] : maxy;

  coord h = maxy / (ng - 1 - std::numeric_limits<double>::epsilon());
*/
  coord h=0.7;
  coord zeta1;
  coord zeta2;
  double timeInfo[7];
  double timeInfo1[6];
  //printf("times Frep Reloc S2G G2G G2S\n" );
  gettimeofday(&t1, NULL);

  zeta1=computeFrepulsive_interpCPU(Frep, y,  n,  d, h,1,timeInfo1);
  gettimeofday(&t2, NULL);

  elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;      // sec to ms
  elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0;   // us to ms
  printf("Host time milliseconds %lf %lf %lf %lf %lf %lf %lf\n", elapsedTime,timeInfo1[0],timeInfo1[1],timeInfo1[2],timeInfo1[3],timeInfo1[4],timeInfo1[5]);
  gettimeofday(&t1, NULL);

  zeta2= computeFrepulsive_interp(Frep_d, y_d, n, d, h,timeInfo);
  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);
  double time1=elapsedTime;
  elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;      // sec to ms
  elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0;   // us to ms
  printf("CUDA elapsedTime %lf\n", elapsedTime);

  printf("Detailed %lf ms in s2g, %lf ms in g2g, %lf ms in g2s\n",timeInfo[1],timeInfo[2],timeInfo[3]);
  printf("%lf ms in zetaAndForce, %lfms  in nuconv, %lf ms Computing Charges\n",timeInfo[4],timeInfo[5],timeInfo[6] );

  //printf("CUDA time milliseconds %lf %lf %lf %lf %lf %lf\n", elapsedTime,timeInfo[0],timeInfo[1],timeInfo[2],timeInfo[3],timeInfo[4]);
  printf("speedup  %lf %lf %lf %lf %lf %lf %lf\n",time1/elapsedTime,timeInfo1[0]/timeInfo[0],timeInfo1[1]/timeInfo[1],timeInfo1[2]/timeInfo[2],timeInfo1[3]/timeInfo[3],timeInfo1[4]/timeInfo[4],timeInfo1[5]/timeInfo[5] );
  printf("zeta1=%lf vs zeta2=%lf\n",zeta1,zeta2 );
  printf("dzeta=%lf\n",(zeta1-zeta2)*(zeta1-zeta2) );
  maxerror(Frep, Frep_d, n, d);
  maxerror(y, y_d, n, d);

  free(y);
  }
  hipFree(y_d);
  free(Frep);
  hipFree(Frep_d);
return 0;
}
