#include "hip/hip_runtime.h"
#include "../gradient_descend.cuh"
#include "../gradient_descend.hpp"
#include "../graph_rescaling.hpp"
#include "../sparsematrix.hpp"
#include "../utils_cuda.cuh"
#include <random>
#include "../prepareMatrix.cuh"
#include "../sparse_reorder.cuh"
#include <iostream>
#include <stdio.h>
#include <sys/time.h>
using namespace std;
#include "../types.hpp"

template <class dataPoint>
dataPoint maxerror(dataPoint *const w, dataPoint *dv,int* perm, int n, int d) {

  dataPoint *v = (dataPoint *)malloc(n * d * sizeof(dataPoint));
  hipMemcpy(v, dv, d * n * sizeof(dataPoint), hipMemcpyDeviceToHost);
  dataPoint maxError = 0;
  dataPoint avgError = 0;

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < d; j++) {
      dataPoint error=fabs(v[i+j*n] - w[perm[i]*d+j]);
         if (error > maxError) {
           maxError =error;

      }
      avgError += error;
    }
  }

  printf("maxError=%lf pos=%d v[i]=%lf vs w[i]=%lf avgError=%lf n=%d size=%d\n",
         maxError, 1, v[1], w[1 * d], avgError / (n * d), n, n * d);
  free(v);
  return maxError;
}

coord *generateRandomCoord(int n, int d) {

  coord *y = (coord *)malloc(n * d * sizeof(coord));

  for (int i = 0; i < n * d; i++)
    y[i] = ((coord)rand() / (RAND_MAX)) * .0001;

  return y;
}

template <class dataPoint>
void copydata(dataPoint *const w, dataPoint *dw, int n, int d) {
  dataPoint *v = (dataPoint *)malloc(sizeof(dataPoint) * n * d);
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < d; j++) {

      v[i + n * j] = w[i * d + j];
    }
  }
  hipMemcpy(dw, v, d * n * sizeof(dataPoint), hipMemcpyHostToDevice);
  free(v);
  return;
}
sparse_matrix *generateRandomCSC(int n) {

  sparse_matrix *P = (sparse_matrix *)malloc(sizeof(sparse_matrix));

  P->n = n;
  P->m = n;

  P->col = (matidx *)malloc((n + 1) * sizeof(matidx));

  for (int j = 0; j < n; j++)
    P->col[j] = rand() % 10 + 2;

  int cumsum = 0;
  for (int i = 0; i < P->n; i++) {
    int temp = P->col[i];
    P->col[i] = cumsum;
    cumsum += temp;
  }
  P->col[P->n] = cumsum;
  P->nnz = cumsum;

  P->row = (matidx *)malloc((P->nnz) * sizeof(matidx));
  P->val = (matval *)malloc((P->nnz) * sizeof(matval));

  std::uniform_real_distribution<double> unif(0, 1);
  std::default_random_engine re;

  for (int l = 0; l < P->nnz; l++) {
    P->row[l] = rand() % n;
    P->val[l] = unif(re);
  }

  return P;
}

//nvcc testing_modules/kl_test.cu gradient_descend.cu timers.cpp utils.cu sparse_reorder.cu pq.cu FrepNoReloc.cu Frep.cpp gradient_descendCPU.cpp graph_rescaling.cpp gridding.cpp gridding.cu non_periodic_conv.cpp non_periodic_convD.cu nuconv.cpp nuconv.cu pq.cpp  relocateData.cpp sparsematrix.cpp  -arch=sm_60 -lfftw3 -lcufft -lcusparse -lcusolver
//./sg_test 8381 8381 251430  <pbmc-graph.mtx
int main(int argc, char **argv) {
  srand(time(NULL));

  int n = atoi(argv[1]);
  int d = atoi(argv[2]);
  int iterations = atoi(argv[3]);
  int nz = atoi(argv[4]);
  int format = atoi(argv[5]);
  int bs= atoi(argv[6]);
  char* method= argv[7];

  int N = n;
  int M = n;
  struct timeval t1, t2;
  double elapsedTime;
  int *I, *J;
  double *val;
  I = (int *)malloc(sizeof(int) * nz);
  J = (int *)malloc(sizeof(int) * nz);
  val = (coord *)malloc(sizeof(coord) * nz);
  for (int i = 0; i < nz; i++) {
    scanf("%d %d %lf\n", &J[i], &I[i], &val[i]);
    I[i]--;
    J[i]--;
  }
  sparse_matrix *P = (sparse_matrix *)malloc(sizeof(sparse_matrix));
  P->val = (double *)calloc(nz, sizeof(double));
  P->row = (int *)calloc(nz, sizeof(int));
  P->col = (int *)calloc(M + 1, sizeof(int));

  for (int i = 0; i < nz; i++) {
    P->val[i] = val[i];
    P->row[i] = J[i];
    P->col[I[i] + 1]++;
  }
  for (int i = 0; i < M; i++) {
    P->col[i + 1] += P->col[i];
  }
  P->n = N;
  P->m = M;
  P->nnz = nz;

  tsneparams params;
  params.d = d;
  params.n = n;
  params.alpha = 12;
  params.maxIter = iterations;
  params.earlyIter = iterations / 4;
  params.np = 1;
  gettimeofday(&t1, NULL);
  coord *y, *y_d;
  CUDA_CALL(hipMallocManaged(&y_d, (d)*n * sizeof(coord)));
  y = generateRandomCoord(n, d);
  copydata(y, y_d, n, d);

  uint32_t nStoch = makeStochastic(*P);
  std::cout << nStoch << " out of " << P->n << " nodes already stochastic"<< std::endl;
  // lambdaRescaling(*P, params.lambda, false, params.dropLeaf);

  symmetrizeMatrix(P);

  double sum_P = .0;
  for (int i = 0; i < P->nnz; i++) {

    sum_P += P->val[i];
  }
  for (int i = 0; i < P->nnz; i++) {
    P->val[i] /= sum_P;
  }

  switch (params.d) {
  case 1:
    params.h = 0.5;
    break;
  case 2:
    params.h = 0.7;
    break;
  case 3:
    params.h = 1.2;
    break;
  }

  int*   perm = static_cast<int *>(malloc(P->n * sizeof(int)));


  //kl_minimizationCPU(y, params, *P);
  gettimeofday(&t2, NULL);
  elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;    // sec to ms
  elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
  double timecpu = elapsedTime;
  printf("timecpu=%lf\n",timecpu );
  params.d = d;
  params.n = n;
  params.alpha = 12;
  params.maxIter = iterations;
  params.earlyIter = iterations / 4;
  params.np = 1;

  gettimeofday(&t1, NULL);
  sparse_matrix *Pd = PrepareSparseMatrix(P,perm,format, method, bs);
  kl_minimization(y_d, params, *Pd);
  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);
  elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;    // sec to ms
  elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
  double timegpu = elapsedTime;
  printf("Speedup=%lf\n", timecpu / timegpu);
  printf("timegpu=%lf\n",timegpu );
  //maxerror(y, y_d,perm, n, d);
  coord *y_copy =static_cast<coord *>(malloc(params.n * params.d * sizeof(coord)));

  CUDA_CALL(hipMemcpy(y_copy, y_d, params.n * params.d * sizeof(coord),hipMemcpyDeviceToHost));

  extractEmbeddingTextT(y_copy, params.n, params.d, "gpuEmbedding.txt");
  //extractEmbeddingText(y, params.n, params.d, "cpuEmbedding.txt");

  hipFree(y_d);
  free(y);
  free(I);
  free(J);
  free(val);
  return 0;
}
