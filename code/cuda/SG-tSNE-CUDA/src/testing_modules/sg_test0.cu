#include "hip/hip_runtime.h"
/*!
  \file   sgtsne.cpp
  \brief  Entry point to SG-t-SNE

  The main procedure definition, responsible for parsing the data
  and the parameters, preprocessing the input, running the
  gradient descent iterations and returning.


*/
#include "sgtsne.cuh"
#define HIP_INF_F __uint2double_rn(0x7ff0000000000000)
#define CUDART_NINF_F __uint2double_rn(0xfff0000000000000)
#include "graph_rescaling.hpp"
#include "prepareMatrix.cu"
template <class dataPoint>
dataPoint *generateRandomCoord(int n, int d) {

  dataPoint *y = (dataPoint *)malloc(n * d * sizeof(dataPoint));

  for (int i = 0; i < n * d; i++)
    y[i] = ((dataPoint)rand() / (RAND_MAX)) * .0001;

  return y;
}
template <class dataPoint>
dataPoint *sgtsneCUDA(sparse_matrix* P, tsneparams params, dataPoint *y_in,
                  double **timeInfo) {
  // ~~~~~~~~~~ unless h is specified, use default ones
  if (params.h <= 0)
    switch (params.d) {
    case 1:
      params.h = 0.5;
      break;
    case 2:
      params.h = 0.7;
      break;
    case 3:
      params.h = 1.2;
      break;
    }

  // ~~~~~~~~~~ print input parameters
  printParams(params);

  // ~~~~~~~~~~ make sure input matrix is column stochastic
  uint32_t nStoch = makeStochastic(*P);
  std::cout << nStoch << " out of " << P->n << " nodes already stochastic"
            << std::endl;

  // ~~~~~~~~~~ prepare graph for SG-t-SNE

  // ----- lambda rescaling
  if (params.lambda == 1)
    std::cout << "Skipping λ rescaling..." << std::endl;
  else
    lambdaRescaling(*P, params.lambda, false, params.dropLeaf);

  // ----- symmetrizing
  symmetrizeMatrix(P);

  // ----- normalize matrix (total sum is 1.0)
  matval sum_P = .0;
  for (int i = 0; i < P->nnz; i++) {

    sum_P += P->val[i];
  }
  for (int i = 0; i < P->nnz; i++) {
    P->val[i] /= sum_P;
  }

  // ~~~~~~~~~~ extracting BSDB permutation
  int*   perm = static_cast<int *>(malloc(P->n * sizeof(int)));
  sparse_matrix *Pd = PrepareSparseMatrix(P,perm,params.format, params.method,params.bs);


  // ~~~~~~~~~~ initial embedding coordinates

  dataPoint *y;
  CUDA_CALL(hipMallocManaged(&y, params.n * params.d * sizeof(dataPoint)));

  if (y_in == NULL) {

    // ----- Initialize Y
    dataPoint *y_rand =generateRandomCoord<dataPoint>(params.n, params.d);
    CUDA_CALL(hipMemcpy(y, y_rand, params.n * params.d * sizeof(dataPoint),
                         hipMemcpyHostToDevice));
    free(y_rand);

  } else {
    CUDA_CALL(hipMemcpy(y, y_in, params.n * params.d * sizeof(dataPoint),
                         hipMemcpyHostToDevice));
  }


  // ~~~~~~~~~~ gradient descent
  kl_minimization(y, params, *Pd);
  dataPoint *y_return =static_cast<dataPoint *>(malloc(params.n * params.d * sizeof(dataPoint)));

  CUDA_CALL(hipMemcpy(y_return, y, params.n * params.d * sizeof(dataPoint),
  hipMemcpyDeviceToHost));



  hipFree(y);
  free(perm);
  return y_return;
}
template
float *sgtsneCUDA(sparse_matrix* P, tsneparams params, float *y_in,
                  double **timeInfo);
