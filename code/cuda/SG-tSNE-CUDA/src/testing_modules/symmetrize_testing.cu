#include <stdio.h>
#include <assert.h>
#include "../types.hpp"
#include "../sparsematrix.hpp"
#include <hipsparse.h>

/*******************/
/* iDivUp FUNCTION */
/*******************/
int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
// --- Credit to http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) { exit(code); }
    }
}

void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }

/***************************/
/* CUSPARSE ERROR CHECKING */
/***************************/
static const char *_cusparseGetErrorEnum(hipsparseStatus_t error)
{
    switch (error)
    {

    case HIPSPARSE_STATUS_SUCCESS:
        return "HIPSPARSE_STATUS_SUCCESS";

    case HIPSPARSE_STATUS_NOT_INITIALIZED:
        return "HIPSPARSE_STATUS_NOT_INITIALIZED";

    case HIPSPARSE_STATUS_ALLOC_FAILED:
        return "HIPSPARSE_STATUS_ALLOC_FAILED";

    case HIPSPARSE_STATUS_INVALID_VALUE:
        return "HIPSPARSE_STATUS_INVALID_VALUE";

    case HIPSPARSE_STATUS_ARCH_MISMATCH:
        return "HIPSPARSE_STATUS_ARCH_MISMATCH";

    case HIPSPARSE_STATUS_MAPPING_ERROR:
        return "HIPSPARSE_STATUS_MAPPING_ERROR";

    case HIPSPARSE_STATUS_EXECUTION_FAILED:
        return "HIPSPARSE_STATUS_EXECUTION_FAILED";

    case HIPSPARSE_STATUS_INTERNAL_ERROR:
        return "HIPSPARSE_STATUS_INTERNAL_ERROR";

    case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
        return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

    case HIPSPARSE_STATUS_ZERO_PIVOT:
        return "HIPSPARSE_STATUS_ZERO_PIVOT";
    }

    return "<unknown>";
}

inline void __cusparseSafeCall(hipsparseStatus_t err, const char *file, const int line)
{
    if (HIPSPARSE_STATUS_SUCCESS != err) {
        fprintf(stderr, "CUSPARSE error in file '%s', line %d, error %s\nterminating!\n", __FILE__, __LINE__, \
            _cusparseGetErrorEnum(err)); \
            assert(0); \
    }
}

extern "C" void cusparseSafeCall(hipsparseStatus_t err) { __cusparseSafeCall(err, __FILE__, __LINE__); }


/* (P+P^T)/2*/
void symmetrizeMatrixGPU(sparse_matrix<matval> *A,hipsparseHandle_t &handle) {
  coord *d_csrValB;       gpuErrchk(hipMalloc(&d_csrValB, A->nnz * sizeof(coord)));
  int *d_csrRowPtrB;      gpuErrchk(hipMalloc(&d_csrRowPtrB, (A->m + 1) * sizeof(int)));
  int *d_csrColIndB;      gpuErrchk(hipMalloc(&d_csrColIndB, A->nnz * sizeof(int)));

  hipsparseScsr2csc(handle, A->m, A->n, A->nnz, A->val, A->row,A->col,
                   d_csrValB, d_csrColIndB, d_csrRowPtrB,
                   HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO);
  hipDeviceSynchronize();
  // --- Summing the two matrices
  int baseC, nnz3;
  hipsparseMatDescr_t descrA;
  hipsparseCreateMatDescr(&descrA);
  coord* sym_val;
  int * sym_col,* sym_row;
  // --- nnzTotalDevHostPtr points to host memory
  int *nnzTotalDevHostPtr = &nnz3;
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
  gpuErrchk(hipMalloc(&sym_row, (A->m + 1) * sizeof(int)));
  hipsparseXcsrgeamNnz(handle, A->m, A->n, descrA, A->nnz, A->row, A->col, descrA, A->nnz, d_csrRowPtrB, d_csrColIndB, descrA, sym_row, nnzTotalDevHostPtr);
  if (NULL != nnzTotalDevHostPtr) {
      nnz3 = *nnzTotalDevHostPtr;
  }
  else{
      gpuErrchk(hipMemcpy(&nnz3, sym_row + A->m, sizeof(int), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(&baseC, sym_row, sizeof(int), hipMemcpyDeviceToHost));
      nnz3 -= baseC;
  }

  gpuErrchk(hipMalloc(&sym_col, nnz3 * sizeof(int)));
  gpuErrchk(hipMalloc(&sym_val, nnz3 * sizeof(coord)));
  coord alpha = 0.5, beta = 0.5;
  hipsparseScsrgeam(handle, A->m, A->n, &alpha, descrA, A->nnz, A->val, A->row, A->col, &beta, descrA, A->nnz, d_csrValB, d_csrRowPtrB, d_csrColIndB, descrA, sym_val, sym_row, sym_col);
  hipDeviceSynchronize();
  A->nnz=nnz3;

  gpuErrchk(hipFree(d_csrValB));
  gpuErrchk(hipFree(d_csrRowPtrB));
  gpuErrchk(hipFree(d_csrColIndB));
  gpuErrchk(hipFree(A->row)); A->row = sym_row;
  gpuErrchk(hipFree(A->col)); A->col = sym_col;
  gpuErrchk(hipFree(A->val)); A->val = sym_val;
}
int main() {

    // --- Initialize cuSPARSE
    hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

    // --- Initialize matrix descriptors
    hipsparseMatDescr_t descrA, descrB, descrC;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
    cusparseSafeCall(hipsparseCreateMatDescr(&descrB));
    cusparseSafeCall(hipsparseCreateMatDescr(&descrC));

    const int M = 6;                                    // --- Number of rows
    const int N = 6;                                    // --- Number of columns

    const int nnz1 = 5400000;                                // --- Number of non-zero blocks for matrix A

    // --- Host vectors defining the first block-sparse matrix
    sparse_matrix<coord> Ah;
    Ah.val = (coord *)malloc(nnz1 * sizeof(coord));
    Ah.row = (int *)malloc((M + 1) * sizeof(int));
    Ah.col = (int *)malloc(nnz1 * sizeof(int));
    for(int i=0;i<n;<nnz1){
      Ah.val[i]
    }



    // --- Device vectors defining the block-sparse matrices
    sparse_matrix<coord> A;
       gpuErrchk(hipMalloc(&(A.val), nnz1 * sizeof(coord)));
        gpuErrchk(hipMalloc(&(A.row), (M + 1) * sizeof(int)));
        gpuErrchk(hipMalloc(&(A.col), nnz1 * sizeof(int)));


    gpuErrchk(hipMemcpy(A.val, Ah.val, nnz1 * sizeof(coord), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(A.row, Ah.row, (M + 1) * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(A.col, Ah.col, nnz1 * sizeof(int), hipMemcpyHostToDevice));
    A.n=N;
    A.nnz=nnz1;
    A.m=M;
    symmetrizeMatrixGPU(&A,handle);


    // --- Transforming csr to dense format
    coord *d_C;             gpuErrchk(hipMalloc(&d_C, M * N * sizeof(coord)));
    cusparseSafeCall(hipsparseScsr2dense(handle, M, N, descrC, A.val, A.row, A.col, d_C, M));

    coord *h_C = (coord *)malloc(M * N * sizeof(coord));
    gpuErrchk(hipMemcpy(h_C, d_C, M * N * sizeof(coord), hipMemcpyDeviceToHost));

    // --- m is row index, n column index
    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            printf("%lf ", h_C[m + n * M]);
        }
        printf("\n");
    }

    return 0;
}
