#include "hip/hip_runtime.h"

#include "../gridding.cuh"
#include "../gridding.hpp"
#include "../relocateData.cuh"
#include "../relocateData.hpp"
#include "../utils_cuda.cuh"
#include <iostream>
#include <stdio.h>
#include <sys/time.h>
using namespace std;
#include "../types.hpp"
template <class dataPoint>
dataPoint maxerror(dataPoint *const w, dataPoint *dv, int n, int d,
              const char *message) {

  dataPoint *v = (dataPoint *)malloc(n * d * sizeof(dataPoint));
  hipMemcpy(v, dv, d * n * sizeof(dataPoint), hipMemcpyDeviceToHost);
  dataPoint maxError = 0;
  dataPoint avgError = 0;
  int s=20;
  for(int j=0;j<d;j++){
  for (int i = 0; i < n ; i++) {

    if((v[i*d+j] - w[i*d+j]) * (v[i*d+j] - w[i*d+j]>0.1 && s>0))
      {printf("i=%d d=%d ",i,j );
        printf("%lf vs %lf\n",v[i*d+j],w[i*d+j] );
        s--;
    }

    if ((v[i*d+j] - w[i*d+j]) * (v[i*d+j] - w[i*d+j]) > maxError) {
      maxError = (v[i*d+j] - w[i*d+j]) * (v[i*d+j] - w[i*d+j]);

    }

    avgError += (v[i*d+j] - w[i*d+j]) * (v[i*d+j] - w[i*d+j]);
  }}

  //printf("maxError=%lf pos=%d v[i]=%lf vs w[i]=%lf avgError=%lf n=%d size=%d\n",maxError, pos, v[pos], w[pos], avgError / (n * d), n, n * d);
  free(v);
  return maxError;
}



coord *generateRandomCoord(int n, int d) {

  coord *y = (coord *)malloc(n * d * sizeof(coord));

  for (int i = 0; i < n * d; i++)
    y[i] = ((coord)rand() / (RAND_MAX)) * 100;

  return y;
}

template <class dataPoint>
void copydata(dataPoint *const w, dataPoint *dw, int n, int d) {
  dataPoint *v = (dataPoint *)malloc(sizeof(dataPoint) * n * d);
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < d; j++) {

      v[i + n * j] = w[i * d + j];
    }
  }
  hipMemcpy(dw, v, d * n * sizeof(dataPoint), hipMemcpyHostToDevice);
  free(v);
  return;
}

__global__ void Normalize(coord *y, uint32_t nPts, uint32_t ng, uint32_t d,
                          coord maxy) {
  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {
    for (int dim = 0; dim < d; dim++) {
      y[TID + dim * nPts] /= maxy;
      if (y[TID + dim * nPts] == 1) {
        y[TID + dim * nPts] = y[TID + dim * nPts] - 0.00000000000001;
      }
      y[TID + dim * nPts] *= (ng - 3);
    }
  }
}
void s2gCPU(coord *VGrid, coord *y, coord *VScat, uint32_t ng, uint32_t n,
            uint32_t d) {
  switch (d) {

  case 1:
    s2g1dCpu(VGrid, y, VScat, ng + 2, 1, n, d, d + 1);
    break;
  case 2:
    s2g2dCpu(VGrid, y, VScat, ng + 2, 1, n, d, d + 1);
    break;
  case 3:
    s2g3dCpu(VGrid, y, VScat, ng + 2, 1, n, d, d + 1);
    break;
  }
}

void testgridding(uint32_t *ib, uint32_t *cb, int *ib_d, uint32_t *cb_d,
                    coord *VGrid, coord *y, coord *VScat, coord *VGrid_d,
                    coord *y_d, coord *VScat_d, uint32_t n, uint32_t d,
                    uint32_t ng, int iterations) {

  double *timecpu = (double *)malloc(iterations * sizeof(double));
  double *timegpu = (double *)malloc(iterations * sizeof(double));
  double *timegpuwarp = (double *)malloc(iterations * sizeof(double));

  int szV = pow(ng + 2, d) * (d + 1);
double errorsimple=0;

  struct timeval t1, t2;
  double elapsedTime;

  for (int i = 0; i < iterations; i++) {
    for (int j = 0; j < szV; j++) {VGrid[j] = 0;}

    gettimeofday(&t1, NULL);
    s2gCPU(VGrid, y, VScat, ng, n, d);

    gettimeofday(&t2, NULL);
    elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;    // sec to ms
    elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
    timecpu[i] = elapsedTime;

    initKernel<<<64, 256>>>(VGrid_d, (coord)0, szV);
    hipDeviceSynchronize();
    gettimeofday(&t1, NULL);
    s2g(VGrid_d, y_d, VScat_d, ng + 2, n, d, d + 1);
    hipDeviceSynchronize();
    gettimeofday(&t2, NULL);
    elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;    // sec to ms
    elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
    timegpu[i] = elapsedTime;
    errorsimple=maxerror(VGrid, VGrid_d, pow(ng + 2, d), d + 1, "Gridding simple");


    initKernel<<<64, 256>>>(VGrid_d, (coord)0, szV);
    hipDeviceSynchronize();

    gettimeofday(&t1, NULL);

    s2gwarp(VGrid_d, y_d, VScat_d, ib_d, ng + 2, n, d, d + 1);

    hipDeviceSynchronize();
    gettimeofday(&t2, NULL);

    elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;    // sec to ms
    elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
    timegpuwarp[i] = elapsedTime;

  }
  double errorwarp=maxerror(VGrid, VGrid_d, pow(ng + 2, d), d + 1, "Gridding warp");

  printf("n=%d d=%d ng=%d\n",n,d,ng );
  printf("cpu: ");
  for (int i = 0; i < iterations; i++) {
    printf("%lf ", timecpu[i]);
  }
  printf("\n");
  printf("gpuwarp: ");
  for (int i = 0; i < iterations; i++) {
    printf("%lf ", timegpuwarp[i]);
  }
  printf("\n");
  printf("gpu: ");
  for (int i = 0; i < iterations; i++) {
    printf("%lf ", timegpu[i]);
  }

  printf("\n");
  printf("Errorsimple=%lf\n",errorsimple );
  //printf("Errorsim=%lf\n",errorsim );

  printf("Errorwarp=%lf\n",errorwarp );
  printf("\n" );
  free(timecpu);
  free(timegpu);
  free(timegpuwarp);
}

int main(int argc, char **argv) {
  CUDA_CALL(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
  CUDA_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
  srand(time(NULL));

  int n = 1 << atoi(argv[1]);
  int d = atoi(argv[2]);
  int ng = atoi(argv[3]);
  int iterations = atoi(argv[4]);

  coord *y, *y_d;

  CUDA_CALL(hipMallocManaged(&y_d, (d)*n * sizeof(coord)));
  y = generateRandomCoord(n, d);
  copydata(y, y_d, n, d);
  uint32_t  *cb, *ib_h, *cb_h;
  int * ib;
  uint32_t points= pow(ng-1, d)+1;
  CUDA_CALL(hipMallocManaged(&ib, points * sizeof(int)));
  CUDA_CALL(hipMallocManaged(&cb, points * sizeof(uint32_t)));
  ib_h = (uint32_t *)calloc(points, sizeof(uint32_t));
  cb_h = (uint32_t *)calloc(points, sizeof(uint32_t));
  hipMemset(cb, 0, points * sizeof(uint32_t));

  uint32_t *iPerm_h = (uint32_t *)malloc(n * sizeof(uint32_t));


  for (int j = 0; j < n; j++) {
    iPerm_h[j] = j;
  }
  uint32_t *iPerm;
  CUDA_CALL(hipMallocManaged(&iPerm, n * sizeof(uint32_t)));
  hipMemcpy(iPerm, iPerm_h, n * sizeof(uint32_t), hipMemcpyHostToDevice);

  relocateCoarseGrid(y_d, iPerm, ib, n, ng, d);

  relocateCoarseGridCPU(&y, &iPerm_h, ib_h, cb_h, n, ng, d, 1);
/*
      uint32_t* ib_c = (uint32_t *)calloc(points, sizeof(uint32_t));
      uint32_t* cb_c = (uint32_t *)calloc(points, sizeof(uint32_t));
      hipMemcpy(ib_c, ib, points * sizeof(uint32_t), hipMemcpyDeviceToHost);
      hipMemcpy(cb_c, cb, points * sizeof(uint32_t), hipMemcpyDeviceToHost);
      for(int i=0;i<points;i++){
        if(ib_c[i]!=ib_h[i]){printf("Error " );}
          printf("i=%d gpu= %d cpu= %d\n",i,ib_c[i],ib_h[i] );
      }
      */
  //hipMemcpy(ib, ib_h, points * sizeof(uint32_t), hipMemcpyHostToDevice);
  //hipMemcpy(cb, cb_h, points * sizeof(uint32_t), hipMemcpyHostToDevice);
  //hipMemcpy(ib_h, ib, points * sizeof(uint32_t), hipMemcpyDeviceToHost);
  //hipMemcpy(cb_h, cb, points * sizeof(uint32_t), hipMemcpyDeviceToHost);

  coord *VScat = generateRandomCoord(n, d + 1);
  coord *VScat_d;
  CUDA_CALL(hipMallocManaged(&VScat_d, (d + 1) * n * sizeof(coord)));
  copydata(VScat, VScat_d, n, d + 1);

  coord maxy = 0;
  for (int i = 0; i < n * d; i++)
    maxy = maxy < y[i] ? y[i] : maxy;
  for (int i = 0; i < n * d; i++) {
    y[i] /= maxy;

    if (1 == y[i])
      y[i] = y[i] - std::numeric_limits<coord>::epsilon();

    y[i] *= (ng - 1);
  }
  Normalize<<<64, 256>>>(y_d, n, ng + 2, d, maxy);

  int szV = pow(ng + 2, d) * (d + 1);

  coord *VGrid = (coord *)calloc(szV, sizeof(coord));
  coord *VGrid_d;
  CUDA_CALL(hipMallocManaged(&VGrid_d, szV * sizeof(coord)));

  initKernel<<<64, 256>>>(VGrid_d, (coord)0, szV);
  testgridding(ib_h, cb_h, ib, cb, VGrid, y, VScat, VGrid_d, y_d, VScat_d, n,
                 d, ng, iterations);
  free(y);
  free(ib_h);
  free(cb_h);
  free(iPerm_h);
  free(VGrid);
  free(VScat);
  hipFree(y_d);
  hipFree(ib);
  hipFree(cb);
  hipFree(iPerm);
  hipFree(VGrid_d);
  hipFree(VScat_d);

  hipDeviceReset();
  return 0;
}
