#include "hip/hip_runtime.h"
#include "relocateData.cuh"
extern hipStream_t streamRep;

template <class dataPoint>
__global__ void generateBoxIdx(uint64_t *Code, const dataPoint *Y, dataPoint scale,
                               const int nPts, const int nDim, const int nGrid,
                               const dataPoint multQuant, const uint32_t qLevel) {
  register uint64_t C[3];
  register dataPoint Yscale;
  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {
    for (int j = 0; j < nDim; j++) {
      Yscale = Y[TID + j * nPts] / scale;
      if (Yscale >= 1) {
        Yscale = 1 - 0.00000000000001;
      }
      C[j] = (uint64_t)abs(floor(multQuant * Yscale));
    }
    switch (nDim) {

    case 1:
      Code[TID] = (uint64_t)C[0];

    case 2:
      Code[TID] = (((uint64_t)C[1]) << qLevel) | (((uint64_t)C[0]));

    case 3:
      Code[TID] = (((uint64_t)C[2]) << 2 * qLevel) |
                  (((uint64_t)C[1]) << qLevel) | ((uint64_t)C[0]);
    }
  }
  return;
}
__inline__ __device__ uint32_t untangleLastDimDevice(int nDim, int TID,
                                                     uint32_t qLevel,
                                                     uint64_t *C) {
  uint64_t mask;
  switch (nDim) {
  case 1:
    mask = (1 << qLevel) - 1;
    return (uint32_t)(C[TID] & mask);

  case 2: {
    mask = (1 << 2 * qLevel) - 1;

    return (uint32_t)((C[TID] & mask) >> qLevel);
  }

  case 3: {
    mask = (1 << 3 * qLevel) - 1;

    return (uint32_t)((C[TID] & mask) >> 2 * qLevel);
  }
  }
  return 0;
}

__inline__ __device__ uint32_t gridIdxKernel(int nDim, int TID, uint32_t qLevel,
                                             uint64_t *C, int nGrid) {
  switch (nDim) {
  case 1:
    return untangleLastDimDevice(1, TID, qLevel, C);
  case 2:
    return untangleLastDimDevice(2, TID, qLevel, C) * nGrid +
           untangleLastDimDevice(1, TID, qLevel, C);
  case 3:
    return untangleLastDimDevice(3, TID, qLevel, C) * nGrid * nGrid +
           untangleLastDimDevice(2, TID, qLevel, C) * nGrid +
           untangleLastDimDevice(1, TID, qLevel, C);
  }
  return 0;
}

__global__ void gridIdxKernelnew(int *ib, uint64_t *C, int nPts, int nDim,
                                 int nGrid, uint32_t qLevel) {
  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x + 1; TID < nPts;
       TID += gridDim.x * blockDim.x) {
    uint32_t idxCur = gridIdxKernel(nDim, TID, qLevel, C, nGrid - 1);
    uint32_t idxPrev = gridIdxKernel(nDim, TID - 1, qLevel, C, nGrid - 1);
    if (idxCur != idxPrev) {
      ib[idxCur] = TID;
      if (idxPrev + 1 != idxCur) {
        ib[idxPrev + 1] = TID;
      }
    }
    if (TID == nPts - 1) {
      ib[idxCur + 1] = nPts;
    }
    if (TID == 1) {
      ib[gridIdxKernel(nDim, 0, qLevel, C, nGrid - 1)] = 0;
    }
  }
}
template <class dataPoint>
void relocateCoarseGrid(dataPoint *Yptr,     // Scattered point coordinates
                        uint32_t *iPerm, // Data relocation permutation
                        int *ib, // Starting index of box (along last dimension)
                        int nPts,  // Number of data points
                        int nGrid, // Grid dimensions (+1)
                        int nDim   // Number of dimensions
) {
  dataPoint *y_d = Yptr;
  thrust::device_ptr<dataPoint> yVec_ptr(y_d);
  dataPoint maxy =
      thrust::reduce(thrust::cuda::par.on(streamRep), yVec_ptr,
                     yVec_ptr + nPts * nDim, 0.0, thrust::maximum<dataPoint>());
  uint64_t *Codes;
  CUDA_CALL(hipMallocManaged(&Codes, nPts * sizeof(uint64_t)));
  dataPoint multQuant = nGrid - 1 - std::numeric_limits<dataPoint>::epsilon();
  uint32_t qLevel = 0;
  qLevel = ceil(log(nGrid) / log(2));
  generateBoxIdx<<<64, 256>>>(Codes, y_d, maxy, nPts, nDim, nGrid, multQuant,
                              qLevel);

  CUDA_CALL(hipDeviceSynchronize());
  thrust ::device_ptr<uint64_t> Codes_ptr(Codes);
  thrust::device_ptr<uint32_t> iPerm_ptr(iPerm);

  switch (nDim) {
  case 1:
    thrust ::stable_sort_by_key(
        Codes_ptr, Codes_ptr + nPts,
        make_zip_iterator(make_tuple(yVec_ptr, iPerm_ptr)));

  case 2:
    thrust ::stable_sort_by_key(
        Codes_ptr, Codes_ptr + nPts,
        make_zip_iterator(make_tuple(yVec_ptr, yVec_ptr + nPts, iPerm_ptr)));

  case 3:
    thrust ::stable_sort_by_key(
        Codes_ptr, Codes_ptr + nPts,
        make_zip_iterator(make_tuple(yVec_ptr, yVec_ptr + nPts,
                                     yVec_ptr + 2 * nPts, iPerm_ptr)));
  }
  CUDA_CALL(hipDeviceSynchronize());

  uint32_t points = pow(nGrid - 1, nDim) + 1;
  CUDA_CALL(hipMemset(ib, -1, points * (sizeof(int))));
  gridIdxKernelnew<<<32, 256>>>(ib, Codes, nPts, nDim, nGrid, qLevel);

  CUDA_CALL(hipFree(Codes));
  return;
}
template
void relocateCoarseGrid(float *Yptr,     // Scattered point coordinates
                        uint32_t *iPerm, // Data relocation permutation
                        int *ib, // Starting index of box (along last dimension)
                        int nPts,  // Number of data points
                        int nGrid, // Grid dimensions (+1)
                        int nDim   // Number of dimensions
);
template
void relocateCoarseGrid(double *Yptr,     // Scattered point coordinates
                        uint32_t *iPerm, // Data relocation permutation
                        int *ib, // Starting index of box (along last dimension)
                        int nPts,  // Number of data points
                        int nGrid, // Grid dimensions (+1)
                        int nDim   // Number of dimensions
);
