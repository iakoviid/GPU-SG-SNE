#include "hip/hip_runtime.h"
#include "matrix_indexing.hpp"
#include "non_periodic_conv.cuh"
#include "utils_cuda.cuh"
#include <sys/time.h>

#define idx2(i, j, d) (SUB2IND2D(i, j, d))
#define idx3(i, j, k, d1, d2) (SUB2IND3D(i, j, k, d1, d2))
#define idx4(i, j, k, l, m, n, o) (SUB2IND4D(i, j, k, l, m, n, o))

#define HIP_PI_F 3.141592654f
// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(Complex *a, const Complex *b,
                                                   int size,uint32_t nVec) {
  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  for (int j = 0; j < nVec; j++) {

  for (int i = threadID; i < size; i += numThreads) {
    a[i+j*size] = ComplexScale(ComplexMul(a[i+j*size], b[i]), 1.0f);
  }
}
}

__global__ void setDataFft1D(Complex *Kc, Complex *Xc, int ng, int nVec,
                             coord *VGrid, coord hsq, int sign) {

  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = threadID; i < ng; i += numThreads) {
    Kc[i].x = kernel1d(hsq, i);
    if (i > 0) {
      Kc[i].x = Kc[i].x + sign * kernel1d(hsq, ng - i);
      if (sign == -1) {

        Complex arg;
        arg.x = 0;
        arg.y = -2 * HIP_PI_F * i / (2 * ng);
        Kc[i] = ComplexMul(Kc[i], my_cexpf(arg));
      }
    }
    for (int j = 0; j < nVec; j++) {
      Xc[i + j * ng].x = VGrid[i + j * ng];
      if (sign == -1) {
        Complex arg;
        arg.x = 0;
        arg.y = -2 * HIP_PI_F * i / (2 * ng);
        Xc[i + j * ng] = ComplexMul(Xc[i + j * ng], my_cexpf(arg));
      }
    }
  }
}

__global__ void setDataFft2D(Complex *Kc, Complex *Xc, int n1, int n2, int nVec,
                             const coord *const VGrid, coord hsq, int signx,
                             int signy) {

  for (uint32_t j = blockIdx.y * blockDim.y + threadIdx.y; j < n2;
       j += blockDim.y * gridDim.y) {
    for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n1;
         i += blockDim.x * gridDim.x) {
      Kc[idx2(i, j, n1)].x = kernel2d(hsq, i, j);
      if (i > 0) {
        Kc[idx2(i, j, n1)].x += signx * kernel2d(hsq, n1 - i, j);
      }
      if (j > 0) {
        Kc[idx2(i, j, n1)].x += signy * kernel2d(hsq, i, n2 - j);
      }
      if (i > 0 && j > 0) {
        Kc[idx2(i, j, n1)].x += signx * signy * kernel2d(hsq, n1 - i, n2 - j);
      }

      for (uint32_t iVec = 0; iVec < nVec; iVec++) {
        Xc[idx3(i, j, iVec, n1, n2)].x = VGrid[idx3(i, j, iVec, n1, n2)];
        if (signx == -1) {
          Complex arg;
          arg.x = 0;
          arg.y = -2 * HIP_PI_F * i / (2 * n1);
          Xc[idx3(i, j, iVec, n1, n2)] =
              ComplexMul(Xc[idx3(i, j, iVec, n1, n2)], my_cexpf(arg));
        }
        if (signy == -1) {
          Complex arg;
          arg.x = 0;
          arg.y = -2 * HIP_PI_F * j / (2 * n2);
          Xc[idx3(i, j, iVec, n1, n2)] =
              ComplexMul(Xc[idx3(i, j, iVec, n1, n2)], my_cexpf(arg));
        }
      }
      if (signx == -1) {
        Complex arg;
        arg.x = 0;
        arg.y = -2 * HIP_PI_F * i / (2 * n1);
        Kc[idx2(i, j, n1)] = ComplexMul(Kc[idx2(i, j, n1)], my_cexpf(arg));
      }

      if (signy == -1) {
        Complex arg;
        arg.x = 0;
        arg.y = -2 * HIP_PI_F * j / (2 * n2);
        Kc[idx2(i, j, n1)] = ComplexMul(Kc[idx2(i, j, n1)], my_cexpf(arg));
      }
    }
  }
}
__global__ void addToPhiGrid(Complex *Xc, coord *PhiGrid, int ng, coord scale) {

  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = threadID; i < ng; i += numThreads) {
    PhiGrid[i] += scale * Xc[i].x;
  }
}

__global__ void normalizeInverse(Complex *Xc, int ng, uint32_t nVec) {

  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = threadID; i < ng; i += numThreads) {
    for (uint32_t iVec = 0; iVec < nVec; iVec++) {
      Complex arg;
      arg.x = 0;
      arg.y = +2 * HIP_PI_F * i / (2 * ng);
      Xc[i + iVec * ng] = ComplexMul(Xc[i + iVec * ng], my_cexpf(arg));
    }
  }
}

__global__ void normalizeInverse2D(Complex *Xc, uint32_t n1, uint32_t n2,
                                   uint32_t nVec, int signx, int signy) {

  for (uint32_t j = blockIdx.y * blockDim.y + threadIdx.y; j < n2;
       j += blockDim.y * gridDim.y) {
    for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n1;
         i += blockDim.x * gridDim.x) {
      for (uint32_t iVec = 0; iVec < nVec; iVec++) {
        if (signx == -1) {
          Complex arg;
          arg.x = 0;
          arg.y = +2 * HIP_PI_F * i / (2 * n1);
          Xc[idx3(i, j, iVec, n1, n2)] =
              ComplexMul(Xc[idx3(i, j, iVec, n1, n2)], my_cexpf(arg));
        }
        if (signy == -1) {
          Complex arg;
          arg.x = 0;
          arg.y = +2 * HIP_PI_F * j / (2 * n2);
          Xc[idx3(i, j, iVec, n1, n2)] =
              ComplexMul(Xc[idx3(i, j, iVec, n1, n2)], my_cexpf(arg));
        }
      }
    }
  }
}

void conv1dnopadcuda(coord *PhiGrid, coord *VGrid, coord h,
                     uint32_t *const nGridDims, uint32_t nVec, int nDim) {

  uint32_t n1 = nGridDims[0];
  coord hsq = h * h;
  Complex *Kc, *Xc;
  CUDA_CALL(hipMallocManaged(&Kc, n1 * sizeof(Complex)));
  CUDA_CALL(hipMallocManaged(&Xc, nVec * n1 * sizeof(Complex)));

  hipfftHandle plan, plan_rhs;
  int ng[1] = {(int)n1};

  hipfftPlan1d(&plan, n1, HIPFFT_C2C, 1);
  hipfftPlanMany(&plan_rhs, 1, ng, NULL, 1, n1, NULL, 1, n1,
                HIPFFT_C2C, nVec);
                struct timeval t1, t2;
                double elapsedTime;
                gettimeofday(&t1, NULL);

  /*even*/
  setDataFft1D<<<32, 256>>>(Kc, Xc, n1, nVec, VGrid, hsq, 1);

  hipDeviceSynchronize(); // why

  hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(Kc),
               reinterpret_cast<hipfftComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_FORWARD);

    ComplexPointwiseMulAndScale<<<32, 256>>>(Xc, Kc, n1,nVec);

  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_BACKWARD);
  addToPhiGrid<<<32, 256>>>(Xc, PhiGrid, n1 * nVec, (0.5 / n1));

  hipDeviceSynchronize(); // why

  setDataFft1D<<<64, 1024>>>(Kc, Xc, n1, nVec, VGrid, hsq, -1);

  hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(Kc),
               reinterpret_cast<hipfftComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_FORWARD);

  //for (int j = 0; j < nVec; j++) {

    //ComplexPointwiseMulAndScale<<<32, 256>>>(&Xc[j * n1], Kc, n1,
                                          //   1.0f);
  //}
  ComplexPointwiseMulAndScale<<<32, 256>>>(Xc, Kc, n1,nVec);

  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_BACKWARD);

  normalizeInverse<<<32, 256>>>(Xc, n1, nVec);

  addToPhiGrid<<<32, 256>>>(Xc, PhiGrid, n1 * nVec, (0.5 / n1));

  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);

  elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;    // sec to ms
  elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
  printf("CUDA nopad time %lf\n",elapsedTime );

  hipFree(Kc);
  hipFree(Xc);
  return;
}
void conv2dnopadcuda(coord *const PhiGrid, const coord *const VGrid,
                     const coord h, uint32_t *const nGridDims,
                     const uint32_t nVec, const uint32_t nDim) {
  coord hsq = h * h;
  Complex *Kc, *Xc;

  // find the size of the last dimension in FFTW (add padding)
  uint32_t n1 = nGridDims[0];
  uint32_t n2 = nGridDims[1];
  int ng[2] = {(int)n1, (int)n2};
  CUDA_CALL(hipMallocManaged(&Kc, n1 * n2 * sizeof(Complex)));
  CUDA_CALL(hipMallocManaged(&Xc, nVec * n1 * n2 * sizeof(Complex)));
  hipfftHandle plan, plan_rhs;
  hipfftPlanMany(&plan, 2, ng, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, 1);
  hipfftPlanMany(&plan_rhs, 2, ng, NULL, 1, n1 * n2, NULL, 1, n1 * n2, HIPFFT_C2C,
                nVec);
                struct timeval t1, t2;
                double elapsedTime;
                gettimeofday(&t1, NULL);
  // ============================== EVEN-EVEN

  setDataFft2D<<<32, 256>>>(Kc, Xc, n1, n2, nVec, VGrid, hsq, 1, 1);
  hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(Kc),
               reinterpret_cast<hipfftComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_FORWARD);

  for (int j = 0; j < nVec; j++) {
    ComplexPointwiseMulAndScale<<<32, 256>>>(&Xc[j * n1 * n2], Kc, n1 * n2,
                                             1.0f);
  }

  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_BACKWARD);
  addToPhiGrid<<<32, 256>>>(Xc, PhiGrid, n1 * n2 * nVec, (0.25 / (n1 * n2)));

  // ============================== ODD-EVEN

  setDataFft2D<<<32, 256>>>(Kc, Xc, n1, n2, nVec, VGrid, hsq, -1, 1);
  hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(Kc),
               reinterpret_cast<hipfftComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_FORWARD);

  for (int j = 0; j < nVec; j++) {
    ComplexPointwiseMulAndScale<<<32, 256>>>(&Xc[j * n1 * n2], Kc, n1 * n2,
                                             1.0f);
  }
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_BACKWARD);
  normalizeInverse2D<<<32, 256>>>(Xc, n1, n2, nVec, -1, 1);
  addToPhiGrid<<<32, 256>>>(Xc, PhiGrid, n1 * n2 * nVec, (0.25 / (n1 * n2)));

  // ============================== EVEN-ODD

  setDataFft2D<<<32, 256>>>(Kc, Xc, n1, n2, nVec, VGrid, hsq, 1, -1);
  hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(Kc),
               reinterpret_cast<hipfftComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_FORWARD);
  for (int j = 0; j < nVec; j++) {
    ComplexPointwiseMulAndScale<<<32, 256>>>(&Xc[j * n1 * n2], Kc, n1 * n2,
                                             1.0f);
  }
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_BACKWARD);
  normalizeInverse2D<<<32, 256>>>(Xc, n1, n2, nVec, 1, -1);

  addToPhiGrid<<<32, 256>>>(Xc, PhiGrid, n1 * n2 * nVec, (0.25 / (n1 * n2)));

  // ============================== ODD-ODD

  setDataFft2D<<<32, 256>>>(Kc, Xc, n1, n2, nVec, VGrid, hsq, -1, -1);
  hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(Kc),
               reinterpret_cast<hipfftComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_FORWARD);

  for (int j = 0; j < nVec; j++) {
    ComplexPointwiseMulAndScale<<<32, 256>>>(&Xc[j * n1 * n2], Kc, n1 * n2,
                                             1.0f);
  }
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_BACKWARD);

  normalizeInverse2D<<<32, 256>>>(Xc, n1, n2, nVec, -1, -1);
  addToPhiGrid<<<32, 256>>>(Xc, PhiGrid, n1 * n2 * nVec, (0.25 / (n1 * n2)));
  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);

  elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;    // sec to ms
  elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
  printf("CUDA nopad time %lf\n",elapsedTime );

}

__global__ void setDataFft3D(Complex *Kc, Complex *Xc, int n1, int n2, int n3, int nVec,
                             const coord *const VGrid, coord hsq, int signx,
                             int signy, int signz) {
  for (uint32_t k = blockIdx.z * blockDim.z + threadIdx.z; k < n3;
       k += blockDim.z * gridDim.z) {

    for (uint32_t j = blockIdx.y * blockDim.y + threadIdx.y; j < n2;
         j += blockDim.y * gridDim.y) {
      for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n1;
           i += blockDim.x * gridDim.x) {
        Kc[idx3(i, j, k, n1, n2)].x = kernel3d(hsq, i, j, k);
        if (i > 0) {
          Kc[idx3(i, j, k, n1, n2)].x += signx * kernel3d(hsq, n1 - i, j, k);
        }
        if (j > 0) {
          Kc[idx3(i, j, k, n1, n2)].x += signx * kernel3d(hsq, i,n2 - j, k);
        }
        if (i > 0 && j > 0) {
          Kc[idx3(i, j, k, n1, n2)].x += signx * kernel3d(hsq, n1 - i,n2 - j, k);
        }
        if (k > 0) {
          Kc[idx3(i, j, k, n1, n2)].x += signx * kernel3d(hsq, i, j,n3 - k);
        }
        if (k > 0 && i > 0) {
          Kc[idx3(i, j, k, n1, n2)].x += signx * kernel3d(hsq, n1 - i, j,n3 - k);
        }
        if (k > 0 && j > 0) {
          Kc[idx3(i, j, k, n1, n2)].x += signx * kernel3d(hsq, i,n2- j,n3- k);
        }
        if (k > 0 && i > 0 && j > 0) {
          Kc[idx3(i, j, k, n1, n2)].x += signx * kernel3d(hsq, n1 - i,n2- j,n3- k);
        }

        for (uint32_t iVec = 0; iVec < nVec; iVec++) {
          Xc[idx4(i, j, k, iVec, n1, n2, n3)].x =
              VGrid[idx4(i, j, k, iVec, n1, n2, n3)];
          if (signx == -1) {
            Complex arg;
            arg.x = 0;
            arg.y = -2 * HIP_PI_F * i / (2 * n1);
            Xc[idx4(i, j, k, iVec, n1, n2, n3)] =
                ComplexMul(Xc[idx4(i, j, k, iVec, n1, n2, n3)], my_cexpf(arg));
          }
          if (signy == -1) {
            Complex arg;
            arg.x = 0;
            arg.y = -2 * HIP_PI_F * j / (2 * n2);
            Xc[idx4(i, j, k, iVec, n1, n2, n3)] =
                ComplexMul(Xc[idx4(i, j, k, iVec, n1, n2, n3)], my_cexpf(arg));
          }
          if (signz == -1) {
            Complex arg;
            arg.x = 0;
            arg.y = -2 * HIP_PI_F * k / (2 * n3);
            Xc[idx4(i, j, k, iVec, n1, n2, n3)] =
                ComplexMul(Xc[idx4(i, j, k, iVec, n1, n2, n3)], my_cexpf(arg));
          }
        }
        if (signx == -1) {
          Complex arg;
          arg.x = 0;
          arg.y = -2 * HIP_PI_F * i / (2 * n1);
          Kc[idx3(i, j, k, n1, n2)] =
              ComplexMul(Kc[idx3(i, j, k, n1, n2)], my_cexpf(arg));
        }

        if (signy == -1) {
          Complex arg;
          arg.x = 0;
          arg.y = -2 * HIP_PI_F * j / (2 * n2);
          Kc[idx3(i, j, k, n1, n2)] =
              ComplexMul(Kc[idx3(i, j, k, n1, n2)], my_cexpf(arg));
        }

        if (signz == -1) {
          Complex arg;
          arg.x = 0;
          arg.y = -2 * HIP_PI_F * k / (2 * n3);
          Kc[idx3(i, j, k, n1, n2)] =
              ComplexMul(Kc[idx3(i, j, k, n1, n2)], my_cexpf(arg));
        }
      }
    }
  }
}

__global__ void normalizeInverse3D(Complex *Xc, uint32_t n1, uint32_t n2,
                                   uint32_t n3, uint32_t nVec, int signx,
                                   int signy, int signz) {

  for (uint32_t k = blockIdx.z * blockDim.z + threadIdx.z; k < n3;
       k += blockDim.z * gridDim.z) {

    for (uint32_t j = blockIdx.y * blockDim.y + threadIdx.y; j < n2;
         j += blockDim.y * gridDim.y) {
      for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n1;
           i += blockDim.x * gridDim.x) {
        for (uint32_t iVec = 0; iVec < nVec; iVec++) {
          if (signx == -1) {
            Complex arg;
            arg.x = 0;
            arg.y = +2 * HIP_PI_F * i / (2 * n1);
            Xc[idx4(i, j, k, iVec, n1, n2, n3)] =
                ComplexMul(Xc[idx4(i, j, k, iVec, n1, n2, n3)], my_cexpf(arg));
          }
          if (signy == -1) {
            Complex arg;
            arg.x = 0;
            arg.y = +2 * HIP_PI_F * j / (2 * n2);
            Xc[idx4(i, j, k, iVec, n1, n2, n3)] =
                ComplexMul(Xc[idx4(i, j, k, iVec, n1, n2, n3)], my_cexpf(arg));
          }
          if (signz == -1) {
            Complex arg;
            arg.x = 0;
            arg.y = +2 * HIP_PI_F * k / (2 * n3);
            Xc[idx4(i, j, k, iVec, n1, n2, n3)] =
                ComplexMul(Xc[idx4(i, j, k, iVec, n1, n2, n3)], my_cexpf(arg));
          }
        }
      }
    }
  }
}
void term3D(Complex *Kc, Complex *Xc, uint32_t n1, uint32_t n2, uint32_t n3,
            uint32_t nVec, const coord *const VGrid, coord *PhiGrid, coord hsq,
            hipfftHandle plan, hipfftHandle plan_rhs, int signx, int signy,
            int signz) {

  setDataFft3D<<<32, 256>>>(Kc, Xc, n1, n2, n3, nVec, VGrid, hsq, signx, signy,
                            signz);
  hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(Kc),
               reinterpret_cast<hipfftComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_FORWARD);

  for (int j = 0; j < nVec; j++) {
    ComplexPointwiseMulAndScale<<<32, 256>>>(&Xc[j * n1 * n2 * n3], Kc,
                                             n1 * n2 * n3, 1.0f);
  }
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_BACKWARD);
  normalizeInverse3D<<<32, 256>>>(Xc, n1, n2, n3, nVec, signx, signy, signz);
  addToPhiGrid<<<32, 256>>>(Xc, PhiGrid, n1 * n2 * n3 * nVec,
                            (0.125 / (n1 * n2 * n3)));
}

void conv3dnopadcuda(coord *const PhiGrid, const coord *const VGrid,
                     const coord h, uint32_t *const nGridDims,
                     const uint32_t nVec, const uint32_t nDim) {

  coord hsq = h * h;
  Complex *Kc, *Xc;

  // find the size of the last dimension in FFTW (add padding)
  uint32_t n1 = nGridDims[0];
  uint32_t n2 = nGridDims[1];
  uint32_t n3 = nGridDims[2];
  int ng[3] = {(int)n1,(int) n2,(int) n3};
  CUDA_CALL(hipMallocManaged(&Kc, n1 * n2 * n3 * sizeof(Complex)));
  CUDA_CALL(hipMallocManaged(&Xc, nVec * n1 * n2 * n3 * sizeof(Complex)));
  hipfftHandle plan, plan_rhs;
  hipfftPlanMany(&plan, 3, ng, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, 1);
  hipfftPlanMany(&plan_rhs, 3, ng, NULL, 1, n1 * n2 * n3, NULL, 1, n1 * n2 * n3,
                HIPFFT_C2C, nVec);

  // ============================== EVEN-EVEN-EVEN

  setDataFft3D<<<32, 256>>>(Kc, Xc, n1, n2, n3, nVec, VGrid, hsq, 1, 1, 1);
  hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(Kc),
               reinterpret_cast<hipfftComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_FORWARD);

  for (int j = 0; j < nVec; j++) {
    ComplexPointwiseMulAndScale<<<32, 256>>>(&Xc[j * n1 * n2 * n3], Kc,
                                             n1 * n2 * n3, 1.0f);
  }

  hipfftExecC2C(plan_rhs, reinterpret_cast<hipfftComplex *>(Xc),
               reinterpret_cast<hipfftComplex *>(Xc), HIPFFT_BACKWARD);
  addToPhiGrid<<<32, 256>>>(Xc, PhiGrid, n1 * n2 * n3 * nVec,
                            (0.125 / (n1 * n2 * n3)));

  // ============================== ODD-EVEN-EVEN

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, -1, 1,
         1);

  // ============================== EVEN-ODD-EVEN

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, 1, -1,
         1);

  // ============================== ODD-ODD-EVEN

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, -1, -1,
         1);

  // ============================== EVEN-EVEN-ODD

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, 1, 1,
         -1);

  // ============================== EVEN-ODD-EVEN

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, 1, -1,
         1);

  // ============================== EVEN-ODD-ODD

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, 1, -1,
         -1);

  // ============================== ODD-ODD-ODD

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, -1, -1,
         -1);
}
