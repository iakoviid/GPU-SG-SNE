#include "hip/hip_runtime.h"
#include "../bsr_pq.cuh"
#include "../matrix_converter.h"
#include "../types.hpp"
#include <algorithm>
#include <hipsparse.h>
#include <fstream>
#include <iostream>
#include <map>
#include <memory>
#include <random>
#include <set>
#include <sys/time.h>
#include <unordered_map>

#define CUDA_CALL(x)                                                           \
  {                                                                            \
    if ((x) != hipSuccess) {                                                  \
      printf("CUDA error at %s:%d\n", __FILE__, __LINE__);                     \
      printf("  %s\n", hipGetErrorString(hipGetLastError()));                \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }
using namespace std;
template <typename data_type>
__global__ void ell_spmv_kernel(unsigned int n, unsigned int elements_in_rows,
                                const unsigned int *col_ids,
                                const data_type *data, const data_type *Y,
                                data_type *Fatr, int d) {
  unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < n) {
    data_type sum1 = 0;
    data_type sum2 = 0;
    data_type sum3 = 0;

    for (unsigned int element = 0; element < elements_in_rows; element++) {
      const unsigned int element_offset = row + element * n;
      uint32_t column = col_ids[element_offset];
      coord dist = 0;
      for (int dim = 0; dim < d; dim++) {
        dist += (Y[row + dim * n] - Y[column + dim * n]) *
                (Y[row + dim * n] - Y[column + dim * n]);
      }
      const data_type value = data[element_offset] / (1 + dist);
      switch (d) {
      case 1:
        sum1 += value * (Y[row] - Y[column]);
        break;
      case 2:
        sum1 += value * (Y[row] - Y[column]);
        sum2 += value * (Y[row + n] - Y[column + n]);
        break;
      case 3:
        sum1 += value * (Y[row] - Y[column]);
        sum2 += value * (Y[row + n] - Y[column + n]);
        sum3 += value * (Y[row + 2 * n] - Y[column + 2 * n]);
        break;
      }
    }

    switch (d) {
    case 1:
      Fatr[row] = sum1;
      break;
    case 2:
      Fatr[row] = sum1;
      Fatr[row + n] = sum2;
      break;
    case 3:
      Fatr[row] = sum1;
      Fatr[row + n] = sum2;
      Fatr[row + 2 * n] = sum3;
      break;
    }
  }
}
template <typename data_type>
__global__ void
coo_spmv_kernel(unsigned int n_elements, const unsigned int *col_ids,
                const unsigned int *row_ids, const data_type *data,
                const data_type *Y, data_type *Fattr, int d, int n) {
  unsigned int element = blockIdx.x * blockDim.x + threadIdx.x;

  if (element < n_elements) {
    data_type dist = 0;
    uint32_t row = row_ids[element];
    uint32_t column = col_ids[element];
    for (int dim = 0; dim < d; dim++) {
      dist += (Y[row + dim * n] - Y[column + dim * n]) *
              (Y[row + dim * n] - Y[column + dim * n]);
    }
    data_type pq = data[element] / (1 + dist);
    switch (d) {
    case 1:
      atomicAdd(Fattr + row, pq * (Y[row] - Y[column]));
      break;
    case 2:
      atomicAdd(Fattr + row, pq * (Y[row] - Y[column]));
      atomicAdd(Fattr + row + n, pq * (Y[row + n] - Y[column + n]));

      break;
    case 3:
      atomicAdd(Fattr + row, pq * (Y[row] - Y[column]));
      atomicAdd(Fattr + row + n, pq * (Y[row + n] - Y[column + n]));
      atomicAdd(Fattr + row + 2 * n, pq * (Y[row + 2 * n] - Y[column + 2 * n]));

      break;
    }
  }
}

template <typename data_type>
void gpu_hybrid_spmv(const hybrid_matrix_class<data_type> &matrix, data_type *Y,
                     unsigned int rows_count, data_type *F,
                     unsigned int *ell_cols, data_type *ell_data,
                     data_type *coo_data, unsigned int *coo_row_ids,
                     unsigned int *coo_col_ids, int d) {

  /// ELL Part
  {
    dim3 block_size = dim3(512);
    dim3 grid_size{};

    grid_size.x = (rows_count + block_size.x - 1) / block_size.x;

    ell_spmv_kernel<<<grid_size, block_size>>>(
        rows_count, matrix.ell_matrix->elements_in_rows, ell_cols, ell_data, Y,
        F, d);
  }

  /// COO Part
  {
    dim3 block_size = dim3(512);
    dim3 grid_size{};

    const auto n_elements = matrix.coo_matrix->get_matrix_size();
    grid_size.x = (n_elements + block_size.x - 1) / block_size.x;

    coo_spmv_kernel<<<grid_size, block_size>>>(
        n_elements, coo_col_ids, coo_row_ids, coo_data, Y, F, d, rows_count);
  }
}
float get_random() {
  static std::default_random_engine e;
  static std::uniform_real_distribution<> dis(0, 1); // rage 0 - 1
  return dis(e);
}
int get_random_int(int max) {
  static std::default_random_engine e;
  std::uniform_int_distribution<int> dis(0, max - 1);
  return dis(e);
}

void generate_candidate_blocks(int R, int C, int BS_R, int BS_C, int num_blocks,
                               int *weight_indptr, int *weight_indices) {
  std::map<int, std::set<int>> blocks;
  int num_r_block = R;
  int num_c_block = C;
  int curr_size = 0;
  while (curr_size < num_blocks) {
    int r = get_random_int(num_r_block);
    int c = get_random_int(num_c_block);
    if (blocks[r].count(c) == 0) {
      blocks[r].insert(c);
      curr_size++;
    }
  }

  int current_ptr = 0;
  int i;
  for (i = 0; i < num_r_block; i++) {
    weight_indptr[i] = current_ptr;
    for (auto block : blocks[i]) {
      weight_indices[current_ptr++] = block;
    }
  }
  weight_indptr[i] = current_ptr;
}
__global__ void ComputePijxQijKernel(coord *__restrict__ attr_forces,
                                     const coord *__restrict__ pij,
                                     const coord *__restrict__ points,
                                     const int *__restrict__ coo_indicesi,
                                     const int *__restrict__ coo_indicesj,
                                     const int num_points,
                                     const int num_nonzero) {
  register int TID, i, j;
  register coord ix, iy, jx, jy, dx, dy, pijqij;
  TID = threadIdx.x + blockIdx.x * blockDim.x;
  if (TID >= num_nonzero)
    return;
  i = coo_indicesi[TID];
  j = coo_indicesj[TID];

  ix = points[i];
  iy = points[num_points + i];
  jx = points[j];
  jy = points[num_points + j];
  dx = ix - jx;
  dy = iy - jy;
  pijqij = pij[TID] / (1 + dx * dx + dy * dy);
  atomicAdd(attr_forces + i, pijqij * dx);
  atomicAdd(attr_forces + num_points + i, pijqij * dy);
}
__host__ __device__ int iDivUp(int a, int b) {
  return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

double tsne_cuda_spmv(coord *Fattr, coord *coov, coord *Y, int *cooi, int *cooj,
                      int n, int nnz, int d) {
  coord *Fattr_d, *Y_d, *coov_d;
  int *cooi_d, *cooj_d;
  CUDA_CALL(hipMalloc(&Fattr_d, n * d * sizeof(coord)));
  CUDA_CALL(hipMemcpy(Fattr_d, Fattr, n * d * sizeof(coord),
                       hipMemcpyHostToDevice));
  CUDA_CALL(hipMalloc(&Y_d, n * d * sizeof(coord)));
  CUDA_CALL(hipMemcpy(Y_d, Y, n * d * sizeof(coord), hipMemcpyHostToDevice));

  CUDA_CALL(hipMalloc(&cooi_d, nnz * sizeof(int)));
  CUDA_CALL(
      hipMemcpy(cooi_d, cooi, nnz * sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMalloc(&cooj_d, nnz * sizeof(int)));
  CUDA_CALL(
      hipMemcpy(cooj_d, cooj, nnz * sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMalloc(&coov_d, nnz * sizeof(coord)));
  CUDA_CALL(
      hipMemcpy(coov_d, coov, nnz * sizeof(coord), hipMemcpyHostToDevice));

  const int BLOCKSIZE = 1024;
  const int NBLOCKS = iDivUp(nnz, BLOCKSIZE);
  struct timeval t1, t2;
  double elapsedTime;
  gettimeofday(&t1, NULL);

  ComputePijxQijKernel<<<NBLOCKS, BLOCKSIZE>>>(Fattr_d, coov_d, Y_d, cooi_d,
                                               cooj_d, n, nnz);
  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);
  elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;    // sec to ms
  elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
  //  printf("COO elapsedTime=%lf\n", elapsedTime);
  CUDA_CALL(hipMemcpy(Fattr, Fattr_d, n * d * sizeof(coord),
                       hipMemcpyDeviceToHost));

  hipFree(Fattr_d);
  hipFree(Y_d);
  hipFree(cooi_d);
  hipFree(cooj_d);
  hipFree(coov_d);
  return elapsedTime;
}
template <class dataPoint>
dataPoint maxerror(dataPoint *const w, dataPoint *v, int n, int d) {

  dataPoint maxError = 0;
  dataPoint avgError = 0;
  int pos = 0;

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < d; j++) {
      if ((v[i + j * n] - w[i + j * n]) * (v[i + j * n] - w[i + j * n]) >
          maxError) {
        maxError =
            (v[i + j * n] - w[i + j * n]) * (v[i + j * n] - w[i + j * n]);
        pos = i;
      }
      avgError += (v[i + j * n] - w[i + j * n]) * (v[i + j * n] - w[i + j * n]);
    }
  }

  // printf("maxError=%lf pos=%d v[i]=%lf vs w[i]=%lf avgError=%lf n=%d
  // size=%d\n",
  //         maxError, pos, v[pos], w[pos], avgError / (n * d), n, n * d);

  return maxError;
}
double test_coo(coord *coov, int *cooi, int *cooj, int nnz, coord *Y, int n,
                int d, coord *Fserial) {
  coord *Fcoo = (coord *)calloc(sizeof(coord), n * d);

  double time = tsne_cuda_spmv(Fcoo, coov, Y, cooi, cooj, n, nnz, d);
  double maxError = maxerror(Fserial, Fcoo, n, d);
  if (maxError > 0.00001) {
    printf("Error coo\n");
  }
  free(Fcoo);
  return time;
}

void serial(coord *val, int *block_ptr, int *col_ind, coord *Y, coord *Fattr,
            int n, int bs, int d, int m) {
  for (int i = 0; i < n; i++) {
    int block_first = block_ptr[i];
    int block_last = block_ptr[i + 1];
    for (int block = block_first; block < block_last; block++) {
      for (int row = 0; row < bs; row++) {
        for (int col = 0; col < bs; col++) {
          // printf("%d  %d  %f \n", i * bs + row,
          int row_v = i * bs + row;

          int column = col_ind[block] * bs + col;
          coord dist = 0;
          for (int dim = 0; dim < d; dim++) {
            dist += (Y[row_v + dim * m] - Y[column + dim * m]) *
                    (Y[row_v + dim * m] - Y[column + dim * m]);
          }
          for (int dim = 0; dim < d; dim++) {
            Fattr[row_v + m * dim] +=
                val[block * bs * bs + row * bs + col] *
                (Y[row_v + m * dim] - Y[column + m * dim]) / (dist + 1);
          }
        }
      }
    }
  }
}

double bsr1run(coord *val, int *block_ptr, int *col_ind, coord *y, coord *Fattr,
               int n, int blockSize, int num_blocks, int rows, int d) {
  coord *cu_y;
  coord *cu_F;
  CUDA_CALL(hipMalloc(&cu_y, rows * d * sizeof(coord)));
  CUDA_CALL(
      hipMemcpy(cu_y, y, rows * d * sizeof(coord), hipMemcpyHostToDevice));
  CUDA_CALL(hipMalloc(&cu_F, rows * d * sizeof(coord)));
  CUDA_CALL(hipMemcpy(cu_F, Fattr, rows * d * sizeof(coord),
                       hipMemcpyHostToDevice));
  coord *val_d;
  int *col_ind_d, *block_ptr_d;
  dim3 block_size = 512;
  dim3 grid_size{};
  grid_size.x = (n * blockSize + block_size.x - 1) / block_size.x;

  hipMalloc(&val_d, num_blocks * blockSize * blockSize * sizeof(coord));
  hipMalloc(&block_ptr_d, (n + 1) * sizeof(int));
  hipMalloc(&col_ind_d, num_blocks * sizeof(int));
  hipMemcpy(val_d, val, num_blocks * blockSize * blockSize * sizeof(coord),
             hipMemcpyHostToDevice);
  hipMemcpy(block_ptr_d, block_ptr, (n + 1) * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(col_ind_d, col_ind, num_blocks * sizeof(int),
             hipMemcpyHostToDevice);
  struct timeval t1, t2;
  double elapsedTime;
  gettimeofday(&t1, NULL);
  bsrpq_r<coord, int><<<grid_size, block_size>>>(
      n, blockSize, col_ind_d, block_ptr_d, val_d, cu_y, cu_F, rows, d);
  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);
  elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;    // sec to ms
  elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
  //  printf("BSR row elapsedTime=%lf\n", elapsedTime);
  hipMemcpy(Fattr, cu_F, rows * d * sizeof(coord), hipMemcpyDeviceToHost);

  CUDA_CALL(hipFree(cu_F));
  CUDA_CALL(hipFree(cu_y));
  CUDA_CALL(hipFree(val_d));
  CUDA_CALL(hipFree(col_ind_d));
  CUDA_CALL(hipFree(block_ptr_d));
  return elapsedTime;
}
template <typename data_type, typename index_type>
void transpose_blocks(data_type *new_values, index_type *row_ptr, index_type bs,
                      index_type n_rows, data_type *values) {
  std::unique_ptr<data_type[]> buffer(new data_type[bs * bs]);

  for (index_type row = 0; row < n_rows; row++) {
    for (index_type block = row_ptr[row]; block < row_ptr[row + 1]; block++) {
      data_type *new_block_data = new_values + bs * bs * block;
      data_type *old_block_data = values + bs * bs * block;
      std::copy_n(old_block_data, bs * bs, buffer.get());

      for (unsigned int i = 0; i < bs; i++)
        for (unsigned int j = 0; j < bs; j++)
          new_block_data[j * bs + i] = buffer[i * bs + j];
    }
  }
}
double bsr_colrun(coord *val, int *block_ptr, int *col_ind, coord *y,
                  coord *Fattr, int n, int blockSize, int num_blocks, int rows,
                  int d) {

  coord *cu_y;
  coord *cu_F;
  CUDA_CALL(hipMalloc(&cu_y, rows * d * sizeof(coord)));
  CUDA_CALL(
      hipMemcpy(cu_y, y, rows * d * sizeof(coord), hipMemcpyHostToDevice));
  CUDA_CALL(hipMalloc(&cu_F, rows * d * sizeof(coord)));
  CUDA_CALL(hipMemcpy(cu_F, Fattr, rows * d * sizeof(coord),
                       hipMemcpyHostToDevice));
  coord *val_t =
      (coord *)malloc(sizeof(coord) * num_blocks * blockSize * blockSize);
  transpose_blocks<coord, int>(val_t, block_ptr, blockSize, n, val);

  coord *val_d;
  int *col_ind_d, *block_ptr_d;
  hipMalloc(&val_d, num_blocks * blockSize * blockSize * sizeof(coord));
  hipMalloc(&block_ptr_d, (n + 1) * sizeof(int));
  hipMalloc(&col_ind_d, num_blocks * sizeof(int));
  hipMemcpy(val_d, val_t, num_blocks * blockSize * blockSize * sizeof(coord),
             hipMemcpyHostToDevice);
  hipMemcpy(block_ptr_d, block_ptr, (n + 1) * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(col_ind_d, col_ind, num_blocks * sizeof(int),
             hipMemcpyHostToDevice);

  dim3 block_size = 32;
  dim3 grid_size{};

  grid_size.x = (n * 32 + block_size.x - 1) / block_size.x;
  struct timeval t1, t2;
  double elapsedTime;
  gettimeofday(&t1, NULL);
  switch (blockSize) {
  case 1:
    bsr_col<coord, int, 1>
        <<<grid_size, block_size, block_size.x * d * sizeof(coord)>>>(
            n, col_ind_d, block_ptr_d, val_d, cu_y, cu_F, rows, d);
    break;
  case 2:
    bsr_col<coord, int, 2>
        <<<grid_size, block_size, block_size.x * d * sizeof(coord)>>>(
            n, col_ind_d, block_ptr_d, val_d, cu_y, cu_F, rows, d);
    break;
  case 3:
    bsr_col<coord, int, 3>
        <<<grid_size, block_size, block_size.x * d * sizeof(coord)>>>(
            n, col_ind_d, block_ptr_d, val_d, cu_y, cu_F, rows, d);
    break;
  case 4:
    bsr_col<coord, int, 4>
        <<<grid_size, block_size, block_size.x * d * sizeof(coord)>>>(
            n, col_ind_d, block_ptr_d, val_d, cu_y, cu_F, rows, d);
    break;
  case 8:
    bsr_col<coord, int, 8>
        <<<grid_size, block_size, block_size.x * d * sizeof(coord)>>>(
            n, col_ind_d, block_ptr_d, val_d, cu_y, cu_F, rows, d);
    break;
  case 16:
    bsr_col<coord, int, 16>
        <<<grid_size, block_size, block_size.x * d * sizeof(coord)>>>(
            n, col_ind_d, block_ptr_d, val_d, cu_y, cu_F, rows, d);
    break;
  case 32:
    bsr_col<coord, int, 32>
        <<<grid_size, block_size, block_size.x * d * sizeof(coord)>>>(
            n, col_ind_d, block_ptr_d, val_d, cu_y, cu_F, rows, d);
    break;
  }
  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);
  elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;    // sec to ms
  elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
  //  printf("BSR col elapsedTime=%lf\n",elapsedTime );

  hipMemcpy(Fattr, cu_F, rows * d * sizeof(coord), hipMemcpyDeviceToHost);
  CUDA_CALL(hipFree(cu_F));
  CUDA_CALL(hipFree(cu_y));
  CUDA_CALL(hipFree(val_d));
  CUDA_CALL(hipFree(col_ind_d));
  CUDA_CALL(hipFree(block_ptr_d));
  return elapsedTime;
}
void test_pq(coord *val, int *block_ptr, int *col_ind, coord *y, int n,
             int blockSize, int num_blocks, int rows, int d, coord *Fserial,
             double *timeInfo) {
  coord *Fbsr1 = (coord *)calloc(sizeof(coord), rows * d);
  timeInfo[0] = bsr1run(val, block_ptr, col_ind, y, Fbsr1, n, blockSize,
                        num_blocks, rows, d);

  double maxError = maxerror(Fserial, Fbsr1, rows, d);
  if (maxError > 0.000001) {
    printf("Error Bsr 1\n");
  }
  coord *Fbsr2 = (coord *)calloc(sizeof(coord), rows * d);
  timeInfo[1] = bsr_colrun(val, block_ptr, col_ind, y, Fbsr2, n, blockSize,
                           num_blocks, rows, d);
  maxError = maxerror(Fserial, Fbsr2, rows, d);
  if (maxError > 0.00001) {
    printf("Error Bsr 2\n");
  }

  free(Fbsr1);
  free(Fbsr2);
}
void bsr2csr(coord *bsrValA, int *bsrRowPtrA, int *bsrColIndA, int n, int bs,
             int num_blocks, int rows, coord *csrVal, int *csrRptr,
             int *csrCidx) {
  // Given BSR format (bsrRowPtrA, bsrcolIndA, bsrValA) and
  // blocks of BSR format are stored in column-major order.
  hipsparseDirection_t dir = HIPSPARSE_DIRECTION_ROW;
  int m = n * bs;
  int nnz = num_blocks * bs * bs; // number of elements
  int *csrRowPtrC, *csrColIndC;
  coord *csrValC;
  hipMalloc((void **)&csrRowPtrC, sizeof(int) * (m + 1));
  hipMalloc((void **)&csrColIndC, sizeof(int) * nnz);
  hipMalloc((void **)&csrValC, sizeof(coord) * nnz);
  coord *bsrValA_d;
  int *bsrColIndA_d, *bsrRowPtrA_d;
  hipMalloc(&bsrValA_d, nnz * sizeof(coord));
  hipMalloc(&bsrRowPtrA_d, (n + 1) * sizeof(int));
  hipMalloc(&bsrColIndA_d, num_blocks * sizeof(int));
  hipMemcpy(bsrValA_d, bsrValA, nnz * sizeof(coord), hipMemcpyHostToDevice);
  hipMemcpy(bsrRowPtrA_d, bsrRowPtrA, (n + 1) * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(bsrColIndA_d, bsrColIndA, num_blocks * sizeof(int),
             hipMemcpyHostToDevice);
  hipsparseMatDescr_t descr = 0;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);
  hipsparseDbsr2csr(handle, dir, n, n, descr, bsrValA_d, bsrRowPtrA_d,
                   bsrColIndA_d, bs, descr, csrValC, csrRowPtrC, csrColIndC);
  hipMemcpy(csrVal, csrValC, nnz * sizeof(coord), hipMemcpyDeviceToHost);
  hipMemcpy(csrCidx, csrColIndC, nnz * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(csrRptr, csrRowPtrC, (rows + 1) * sizeof(int),
             hipMemcpyDeviceToHost);

  hipFree(bsrValA_d);
  hipFree(bsrColIndA_d);
  hipFree(bsrRowPtrA_d);
  hipFree(csrRowPtrC);
  hipFree(csrColIndC);
  hipFree(csrValC);
}
void csr2bsr(int blockDim, int n, int m, int nnz, int *csrRowptr,
             int *csrColInd, coord *csrVal, int **bsrRowPtr, int **bsrColInd,
             coord **bsrVal, int* nnzblocks,int* n_block_rows) {

  int *csrRowPtrA, *csrColIndA;
  coord *csrValA;
  hipMalloc((void **)&csrRowPtrA, sizeof(int) * (m + 1));
  hipMalloc((void **)&csrColIndA, sizeof(int) * nnz);
  hipMalloc((void **)&csrValA, sizeof(coord) * nnz);
  hipMemcpy(csrValA, csrVal, nnz * sizeof(coord), hipMemcpyHostToDevice);
  hipMemcpy(csrColIndA, csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(csrRowPtrA, csrRowptr, (m + 1) * sizeof(int),
             hipMemcpyHostToDevice);
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);
  hipsparseMatDescr_t descr = 0;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseDirection_t dir = HIPSPARSE_DIRECTION_ROW;
  int base, nnzb;
  int mb = (m + blockDim - 1) / blockDim;
  hipMalloc(bsrRowPtr, sizeof(int) * (mb + 1));
  // nnzTotalDevHostPtr points to host memory
  int *nnzTotalDevHostPtr = &nnzb;
  hipsparseXcsr2bsrNnz(handle, dir, m, n, descr, csrRowPtrA, csrColIndA,
                      blockDim, descr, *bsrRowPtr, nnzTotalDevHostPtr);
  if (NULL != nnzTotalDevHostPtr) {
    nnzb = *nnzTotalDevHostPtr;
  } else {
    hipMemcpy(&nnzb, *bsrRowPtr + mb, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&base, *bsrRowPtr, sizeof(int), hipMemcpyDeviceToHost);
    nnzb -= base;
  }
  hipMalloc(bsrColInd, sizeof(int) * nnzb);
  hipMalloc(bsrVal, sizeof(coord) * (blockDim * blockDim) * nnzb);
  hipsparseDcsr2bsr(handle, dir, m, n, descr, csrValA, csrRowPtrA, csrColIndA,
                   blockDim, descr, *bsrVal, *bsrRowPtr, *bsrColInd);
  *nnzblocks=nnzb;
  *n_block_rows=mb;
  hipFree(csrRowPtrA);
  hipFree(csrColIndA);
  hipFree(csrValA);
}
int main(int argc, char **argv) {
  struct timeval t1, t2;
  double elapsedTime;
  gettimeofday(&t1, NULL);

  ofstream myfile;
  ofstream myfile2;
  myfile.open("matrix.txt");
  myfile2.open("Y.txt");
  int N = 1 << atoi(argv[1]);
  int K = N;
  int bs = 1 << atoi(argv[2]);
  myfile << "N= " << N << " bs= " << bs << "\n";
  float density = (float)1 / (1 << (atoi(argv[3])));
  int d = atoi(argv[4]);
  int iterations = atoi(argv[5]);
  cout << "N= " << N << " d= " << d << " bs= " << bs << " density= " << density
       << "\n";
  /*Make random bsr matrix N*bs x K*bs */
  coord *weight;
  int *weight_ind;
  int *weight_ptr;
  int nnz = int(density * K * N * bs * bs); // nnz
  int num_blocks = int(nnz / (bs * bs)) + 1;
  printf("num_blocks=%d\n", num_blocks); // will be generated from blocks
  nnz = num_blocks * bs * bs;
  printf("nnz=%d\n", nnz);

  weight = (coord *)malloc(num_blocks * bs * bs * sizeof(coord));
  weight_ind = (int *)malloc(num_blocks * sizeof(int));
  weight_ptr = (int *)malloc((N + 1) * sizeof(int));

  for (int i = 0; i < num_blocks * bs * bs; i++) {
    weight[i] = (coord)10 * get_random();
  }
  generate_candidate_blocks(N, K, bs, bs, num_blocks, weight_ptr, weight_ind);
  int rows = N * bs;
  coord *x = (coord *)malloc(rows * d * sizeof(coord));
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < d; j++) {
      x[i + j * rows] = (coord)10 * get_random();
      myfile2 << x[i + j * rows] << " ";
    }
    myfile2 << "\n";
  }
  gettimeofday(&t2, NULL);
  elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;    // sec to ms
  elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
  printf("Making elapsedTime=%lf\n", elapsedTime);
  // Make coo format

  coord *coov = (coord *)malloc(sizeof(coord) * nnz);
  int *cooi = (int *)malloc(sizeof(int) * nnz);
  int *cooj = (int *)malloc(sizeof(int) * nnz);
  int nnzcntr = 0;
  gettimeofday(&t1, NULL);

  for (int i = 0; i < N; i++) {
    int block_first = weight_ptr[i];
    int block_last = weight_ptr[i + 1];
    for (int block = block_first; block < block_last; block++) {
      for (int row = 0; row < bs; row++) {
        for (int col = 0; col < bs; col++) {

          myfile << i * bs + row << " " << weight_ind[block] * bs + col << " "
                 << weight[block * bs * bs + row * bs + col] << "\n";

          coov[nnzcntr] = weight[block * bs * bs + row * bs + col];
          cooi[nnzcntr] = i * bs + row;
          cooj[nnzcntr] = weight_ind[block] * bs + col;

          nnzcntr++;
          // bsr[ i * bs + row][weight_ind[block] * bs + col]=weight[block * bs
          // * bs + row * bs + col];
        }
      }
    }
  }
  gettimeofday(&t2, NULL);
  elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;    // sec to ms
  elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
  printf("Coo and file elapsedTime=%lf\n", elapsedTime);

  myfile.close();
  myfile2.close();
  coord *Fserial = (coord *)calloc(sizeof(coord), rows * d);

  gettimeofday(&t1, NULL);
  serial(weight, weight_ptr, weight_ind, x, Fserial, N, bs, d, rows);
  gettimeofday(&t2, NULL);
  elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;    // sec to ms
  elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
  printf("serial  elapsedTime=%lf\n", elapsedTime);
  double timeInfo[2];
  double *time_coo = (double *)malloc(iterations * sizeof(double));
  double *time_bsr_row = (double *)malloc(iterations * sizeof(double));
  double *time_bsr_col = (double *)malloc(iterations * sizeof(double));


  coord *csrVal = (coord *)malloc(sizeof(coord) * nnz);
  int *csrCidx = (int *)malloc(sizeof(int) * nnz);
  int *csrRptr = (int *)malloc(sizeof(int) * (rows + 1));
  bsr2csr(weight, weight_ptr, weight_ind, N, bs, num_blocks, rows, csrVal,
          csrRptr, csrCidx);
  coord* bsrValC;
  int* bsrRowPtrC,*bsrColIndC;
  int mb,nnzb;
  csr2bsr( bs, rows, rows,  nnz,csrRptr,csrCidx,csrVal, &bsrRowPtrC, &bsrColIndC, &bsrValC,&nnzb,&mb);

  coord* bsrValh=(coord*)malloc(sizeof(coord)*nnzb*bs*bs );
  int* bsrColh=(int*)malloc(sizeof(int)*nnzb);
  int* bsrRowh=(int* )malloc(sizeof(int)*(mb+1));

  CUDA_CALL(hipMemcpy(bsrRowh,bsrRowPtrC , (mb+1) *sizeof(int),hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(bsrColh, bsrColIndC,  nnzb*sizeof(int),hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(bsrValh, bsrValC,  bs*bs*nnzb*sizeof(coord),hipMemcpyDeviceToHost));

  printf("Index: %d\n",maxerror(bsrColh,weight_ind, nnzb, 1) );
  printf("Index: %lf\n",maxerror(bsrValh,weight, nnzb*bs*bs, 1) );
  printf("Index: %d\n",maxerror(bsrRowh,weight_ptr, N+1, 1) );
  printf("nnzb=%d mb=%d\n",nnzb,mb );
  printf("num_blocks=%d N=%d\n", num_blocks,N); // will be generated from blocks

      //------------------------------------------------------------------------------


        csr_matrix_class<coord> A;
        A.nnz=nnz;
        A.n=rows;
        A.data.reset (new coord[nnz]);
        A.columns.reset (new unsigned int[nnz]);
        A.row_ptr.reset (new unsigned int[rows+1]);
        for(int i=0;i<nnz;i++){
          A.columns[i]=(unsigned int)csrCidx[i];
          A.data[i]=csrVal[i];
        }
        for(int i=0;i<rows+1;i++){
          A.row_ptr[i]=csrRptr[i];
        }
        hybrid_matrix_class<coord> D(A);
        D.allocate(A,0.001);
        coord* hybridy;
        coord* dx;
        CUDA_CALL(hipMalloc(&hybridy, rows*d * sizeof(coord)));
        CUDA_CALL(hipMalloc(&dx, rows*d * sizeof(coord)));
        hipMemcpy(dx, x, rows*d * sizeof (coord),hipMemcpyHostToDevice);

        unsigned int* ell_cols,* coo_col_ids,*coo_row_ids;
        coord* ell_data, *coo_data;
        const size_t A_size = D.ell_matrix->get_matrix_size ();
        const size_t col_ids_size = A_size;
        CUDA_CALL(hipMalloc(&ell_data, A_size * sizeof(coord)));
        CUDA_CALL(hipMalloc(&ell_cols, A_size * sizeof(unsigned int)));
        hipMemcpy(ell_data, D.ell_matrix->data.get(), A_size * sizeof (coord),hipMemcpyHostToDevice);
        hipMemcpy(ell_cols, D.ell_matrix->columns.get(), col_ids_size *sizeof (unsigned int), hipMemcpyHostToDevice);

        const size_t coo_size = D.coo_matrix->get_matrix_size ();
        CUDA_CALL(hipMalloc(&coo_data, coo_size * sizeof(coord)));
        CUDA_CALL(hipMalloc(&coo_col_ids, coo_size * sizeof(unsigned int)));
        CUDA_CALL(hipMalloc(&coo_row_ids, coo_size * sizeof(unsigned int)));

        hipMemcpy (coo_data, D.coo_matrix->data.get(), coo_size * sizeof (coord),hipMemcpyHostToDevice);
        hipMemcpy (coo_col_ids,D.coo_matrix->cols.get(), coo_size * sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy (coo_row_ids, D.coo_matrix->rows.get(),coo_size * sizeof (unsigned int), hipMemcpyHostToDevice);
        double *time_hybrid = (double *)malloc(iterations * sizeof(double));

        for (int i = 0; i < iterations; i++) {
          time_coo[i] = test_coo(coov, cooi, cooj, nnz, x, rows, d, Fserial);
          test_pq(weight, weight_ptr, weight_ind, x, N, bs, num_blocks, rows, d,
                  Fserial, timeInfo);
          gettimeofday(&t1, NULL);

          gpu_hybrid_spmv(D,dx,rows,hybridy,ell_cols,ell_data,coo_data,coo_row_ids,coo_col_ids,d);
          hipDeviceSynchronize();
          gettimeofday(&t2, NULL);
          elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;    // sec to ms
          elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
          time_hybrid[i]=elapsedTime;
          time_bsr_col[i] = timeInfo[1];
          time_bsr_row[i] = timeInfo[0];
        }
        for (int i = 0; i < iterations; i++) {
          printf("%lf ", time_coo[i]);
        }
        printf("\n");

        for (int i = 0; i < iterations; i++) {
          printf("%lf ", time_bsr_row[i]);
        }
        printf("\n");

        for (int i = 0; i < iterations; i++) {
          printf("%lf ", time_bsr_col[i]);
        }

        printf("\n");
        for (int i = 0; i < iterations; i++) {
          printf("%lf ", time_hybrid[i]);
        }

        printf("\n");


        free(time_coo);
        free(time_bsr_col);
        free(time_bsr_row);
        free(time_hybrid);


      //  printf("Hybrid elapsedTime=%lf\n", elapsedTime);

        //coord* result2=(coord *)malloc(sizeof(coord)*rows*d);
        //hipMemcpy(result2, hybridy, rows*d * sizeof (coord),hipMemcpyDeviceToHost);
        //printf("Hybrid error %f\n",maxerror(result2,Fserial , rows,  d) );
        hipFree(ell_data);
        hipFree(coo_data);
        hipFree(ell_cols);
        hipFree(coo_col_ids);
        hipFree(coo_row_ids);
        hipFree(hybridy);
        hipFree(dx);

      //------------------------------------------------------------------------------
      free(weight);
  free(weight_ind);
  free(weight_ptr);
  free(x);
  free(coov);
  free(cooi);
  free(cooj);
  free(Fserial);
  free(csrVal);
  free(csrRptr);
  free(csrCidx);
  hipFree(bsrRowPtrC);
  hipFree(bsrValC);
  hipFree(bsrColIndC);


  return 0;
}
