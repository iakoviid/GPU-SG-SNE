#include "../gridding.cuh"
#include "../gridding.hpp"
#include "../relocateData.cuh"
#include "../relocateData.hpp"
#include "../utils_cuda.cuh"
#include "../Frep.cuh"
#include "../Frep.hpp"
#include "../gradient_descend.hpp"
#include "../gradient_descend.cuh"
#include "../sparsematrix.cuh"
#include "../sparsematrix.hpp"
#include <random>
#include "../graph_rescaling.hpp"

#include <iostream>
#include <stdio.h>
#include <sys/time.h>
using namespace std;
#include "../types.hpp"

template <class dataPoint>
dataPoint maxerror(dataPoint *const w, dataPoint *dv, int n, int d) {

  dataPoint *v = (dataPoint *)malloc(n * d * sizeof(dataPoint));
  hipMemcpy(v, dv, d * n * sizeof(dataPoint), hipMemcpyDeviceToHost);
  dataPoint maxError = 0;
  dataPoint avgError = 0;
  int pos = 0;

  for (int i = 0; i < n ; i++) {
    for(int j=0;j<d;j++){
    if ((v[i+j*n] - w[i*d+j]) * (v[i+j*n] - w[i*d+j]) > maxError) {
      maxError = (v[i+j*n] - w[i*d+j]) * (v[i+j*n] - w[i*d+j]);
      pos = i;
    }
    avgError += (v[i+j*n] - w[i*d+j]) * (v[i+j*n] - w[i*d+j]);
  }}

  printf("maxError=%lf pos=%d v[i]=%lf vs w[i]=%lf avgError=%lf n=%d size=%d\n",
         maxError, 1, v[1], w[1*d], avgError / (n * d), n, n * d);
  free(v);
  return maxError;
}

coord *generateRandomCoord(int n, int d) {

  coord *y = (coord *)malloc(n * d * sizeof(coord));

  for (int i = 0; i < n * d; i++)
    y[i] = ((coord)rand() / (RAND_MAX)) *  .0001;

  return y;
}

template <class dataPoint>
void copydata(dataPoint *const w, dataPoint *dw, int n, int d) {
  dataPoint *v = (dataPoint *)malloc(sizeof(dataPoint) * n * d);
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < d; j++) {

      v[i + n * j] = w[i * d + j];
    }
  }
  hipMemcpy(dw, v, d * n * sizeof(dataPoint), hipMemcpyHostToDevice);
  free(v);
  return;
}
sparse_matrix *generateRandomCSC(int n){

  sparse_matrix *P = (sparse_matrix *) malloc(sizeof(sparse_matrix));

  P->n = n; P->m = n;

  P->col = (matidx *) malloc( (n+1)*sizeof(matidx) );

  for (int j=0 ; j<n ; j++)
    P->col[j] = rand() % 10 + 2;

  int cumsum = 0;
  for(int i = 0; i < P->n; i++){
    int temp = P->col[i];
    P->col[i] = cumsum;
    cumsum += temp;
  }
  P->col[P->n] = cumsum;
  P->nnz = cumsum;

  P->row = (matidx *) malloc( (P->nnz)*sizeof(matidx) );
  P->val = (matval *) malloc( (P->nnz)*sizeof(matval) );

  std::uniform_real_distribution<double> unif(0,1);
  std::default_random_engine re;

  for (int l = 0; l < P->nnz; l++){
    P->row[l] = rand() % n;
    P->val[l] = unif(re);
  }

  return P;

}
/*
void  PrepareSparseMatrix(sparse_matrix P,sparse_matrix Pd,) {


}
*/
//./sg_test 8381 8381 251430  <pbmc-graph.mtx
int main(int argc, char **argv) {
  srand(time(NULL));

  int n = atoi(argv[1]);
  int d = atoi(argv[2]);
  int iterations=atoi(argv[3]);
  int nz=  atoi(argv[4]);
  int format=atoi(argv[5]);
  int N=n;
  int M=n;
  coord *y, *y_d;
  struct timeval t1, t2;
  double elapsedTime;
  CUDA_CALL(hipMallocManaged(&y_d, (d)*n * sizeof(coord)));
  y = generateRandomCoord(n, d);
  copydata(y, y_d, n, d);
  int *I, *J;
  double *val;
  I = (int *)malloc(sizeof(int) * nz);
  J = (int *)malloc(sizeof(int) * nz);
  val = (coord *)malloc(sizeof(coord) * nz);
  for (int i = 0; i < nz; i++) {
    scanf("%d %d %lf\n", &J[i], &I[i], &val[i]);
    I[i]--;
    J[i]--;
  }
  sparse_matrix *P=(sparse_matrix *)malloc(sizeof(sparse_matrix));
  P->val = (double *)calloc(nz, sizeof(double));
  P->row = (int *)calloc(nz, sizeof(int));
  P->col = (int *)calloc(M + 1, sizeof(int));

 for (int i = 0; i < nz; i++) {
   P->val[i] = val[i];
   P->row[i] = J[i];
   P->col[I[i] + 1]++;
 }
 for (int i = 0; i < M; i++) {
   P->col[i + 1] += P->col[i];
 }
 P->n=N;
 P->m=M;
 P->nnz=nz;

 tsneparams params;
 params.d=d;
 params.n=n;
 params.alpha=12;
 params.maxIter=iterations;
 params.earlyIter=iterations/4;
 params.np=1;
 uint32_t nStoch = makeStochastic(*P);
 //lambdaRescaling(*P, params.lambda, false, params.dropLeaf);

symmetrizeMatrix( P );

double sum_P = .0;
for (int i = 0; i < P->nnz; i++) {
  sum_P += P->val[i];
}
for (int i = 0; i < P->nnz; i++) {
  P->val[i] /= sum_P;
}

    switch (params.d){
    case 1:
      params.h = 0.5;
      break;
    case 2:
      params.h = 0.7;
      break;
    case 3:
      params.h = 1.2;
      break;
    }
    matval *vald;
    matidx *cold,*rowd;
    int nnz=P->nnz;
    if(format==0){
    CUDA_CALL(hipMallocManaged(&cold, nnz * sizeof(matidx)));
    CUDA_CALL(hipMallocManaged(&vald,nnz * sizeof(matval)));
    CUDA_CALL(hipMallocManaged(&rowd, (n+1) * sizeof(matidx)));

    hipMemcpy(cold,  P->row,  nnz * sizeof(matidx), hipMemcpyHostToDevice);
    hipMemcpy(vald,  P->val, nnz * sizeof(matval), hipMemcpyHostToDevice);
    hipMemcpy(rowd,  P->col, (n+1) * sizeof(matidx), hipMemcpyHostToDevice);
  }else if(format==2){
    //csr to coo

    CUDA_CALL(hipMallocManaged(&cold, nnz * sizeof(matidx)));
    CUDA_CALL(hipMallocManaged(&vald,nnz * sizeof(matval)));
    CUDA_CALL(hipMallocManaged(&rowd, nnz * sizeof(matidx)));
    matidx* coorow=(matidx *)malloc(sizeof(matidx)*nnz);
    for(int i=0;i<n;i++){
      for(int j=P->col[j];j<P->col[j+1];j++ ){
              coorow[j]=i;
            }
    }
    hipMemcpy(cold,  P->row,  nnz * sizeof(matidx), hipMemcpyHostToDevice);
    hipMemcpy(vald,  P->val, nnz * sizeof(matval), hipMemcpyHostToDevice);
    hipMemcpy(rowd,  coorow, nnz * sizeof(matidx), hipMemcpyHostToDevice);

  }
    sparse_matrix *Pd=(sparse_matrix *) malloc(sizeof(sparse_matrix));
    Pd->val=vald;
    Pd->col=cold;
    Pd->row=rowd;
    Pd->n=n;
    Pd->nnz=nnz;
    Pd->format=format;
  gettimeofday(&t1, NULL);

  kl_minimizationCPU(y, params, *P);
  gettimeofday(&t2, NULL);

  elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;      // sec to ms
  elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0;   // us to ms
  double timecpu=elapsedTime;
  params.d=d;
  params.n=n;
  params.alpha=12;
  params.maxIter=iterations;
  params.earlyIter=iterations/4;
  params.np=1;

  gettimeofday(&t1, NULL);
  kl_minimization(y_d, params, *Pd);
  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);
  elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;      // sec to ms
  elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0;   // us to ms
  double timegpu=elapsedTime;
  printf("Speedup=%lf\n",timecpu/timegpu );
  maxerror(y, y_d, n, d);
  coord *y_copy =
      static_cast<coord *>(malloc(params.n * params.d * sizeof(coord)));

  CUDA_CALL(hipMemcpy(y_copy, y_d, params.n * params.d * sizeof(coord),
                       hipMemcpyDeviceToHost));

  extractEmbeddingTextT(y_copy,params.n,params.d,"gpuEmbedding.txt");
  extractEmbeddingText(y,params.n,params.d,"cpuEmbedding.txt");


  hipFree(y_d);
  free(y);
  free(I);
  free(J);
  free(val);
return 0;
}
