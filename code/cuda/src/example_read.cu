
#include "hipsolver.h"
#include "sparsematrix.hpp"
#include "types.hpp"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "helper/hip/hip_runtime_api.h"
#include "sparse_reorder.cuh"

void csr2bsr(int blockDim, int n, int m, int nnz, int *csrRowptr,
             int *csrColInd, coord *csrVal, int **bsrRowPtr, int **bsrColInd,
             coord **bsrVal, int *nnzblocks, int *n_block_rows,
             hipsparseHandle_t handle) {

  int *csrRowPtrA, *csrColIndA;
  coord *csrValA;
  hipMalloc((void **)&csrRowPtrA, sizeof(int) * (m + 1));
  hipMalloc((void **)&csrColIndA, sizeof(int) * nnz);
  hipMalloc((void **)&csrValA, sizeof(coord) * nnz);
  hipMemcpy(csrValA, csrVal, nnz * sizeof(coord), hipMemcpyHostToDevice);
  hipMemcpy(csrColIndA, csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(csrRowPtrA, csrRowptr, (m + 1) * sizeof(int),
             hipMemcpyHostToDevice);

  hipsparseMatDescr_t descr = 0;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseDirection_t dir = HIPSPARSE_DIRECTION_ROW;
  int base, nnzb;
  int mb = (m + blockDim - 1) / blockDim;
  hipMalloc(bsrRowPtr, sizeof(int) * (mb + 1));
  // nnzTotalDevHostPtr points to host memory
  int *nnzTotalDevHostPtr = &nnzb;
  hipsparseXcsr2bsrNnz(handle, dir, m, n, descr, csrRowPtrA, csrColIndA,
                      blockDim, descr, *bsrRowPtr, nnzTotalDevHostPtr);
  if (NULL != nnzTotalDevHostPtr) {
    nnzb = *nnzTotalDevHostPtr;
  } else {
    hipMemcpy(&nnzb, *bsrRowPtr + mb, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&base, *bsrRowPtr, sizeof(int), hipMemcpyDeviceToHost);
    nnzb -= base;
  }
  hipMalloc(bsrColInd, sizeof(int) * nnzb);
  hipMalloc(bsrVal, sizeof(coord) * (blockDim * blockDim) * nnzb);
  hipsparseDcsr2bsr(handle, dir, m, n, descr, csrValA, csrRowPtrA, csrColIndA,
                   blockDim, descr, *bsrVal, *bsrRowPtr, *bsrColInd);
  *nnzblocks = nnzb;
  *n_block_rows = mb;
  hipFree(csrRowPtrA);
  hipFree(csrColIndA);
  hipFree(csrValA);
}
#define FLAG_BSDB_PERM
int main(int argc, char *argv[]) {
  // int ret_code;
  // MM_typecode matcode;
  // FILE *f;

  hipsolverSpHandle_t handle = NULL;
  hipsparseHandle_t cusparseHandle = NULL; /* used in residual evaluation */
  hipStream_t stream = NULL;
  hipsparseMatDescr_t descrA = NULL;
  checkCudaErrors(hipsolverSpCreate(&handle));
  checkCudaErrors(hipsparseCreate(&cusparseHandle));

  checkCudaErrors(hipStreamCreate(&stream));
  /* bind stream to cusparse and cusolver*/
  checkCudaErrors(hipsolverSpSetStream(handle, stream));
  checkCudaErrors(hipsparseSetStream(cusparseHandle, stream));

  /* configure matrix descriptor*/
  checkCudaErrors(hipsparseCreateMatDescr(&descrA));
  checkCudaErrors(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
  int baseA = 0; /* base index in CSR format */
  if (baseA) {
    checkCudaErrors(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));
  } else {
    checkCudaErrors(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));
  }

  int M, N, nz;
  int *I, *J;
  double *val;
  // ReadMatrix(&M,&N,&nz,&I,&J, &val,argc,argv);
  N = atoi(argv[1]);
  M = atoi(argv[2]);
  nz = atoi(argv[3]);
  int bs = atoi(argv[4]);
  I = (int *)malloc(sizeof(int) * nz);
  J = (int *)malloc(sizeof(int) * nz);
  val = (coord *)malloc(sizeof(coord) * nz);

  for (int i = 0; i < nz; i++) {
    scanf("%d %d %lf\n", &J[i], &I[i], &val[i]);
    I[i]--;
    J[i]--;
  }

  /************************/
  /* now write out matrix */
  /************************/
  double *csr_val = (double *)calloc(nz, sizeof(double));
  int *csr_col = (int *)calloc(nz, sizeof(int));
  int *csr_row = (int *)calloc(M + 1, sizeof(int));

  for (int i = 0; i < nz; i++) {
    csr_val[i] = val[i];
    csr_col[i] = J[i];
    csr_row[I[i] + 1]++;
  }
  for (int i = 0; i < M; i++) {
    csr_row[i + 1] += csr_row[i];
  }
  coord *bsrValC;
  int *bsrRowPtrC, *bsrColIndC;
  int mb, nnzb;
  sparse_matrix P;
  P.n = N;
  P.m = M;
  P.nnz = nz;
  P.val = csr_val;
  P.col = csr_row;
  P.row = csr_col;

  symmetrizeMatrix(&P);
  N = P.n;
  M = P.m;
  nz = P.nnz;
  printf("nnz= %d\n", P.nnz);

  // csr_val=P.val;
  // csr_row=P.col;
  // csr_col=P.row;

  //csr2bsr(bs, N, M, nz, P.col, P.row, P.val, &bsrRowPtrC, &bsrColIndC, &bsrValC,&nnzb, &mb, cusparseHandle);
  //printf("nnzb=%d mb=%d\n", nnzb, mb);
  /*
  sparse_matrix P;
  P.n=N;
  P.m=M;
  P.nnz=nz;
  P.row=csr_row;
  P.col=csr_col;
  P.val=csr_val;

  //symmetrizeMatrix( &P );
  // ~~~~~~~~~~ extracting BSDB permutation
  idx_t *perm = static_cast<idx_t *>( malloc(P.n * sizeof(idx_t)) );
  idx_t *iperm = static_cast<idx_t *>( malloc(P.n * sizeof(idx_t)) );

#ifdef FLAG_BSDB_PERM

  std::cout << "Nested dissection permutation..." << std::flush;
  // idx_t options[METIS_NOPTIONS];
  // METIS_SetDefaultOptions(options);
  // options[METIS_OPTION_NUMBERING] = 0;

  int status = METIS_NodeND( &P.n,
                             reinterpret_cast<idx_t *> (P.row),
                             reinterpret_cast<idx_t *> (P.col),
                             NULL, NULL,
                             perm, iperm );


  permuteMatrix( &P, perm, iperm );


  if( status != METIS_OK ) {
    std::cerr << "METIS error."; exit(1);
  }

  std::cout << "DONE" << std::endl;

#else

  for( int i = 0; i < P.n; i++ ){
    perm[i]  = i;
    iperm[i] = i;
  }

#endif
*/
  int *perm = static_cast<int *>(malloc(N * sizeof(int)));
  double *csr_val_permuted = (double *)calloc(nz, sizeof(double));
  int *csr_col_permuted = (int *)calloc(nz, sizeof(int));
  int *csr_row_permuted = (int *)calloc(M + 1, sizeof(int));

  SparseReorder(argv[5], handle, descrA, M, N, nz, P.col, P.row, P.val,
               csr_row_permuted, csr_col_permuted, csr_val_permuted, perm);
/*
  for (int i = 0; i < N; i++) {
    for (int element = csr_row_permuted[i]; element < csr_row_permuted[i + 1];
         element++) {
      printf("%d %d %lf\n", i, csr_col_permuted[element],
             csr_val_permuted[element]);
    }
  }
  */
  csr2bsr(bs, N, M, nz, csr_row_permuted, csr_col_permuted, csr_val_permuted, &bsrRowPtrC, &bsrColIndC, &bsrValC,&nnzb, &mb, cusparseHandle);
  printf("nnzb=%d mb=%d\n", nnzb, mb);

  /*for(int i=0;i<N;i++){
    printf("%d\n",perm[i] );
  }*/
  if (handle) {
    checkCudaErrors(hipsolverSpDestroy(handle));
  }
  if (cusparseHandle) {
    checkCudaErrors(hipsparseDestroy(cusparseHandle));
  }
  if (stream) {
    checkCudaErrors(hipStreamDestroy(stream));
  }
  if (descrA) {
    checkCudaErrors(hipsparseDestroyMatDescr(descrA));
  }
  return 0;
}
