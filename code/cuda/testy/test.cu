#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <limits>
#include <math.h>
#include <numeric>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#define LIMIT_SEQ 512
void conv1dnopad(double *const PhiGrid, const double *const VGrid,
                 const double h, uint32_t *const nGridDims, const uint32_t nVec,
                 const uint32_t nDim, const uint32_t nProc);

#define CUDA_CALL(x)                                                           \
  {                                                                            \
    if ((x) != hipSuccess) {                                                  \
      printf("CUDA error at %s:%d\n", __FILE__, __LINE__);                     \
      printf("  %s\n", hipGetErrorString(hipGetLastError()));                \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

double *generateRandomCoord(int n, int d) {

  double *y = (double *)malloc(n * d * sizeof(double));
  srand(time(0));

  for (int i = 0; i < n * d; i++)
    y[i] = ((double)rand() / (RAND_MAX)) * 100;

  return y;
}
__global__ void generateBoxIdx(uint64_t *Code, const double *Y, double scale,
                               const int nPts, const int nDim, const int nGrid,
                               const double multQuant, const uint32_t qLevel) {
  register uint64_t C[3];
  register double Yscale;
  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {
    for (int j = 0; j < nDim; j++) {
      Yscale = Y[TID + j * nPts] / scale;
      if (Yscale >= 1) {
        Yscale = 1 - 0.00000000000001;
        // printf("Yscale= %lf\n",Yscale );
      }
      C[j] = (uint64_t)abs(floor(multQuant * Yscale));
    }
    switch (nDim) {

    case 1:
      Code[TID] = (uint64_t)C[0];

    case 2:
      Code[TID] = (((uint64_t)C[1]) << qLevel) | (((uint64_t)C[0]));

    case 3:
      Code[TID] = (((uint64_t)C[2]) << 2 * qLevel) |
                  (((uint64_t)C[1]) << qLevel) | ((uint64_t)C[0]);
    }
  }
  return;
}

uint64_t tangleCode(const double *const YScat, const double scale,
                    const double multQuant, const uint32_t nGrid,
                    const uint32_t nDim) {

  uint32_t qLevel = ceil(log(nGrid) / log(2));

  uint64_t C[3];

  for (uint32_t j = 0; j < nDim; j++) {

    // get scaled input
    double Yscale = YScat[j] / scale;
    if (Yscale >= 1) {
      Yscale = 1 - std::numeric_limits<double>::epsilon();
      // printf("Yscale= %lf\n",Yscale );
    }

    // scale data points
    C[j] = (uint32_t)abs(floor(multQuant * Yscale));
  }

  switch (nDim) {

  case 1:
    return (uint64_t)C[0];

  case 2:
    return (((uint64_t)C[1]) << qLevel) | (((uint64_t)C[0]));

  case 3:
    return (((uint64_t)C[2]) << 2 * qLevel) | (((uint64_t)C[1]) << qLevel) |
           (((uint64_t)C[0]));

  default:
    return 0;
  }
}
void quantizeAndComputeCodes(uint64_t *const C, const double *const YScat,
                             const double scale, const uint32_t nPts,
                             const uint32_t nDim, const uint32_t nGrid) {

  // get quantization multiplier
  double multQuant = nGrid - 1 - std::numeric_limits<double>::epsilon();

  // add codes and ID to struct to sort them
  for (int i = 0; i < nPts; i++) {
    C[i] = tangleCode(&YScat[i * nDim], scale, multQuant, nGrid, nDim);
  }
}

template <typename dataval>
void doSort(uint64_t *const Cs, uint64_t *const Ct, uint32_t *const Ps,
            uint32_t *const Pt, dataval *const Ys, dataval *const Yt,
            uint32_t prev_off, const uint32_t nbits, const uint32_t sft,
            const uint32_t n, const uint32_t d, uint32_t nb) {

  // prepare bins
  uint32_t nBin = (0x01 << (nbits));
  // uint32_t *BinCursor  = new uint32_t[ nBin ]();
  uint32_t *BinCursor = (uint32_t *)calloc(nBin, sizeof(uint32_t));

  // current code
  uint32_t *code = new uint32_t[d]();

  // get mask for required number of bits
  uint64_t mask = (0x01 << (nbits)) - 1;

  for (int i = 0; i < n; i++) {
    uint32_t const ii = (Cs[i] >> sft) & mask;
    BinCursor[ii]++;
  }

  // scan prefix (can be better!)
  int offset = 0;
  for (int i = 0; i < nBin; i++) {
    int const ss = BinCursor[i];
    BinCursor[i] = offset;
    offset += ss;
  }

  // permute points
  for (int i = 0; i < n; i++) {
    uint32_t const ii = (Cs[i] >> sft) & mask;
    Ct[BinCursor[ii]] = Cs[i];
    for (int kapa = 0; kapa < d; kapa++) {
      Yt[BinCursor[ii] * d + kapa] = Ys[i * d + kapa];
    }
    Pt[BinCursor[ii]] = Ps[i];
    BinCursor[ii]++;
  }

  if (sft >= nbits) {

    offset = 0;
    for (int i = 0; i < nBin; i++) {
      uint32_t nPts = BinCursor[i] - offset;

      if (nPts > LIMIT_SEQ) {
        doSort(&Ct[offset], &Cs[offset], &Pt[offset], &Ps[offset],
               &Yt[offset * d], &Ys[offset * d], prev_off + offset, nbits,
               sft - nbits, nPts, d, nb);
      } else if (nPts > 0) {
        doSort(&Ct[offset], &Cs[offset], &Pt[offset], &Ps[offset],
               &Yt[offset * d], &Ys[offset * d], prev_off + offset, nbits,
               sft - nbits, nPts, d, nb);
      }
      offset = BinCursor[i];
    }
  }

  ;

  // delete BinCursor;
  free(BinCursor);
  delete code;
}

template <typename dataval>
void doSort_top(uint64_t *const Cs, uint64_t *const Ct, uint32_t *const Ps,
                uint32_t *const Pt, dataval *const Ys, dataval *const Yt,
                uint32_t prev_off, const uint32_t nbits, const uint32_t sft,
                const uint32_t n, const uint32_t d, uint32_t nb, uint32_t np) {

  // prepare bins
  uint32_t nBin = (0x01 << (nbits));

  // retrive active block per thread
  int m = (int)std::ceil((float)n / (float)np);

  uint32_t *BinCursor = (uint32_t *)calloc(nBin * np, sizeof(uint32_t));

  // current code
  uint32_t *code = new uint32_t[d]();

  // get mask for required number of bits
  uint64_t mask = (0x01 << (nbits)) - 1;

  for (int i = 0; i < np; i++) {
    int size = ((i + 1) * m < n) ? m : (n - i * m);
    for (int j = 0; j < size; j++) {
      uint32_t const ii = (Cs[i * m + j] >> sft) & mask;
      BinCursor[i * nBin + ii]++;
    }
  }

  int offset = 0;
  for (int i = 0; i < nBin; i++) {
    for (int j = 0; j < np; j++) {
      int const ss = BinCursor[j * nBin + i];
      BinCursor[j * nBin + i] = offset;
      offset += ss;
    }
  }

  // permute points
  for (int j = 0; j < np; j++) {
    int size = ((j + 1) * m < n) ? m : (n - j * m);
    for (int i = 0; i < size; i++) {
      uint32_t const idx = j * m + i;
      uint32_t const ii = (Cs[idx] >> sft) & mask;
      uint32_t const jj = BinCursor[j * nBin + ii];
      Ct[jj] = Cs[idx];
      for (int kapa = 0; kapa < d; kapa++) {
        Yt[jj * d + kapa] = Ys[idx * d + kapa];
      }
      Pt[jj] = Ps[idx];
      BinCursor[j * nBin + ii]++;
    }
  }

  if (sft >= nbits) {

    offset = 0;
    for (int i = 0; i < nBin; i++) {
      uint32_t nPts = BinCursor[(np - 1) * nBin + i] - offset;

      if (nPts > LIMIT_SEQ) {
        doSort(&Ct[offset], &Cs[offset], &Pt[offset], &Ps[offset],
               &Yt[offset * d], &Ys[offset * d], prev_off + offset, nbits,
               sft - nbits, nPts, d, nb);
      } else if (nPts > 0) {
        doSort(&Ct[offset], &Cs[offset], &Pt[offset], &Ps[offset],
               &Yt[offset * d], &Ys[offset * d], prev_off + offset, nbits,
               sft - nbits, nPts, d, nb);
      }
      offset = BinCursor[(np - 1) * nBin + i];
    }
  }

  // delete BinCursor;
  free(BinCursor);
  delete code;
}
__inline__ uint32_t untangleLastDim(const uint64_t C, const uint32_t nDim,
                                    const uint32_t qLevel) {

  uint32_t Cout = 0;

  switch (nDim) {

  case 1:
    Cout = (uint32_t)C;
    break;

  case 2: {
    uint64_t mask = (1 << 2 * qLevel) - 1;

    Cout = (uint32_t)((C & mask) >> qLevel);
    break;
  }

  case 3: {
    uint64_t mask = (1 << 3 * qLevel) - 1;

    Cout = (uint32_t)((C & mask) >> 2 * qLevel);
    break;
  }

  default: {
    std::cerr << "Supporting up to 3D" << std::endl;
    exit(1);
  }
  }

  return Cout;
}

void gridSizeAndIdx(uint32_t *const ib, uint32_t *const cb,
                    uint64_t const *const C, const uint32_t nPts,
                    const uint32_t nDim, const uint32_t nGridDim) {

  uint32_t qLevel = ceil(log(nGridDim) / log(2));
  uint32_t idxCur = -1;
  // printf("-----------------------Punch it mr sulu---------------------\n" );
  for (uint32_t i = 0; i < nPts; i++) {

    uint32_t idxNew = untangleLastDim(C[i], nDim, qLevel);
    // if(i<10){printf("idxNew=%d\n",idxNew );}
    cb[idxNew]++;

    if (idxNew != idxCur)
      ib[idxNew + 1] = i + 1;
  }
}
__inline__ __device__ uint32_t untangleLastDimDevice(int nDim, int TID,
                                                     uint32_t qLevel,
                                                     uint64_t *C) {
  uint64_t mask;
  switch (nDim) {
  case 1:
    return (uint32_t)C[TID];
    break;

  case 2: {
    mask = (1 << 2 * qLevel) - 1;

    return (uint32_t)((C[TID] & mask) >> qLevel);
    break;
  }

  case 3: {
    mask = (1 << 3 * qLevel) - 1;

    return (uint32_t)((C[TID] & mask) >> 2 * qLevel);
    break;
  }
  }
}
// Concern about point 0
__global__ void gridSizeAndIdxKernel(uint32_t *ib, uint32_t *cb, uint64_t *C,
                                     int nPts, int nDim, int nGrid,
                                     uint32_t qLevel) {
  uint32_t idxCur;
  uint32_t idxNew;
  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {

    if (TID < nPts - 1) {
      idxNew = untangleLastDimDevice(nDim, TID, qLevel, C);
      idxCur = untangleLastDimDevice(nDim, TID + 1, qLevel, C);
      if (idxNew != idxCur) {
        ib[idxCur] = TID + 1;
      }
      if (idxCur - idxNew > 1) {
        ib[idxNew + 1] = TID + 1;
      }
    } else {
      idxNew = untangleLastDimDevice(nDim, TID, qLevel, C);
      if (idxNew != idxCur)
        ib[idxNew + 1] = TID + 1;
    }
  }
  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x;
       TID < nGrid - 1; TID += gridDim.x * blockDim.x) {
    idxCur = ib[TID];
    idxNew = ib[TID + 1];
    cb[TID] = idxNew - idxCur;
  }
}

__global__ void ComputeChargesKernel(double *__restrict__ VScat,
                                     const double *const y_d, const int n,
                                     const int d, const int n_terms) {

  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < n;
       TID += gridDim.x * blockDim.x) {
    for (int j = 0; j < d; j++) {
      VScat[TID + (j + 1) * n] = y_d[TID + (j)*n];
      // if(threadIdx.x==0){printf("y_d[%d]=%lf\n",TID+(j)*n ,y_d[TID+(j)*n]);}
    }
    VScat[TID] = 1;
  }
}
void ComputeCharges(double *VScat, double *y_d, int n, int d) {
  int threads = 1024;
  int Blocks = 64;
  ComputeChargesKernel<<<Blocks, threads>>>(VScat, y_d, n, d, d + 1);
}
__global__ void compute_repulsive_forces_kernel(
    volatile double *__restrict__ frep, const double *const Y,
    const int num_points, const int nDim, const double *const Phi,
    volatile double *__restrict__ zetaVec, uint32_t *iPerm) {
  register double Ysq = 0;
  register double z = 0;

  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x;
       TID < num_points; TID += gridDim.x * blockDim.x) {

    for (uint32_t j = 0; j < nDim; j++) {
      Ysq += Y[TID + j * num_points] * Y[TID + j * num_points];
      z -= 2 * Y[TID + j * num_points] * Phi[TID * (nDim + 1) + j + 1];
    }
    z += (1 + 2 * Ysq) * Phi[TID * (nDim + 1)];
    zetaVec[TID] = z;
    for (uint32_t j = 0; j < nDim; j++) {
      frep[iPerm[TID] + j * num_points] =
          Y[TID + j * num_points] * Phi[TID * (nDim + 1)] -
          Phi[TID * (nDim + 1) + j + 1];
    }
  }
}
double zetaAndForce(double *Ft_d, double *y_d, int n, int d, double *Phi,
                    thrust::device_vector<uint32_t> &iPerm,
                    thrust::device_vector<double> &zetaVec) {

  int threads = 1024;
  int Blocks = 64;
  compute_repulsive_forces_kernel<<<Blocks, threads>>>(
      Ft_d, y_d, n, d, Phi, thrust::raw_pointer_cast(zetaVec.data()),
      thrust::raw_pointer_cast(iPerm.data()));
  double z = thrust::reduce(zetaVec.begin(), zetaVec.end()) - n;
  return z;
}
template <typename dataval>
dataval zetaAndForce2(dataval *const F,            // Forces
                      const dataval *const Y,      // Coordinates
                      const dataval *const Phi,    // Values
                      const uint32_t *const iPerm, // Permutation
                      const uint32_t nPts,         // #points
                      const uint32_t nDim) {       // #dimensions

  dataval Z = 0;

  // compute normalization term
  for (uint32_t i = 0; i < nPts; i++) {
    dataval Ysq = 0;
    for (uint32_t j = 0; j < nDim; j++) {
      Ysq += Y[i * nDim + j] * Y[i * nDim + j];
      Z -= 2 * (Y[i * nDim + j] * Phi[i * (nDim + 1) + j + 1]);
    }
    Z += (1 + 2 * Ysq) * Phi[i * (nDim + 1)];
  }

  Z = Z - nPts;

  // Compute repulsive forces
  for (uint32_t i = 0; i < nPts; i++) {
    for (uint32_t j = 0; j < nDim; j++)
      F[iPerm[i] * nDim + j] = (Y[i * nDim + j] * Phi[i * (nDim + 1)] -
                                Phi[i * (nDim + 1) + j + 1]) /
                               Z;
  }

  return Z;
}

__global__ void s2g1dCuda(double *V, double *y, double *q, uint32_t ng,
                          uint32_t nPts, uint32_t nDim, uint32_t nVec,
                          double maxy);

void conv1dnopadcuda(double *PhiGrid_d, double *VGrid_d, double h, int nGridDim,
                     int m, int d);

void s2g1d(double *V, double *y, double *q, uint32_t ng, uint32_t np,
           uint32_t nPts, uint32_t nDim, uint32_t nVec);

void s2g1drb(double *V, double *y, double *q, uint32_t *ib, uint32_t *cb,
             uint32_t ng, uint32_t np, uint32_t nPts, uint32_t nDim,
             uint32_t nVec);
__global__ void g2s1dCuda(double *Phi, double *V, double *y, uint32_t ng,
                          uint32_t nPts, uint32_t nDim, uint32_t nVec);

void g2s1d(double *Phi, double *V, double *y, uint32_t ng, uint32_t nPts,
           uint32_t nDim, uint32_t nVec);

void nuconv(double *PhiScat, double *y, double *VScat, uint32_t *ib,
            uint32_t *cb, int n, int d, int m, int np, int nGridDim,
            double *y_d, uint32_t *id_d, uint32_t *cb_d, double *VScat_d,
            double *Phi_d) {

  // ~~~~~~~~~~ normalize coordinates (inside bins)
  double maxy = 0;
  int Blocks = 64;
  int threads = 1024;
  for (int i = 0; i < n * d; i++)
    maxy = maxy < y[i] ? y[i] : maxy;
  for (int i = 0; i < n * d; i++) {
    y[i] /= maxy;
  }

  thrust::device_ptr<double> yVec_ptr(y_d);
  thrust::device_vector<double> yVec_d(yVec_ptr, yVec_ptr + n * d);
  thrust::device_vector<double>::iterator iter =
      thrust::max_element(yVec_d.begin(), yVec_d.end());
  unsigned int position = iter - yVec_d.begin();
  maxy = yVec_d[position];
  // ~~~~~~~~~~ scale them from 0 to ng-1

  for (int i = 0; i < n * d; i++) {
    if (y[i] == 1) {
      y[i] = y[i] - std::numeric_limits<double>::epsilon();
    }
    y[i] *= (nGridDim - 1);
  }

  for (int i = 0; i < n * d; i++)
    if ((y[i] >= nGridDim - 1) || (y[i] < 0))
      exit(1);

  // ~~~~~~~~~~ find exact h

  double h = maxy / (nGridDim - 1 - std::numeric_limits<double>::epsilon());

  // ~~~~~~~~~~ scat2grid
  int szV = pow(nGridDim + 2, d) * m;
  // printf("m=%d d=%d\n",m,d );
  double *VGrid = static_cast<double *>(calloc(szV * np, sizeof(double)));
  double *VGrid_d;
  CUDA_CALL(hipMallocManaged(&VGrid_d,
                              pow(nGridDim + 2, d) * (d + 1) * sizeof(double)));
  double *V = (double *)(calloc(szV * np, sizeof(double)));
  double *VScat2 = (double *)malloc(n * (d + 1) * sizeof(double));

  CUDA_CALL(hipMemcpy(VScat2, VScat_d, n * (d + 1) * sizeof(double),
                       hipMemcpyDeviceToHost));
  printf("====================================================================="
         "================\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < d + 1; j++) {
      if (abs(VScat[i * (d + 1) + j] - VScat2[i + j * n]) < 0.01) {
        // printf(" Succes VScat= %lf
        // VScat_d=%lf\n",VScat[i*(d+1)+j],VScat2[i+j*n] );
      } else {
        printf(" Error VScat= %lf VScat_d=%lf\n", VScat[i * (d + 1) + j],
               VScat2[i + j * n]);
      }
    }
  }
  printf("====================================================================="
         "==============\n");
  int tpoints = pow(nGridDim + 2, d);

  if (d == 1) {
    s2g1d(VGrid, y, VScat, nGridDim + 2, np, n, d, m);
    s2g1dCuda<<<1, 1>>>(VGrid_d, y_d, VScat_d, nGridDim + 2, n, d, m, maxy);
    // Differnt strategy every point must take 1 f1

    CUDA_CALL(
        hipMemcpy(V, VGrid_d, szV * sizeof(double), hipMemcpyDeviceToHost));

    for (int i = 0; i < pow(nGridDim + 2, d); i++) {
      for (int j = 0; j < m; j++) {
        if (abs(VGrid[i + j * tpoints] - V[i + j * tpoints]) < 0.00001) {
          // printf("Succes V1=%lf vs V=%lf \n",VGrid[i+j*tpoints] ,
          // V[i+j*tpoints]);
        } else {
          printf("Error V1=%lf vs V=%lf \n", VGrid[i + j * tpoints],
                 V[i + j * tpoints]);
        }
      }
    }

    printf("==================================================================="
           "================\n");
  }

  double *PhiGrid = static_cast<double *>(calloc(szV, sizeof(double)));
  double *PhiGrid_d;
  CUDA_CALL(hipMallocManaged(&PhiGrid_d, szV * sizeof(double)));

  uint32_t *const nGridDims = new uint32_t[d]();
  for (int i = 0; i < d; i++) {
    nGridDims[i] = nGridDim + 2;
  }
  if (d == 1) {
    conv1dnopad(PhiGrid, VGrid, h, nGridDims, m, d, 1);
    conv1dnopadcuda(PhiGrid_d, VGrid_d, h, nGridDim + 2, m, d);
    double *PhiGrid_dtest = (double *)malloc(sizeof(double) * szV);

    CUDA_CALL(hipMemcpy(PhiGrid_dtest, PhiGrid_d, szV * sizeof(double),
                         hipMemcpyDeviceToHost));
    printf("==================================================================="
           "=\n");
    for (int i = 0; i < szV; i++) {
      printf("Vhost=%lf Vcuda=%lf\n", PhiGrid[i], PhiGrid_dtest[i]);
    }
    printf("==================================================================="
           "=\n");
    g2s1d(PhiScat, PhiGrid, y, nGridDim + 2, n, d, m);
    printf("======================================================\n");
    g2s1dCuda<<<32, 256>>>(Phi_d, PhiGrid_d, y_d, nGridDim + 2, n, d, m);
    double *phitest = (double *)malloc(m * n * sizeof(double));
    CUDA_CALL(hipMemcpy(phitest, Phi_d, n * (d + 1) * sizeof(double),
                         hipMemcpyDeviceToHost));
    for (int i = 0; i < n * m; i++) {
      // printf("Phih=%lf vs phicuda=%lf\n",PhiScat[i],phitest[i] );
    }
  }
}

__global__ void toZero(double *y, int j, double miny, int n) {

  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < n;
       TID += gridDim.x * blockDim.x) {
    y[TID + j * n] = y[TID + j * n] - miny;
  }
}

int main(int argc, char **argv) {
  int d = atoi(argv[1]);
  int N = 1 << atoi(argv[2]);
  int threads = 1 << atoi(argv[3]);
  int blocks = 1 << atoi(argv[4]);
  int nGrid = atoi(argv[5]);

  double *y = generateRandomCoord(N, d);
  uint32_t *iPermpa = (uint32_t *)malloc(sizeof(uint32_t) * 1 * N);

  y[1] = 100;
  y[2] = 100;
  y[3] = 100;
  double *yc = (double *)malloc(N * d * sizeof(double));
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < d; j++) {
      yc[i + N * j] = y[i * d + j];
      // printf("%lf  ",y[i*d+j] );
    }
    // printf("\n" );
  }
  double *y_d;
  int n = N;

  CUDA_CALL(hipMallocManaged(&y_d, d * n * sizeof(double)));
  CUDA_CALL(
      hipMemcpy(y_d, yc, n * d * sizeof(double), hipMemcpyHostToDevice));

  thrust::device_ptr<double> yVec_ptr(y_d);
  thrust::device_vector<double> yVec_d(yVec_ptr, yVec_ptr + n * d);
  unsigned int position;
  double *miny = (double *)malloc(sizeof(double) * d);
  for (int j = 0; j < d; j++) {

    thrust::device_vector<double>::iterator iter = thrust::min_element(
        yVec_d.begin() + j * n, yVec_d.begin() + n * (j + 1));

    position = iter - (yVec_d.begin());

    miny[j] = yVec_d[position];
    // thrust::for_each(vec.begin()+j*n,vec.begin()+(j+1)*n , _1 -= miny[j]);
    toZero<<<blocks, threads>>>(y_d, j, miny[j], n);
    // printf("%lf\n",miny[j] );
  }
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < d; j++) {
      y[i * d + j] -= miny[j];
    }
  }

  thrust::device_vector<double>::iterator iter =
      thrust::max_element(yVec_d.begin(), yVec_d.end());
  position = iter - yVec_d.begin();
  double maxy = yVec_d[position];

  uint64_t *Codes;
  CUDA_CALL(hipMallocManaged(&Codes, n * sizeof(uint64_t)));
  double multQuant = nGrid - 1 - std::numeric_limits<double>::epsilon();

  uint32_t qLevel = 0;
  qLevel = ceil(log(nGrid) / log(2));
  generateBoxIdx<<<blocks, threads>>>(Codes, y_d, 100, N, d, nGrid, multQuant,
                                      qLevel);
  uint64_t *Codes1;
  Codes1 = (uint64_t *)malloc(sizeof(uint64_t) * n);
  CUDA_CALL(
      hipMemcpy(Codes1, Codes, sizeof(uint64_t) * n, hipMemcpyDeviceToHost));

  uint64_t *Codes2;
  Codes2 = (uint64_t *)malloc(sizeof(uint64_t) * n);
  quantizeAndComputeCodes(Codes2, y, 100, n, d, nGrid);
  // printf("----------------------------------------\n" );
  for (int i = 0; i < n; i++) {
    if (Codes1[i] != Codes2[i]) {
      // printf("------------Error i=%d----------\n",i );}
    }

    hipDeviceSynchronize(); // why

    thrust ::device_ptr<uint64_t> Codes_ptr(Codes);
    thrust::device_vector<uint32_t> iPerm(n);
    thrust::sequence(iPerm.begin(), iPerm.end());

    switch (d) {

    case 1:
      thrust ::stable_sort_by_key(
          Codes_ptr, Codes_ptr + n,
          make_zip_iterator(make_tuple(yVec_ptr, iPerm.begin())));

    case 2:
      thrust ::stable_sort_by_key(
          Codes_ptr, Codes_ptr + n,
          make_zip_iterator(make_tuple(yVec_ptr, yVec_ptr + n, iPerm.begin())));

    case 3:
      thrust ::stable_sort_by_key(
          Codes_ptr, Codes_ptr + n,
          make_zip_iterator(make_tuple(yVec_ptr, yVec_ptr + n, yVec_ptr + 2 * n,
                                       iPerm.begin())));
    }
    CUDA_CALL(hipMemcpy(Codes1, Codes, sizeof(uint64_t) * n,
                         hipMemcpyDeviceToHost));
    CUDA_CALL(
        hipMemcpy(yc, y_d, sizeof(double) * d * n, hipMemcpyDeviceToHost));
    uint32_t *perm = (uint32_t *)malloc(n * sizeof(uint32_t));
    CUDA_CALL(hipMemcpy(perm, thrust::raw_pointer_cast(iPerm.data()),
                         sizeof(uint32_t) * n, hipMemcpyDeviceToHost));
    /*
      for(int i=0;i<N;i++){
        printf("%ld ",Codes1[i] );
        for(int j=0;j<d;j++){

          printf("%lf  ",yc[i+N*j] );
        }
        printf(" --> %d ",perm[i] );
        printf("\n" );
      }*/
    uint64_t *const C2 = (uint64_t *)malloc(sizeof(uint64_t) * 1 * n);
    double *const Y2 = (double *)malloc(sizeof(double) * d * n);
    uint32_t *iPerm2 = (uint32_t *)malloc(sizeof(uint32_t) * 1 * n);
    for (int i = 0; i < n; i++) {
      iPermpa[i] = i;
    }
    doSort_top(Codes2, C2, iPermpa, iPerm2, y, Y2, 0, qLevel, (d - 1) * qLevel,
               n, d, nGrid, 1);

    // printf("----------------------------------------\n" );
    /*
      for(int i=0;i<n;i++){
        printf("C1=%ld C2=%ld  Y=",Codes2[i],C2[i]);
        for(int j=0;j<d;j++){

          printf("%lf  ",y[i*d+j] );
        }
        printf("Y2= " );
        for(int j=0;j<d;j++){

          printf("%lf  ",Y2[i*d+j] );
        }
        printf("\n" );
      }
    */
    uint32_t *ibh; // Starting index of box (along last dimension)
    uint32_t *cbh; // Number of scattered points per box (along last dimension)
    ibh = (uint32_t *)calloc(nGrid, sizeof(uint32_t));
    cbh = (uint32_t *)calloc(nGrid, sizeof(uint32_t));
    uint32_t *ib;
    uint32_t *cb;
    CUDA_CALL(hipMallocManaged(&ib, nGrid * sizeof(uint32_t)));
    CUDA_CALL(
        hipMemcpy(ib, ibh, nGrid * sizeof(uint32_t), hipMemcpyHostToDevice));
    CUDA_CALL(hipMallocManaged(&cb, nGrid * sizeof(uint32_t)));
    CUDA_CALL(
        hipMemcpy(cb, cbh, nGrid * sizeof(uint32_t), hipMemcpyHostToDevice));
    if ((d % 2) == 1) {

      // ========== get starting index and size of each grid box
      gridSizeAndIdx(ibh, cbh, C2, n, d, nGrid);
      y = Y2;
      iPermpa = iPerm2;

    } else {

      // ========== get starting index and size of each grid box
      gridSizeAndIdx(ibh, cbh, Codes2, n, d, nGrid);
    }

    // printf("------------Please god help me-------\n" );

    for (int i = 0; i < n; i++) {
      if (iPermpa[i] != perm[i]) {
        printf("Error %d   vs %d\n", iPermpa[i], perm[i]);
      }
    }

    gridSizeAndIdxKernel<<<blocks, threads>>>(ib, cb, Codes, n, d, nGrid,
                                              qLevel);
    uint32_t *ib2 = (uint32_t *)calloc(nGrid, sizeof(uint32_t));

    CUDA_CALL(
        hipMemcpy(ib2, ib, nGrid * sizeof(uint32_t), hipMemcpyDeviceToHost));
    for (int i = 0; i < nGrid; i++) {
      if (ib2[i] != ibh[i]) {
        printf("Error ib=%d ibh=%d \n", ib2[i], ibh[i]);
      } else {
        printf("Succes ib=%d ibh=%d \n", ib2[i], ibh[i]);
      }
    }
    uint32_t *cb2 = (uint32_t *)calloc(nGrid, sizeof(uint32_t));
    CUDA_CALL(
        hipMemcpy(cb2, cb, nGrid * sizeof(uint32_t), hipMemcpyDeviceToHost));

    for (int i = 0; i < nGrid; i++) {
      if (cbh[i] != cb2[i]) {
        printf("Error cbh=%d cb=%d\n", cbh[i], cb2[i]);
      } else {
        printf("hipSuccess cbh=%d cb=%d\n", cbh[i], cb2[i]);
      }
    }

    double *VScat = (double *)malloc(n * (d + 1) * sizeof(double));
    double *VScat_d;
    CUDA_CALL(hipMallocManaged(&VScat_d, n * (d + 1) * sizeof(double)));
    ComputeCharges(VScat_d, y_d, n, d);
    double *VScat2 = (double *)malloc(n * (d + 1) * sizeof(double));

    CUDA_CALL(hipMemcpy(VScat2, VScat_d, n * (d + 1) * sizeof(double),
                         hipMemcpyDeviceToHost));

    for (int i = 0; i < n; i++) {

      VScat[i * (d + 1)] = 1.0;
      for (int j = 0; j < d; j++)
        VScat[i * (d + 1) + j + 1] = y[i * d + j];
    }

    for (int i = 0; i < n; i++) {
      for (int j = 0; j < d + 1; j++) {
        if (VScat[i * (d + 1) + j] == VScat2[i + j * n]) {
          // printf(" Succes VScat= %lf
          // VScat_d=%lf\n",VScat[i*(d+1)+j],VScat2[i+j*n] );
        } else {
          printf(" Error VScat= %lf VScat_d=%lf\n", VScat[i * (d + 1) + j],
                 VScat2[i + j * n]);
        }
      }
    }

    for (int i = 0; i < n; i++) {
      for (int j = 0; j < d; j++)
        if (y[i * (d) + j] == yc[i + j * n]) {
          //   printf(" Succes y= %lf yc=%lf\n",y[i*(d)+j],yc[i+j*n] );
        } else {
          printf(" Error y= %lf yc=%lf\n", y[i * (d) + j], yc[i + j * n]);
        }
    }

    double *Phi = generateRandomCoord(n, d + 1);
    double *Phi_d;
    CUDA_CALL(hipMallocManaged(&Phi_d, n * (d + 1) * sizeof(double)));
    nuconv(Phi, y, VScat, ibh, cbh, n, d, d + 1, 1, nGrid, y_d, ib, cb, VScat_d,
           Phi_d);

    // thrust::device_vector<double> zetaVec(n);
    // double* Ft=(double *)malloc(d*n*sizeof(double));
    // double* Ft_d;
    // CUDA_CALL(hipMallocManaged(&Ft_d,n*d * sizeof( double )));
    // CUDA_CALL(hipMemcpy(Phi_d,Phi,n*(d+1) * sizeof( double ),
    // hipMemcpyHostToDevice));

    /*
      double z= zetaAndForce( Ft_d, y_d, n, d, Phi_d,iPerm,zetaVec);
      double z2=zetaAndForce2(Ft,y,Phi,iPermpa,n,d);
      printf("%lf vs %lf \n",z,z2 );
      double* Ft2=(double *)malloc(d*n*sizeof(double));
      CUDA_CALL(hipMemcpy(Ft2,Ft_d,n*(d) * sizeof( double ),
      hipMemcpyDeviceToHost)); for(int i=0;i<n;i++){ for(int j=0;j<d;j++){
          if(abs(Ft[i*d+j]-Ft2[i+j*n]/z)>0.0000001){
          printf("Error F1=%lf vs Fd=%lf \n",Ft[i*d+j],Ft2[i+j*n]/z );}
        }
      }
    */

    return 0;
  }
