#include "hip/hip_runtime.h"
#include "gridding.cuh"


__global__ void s2g1d(coord *V, coord *y, coord *q, uint32_t ng,
                          uint32_t nPts, uint32_t nDim, uint32_t nVec,
                          coord maxy) {
  coord v1[4];
  uint32_t f1;
  coord d;

  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {
    y[TID] /= maxy;
    if (y[TID] == 1) {
      y[TID] = y[TID] - 0.00000000000001;
    }
    y[TID] *= (ng - 3);

    f1 = (uint32_t)floor(y[TID]);
    d = y[TID] - (coord)f1;
    v1[0] = g2(1 + d);
    v1[1] = g1(d);
    v1[2] = g1(1 - d);
    v1[3] = g2(2 - d);

    for (int j = 0; j < nVec; j++) {
      coord qv = q[nPts * j + TID];
      for (int idx1 = 0; idx1 < 4; idx1++) {
        atomicAdd(&V[f1 + idx1 + j * ng],qv * v1[idx1]);
        //V[f1 + idx1 + j * ng] += qv * v1[idx1]
      }
    }
  }
}


__global__ void g2s1d(coord *Phi, coord *V, coord *y, uint32_t ng,
                          uint32_t nPts, uint32_t nDim, uint32_t nVec) {
  coord v1[4];
  uint32_t f1;
  coord d;

  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {
    f1 = (uint32_t)floor(y[TID]);
    d = y[TID] - (coord)f1;
    v1[0] = g2(1 + d);
    v1[1] = g1(d);
    v1[2] = g1(1 - d);
    v1[3] = g2(2 - d);

    for (uint32_t j = 0; j < nVec; j++) {
      coord accum = 0;
      for (uint32_t idx1 = 0; idx1 < 4; idx1++) {
        //printf("CUDA, V[%d]=%lf\n",f1 + idx1 + j * ng,V[f1 + idx1 + j * ng]  );
        accum += V[f1 + idx1 + j * ng] * v1[idx1];
      }
      Phi[TID + j * nPts] = accum;
    }
  }
}
