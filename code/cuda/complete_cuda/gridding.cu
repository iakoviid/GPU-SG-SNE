#include "hip/hip_runtime.h"
#include "gridding.cuh"
#include "matrix_indexing.hpp"
#define idx2(i, j, d) (SUB2IND2D(i, j, d))
#define idx4(i,j,k,l,m,n,o)  (SUB2IND4D(i,j,k,l,m,n,o))
__global__ void s2g1d(coord *V, coord *y, coord *q, uint32_t ng, uint32_t nPts,
                      uint32_t nDim, uint32_t nVec, coord maxy) {
  coord v1[4];
  uint32_t f1;
  coord d;

  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {
    y[TID] /= maxy;
    if (y[TID] == 1) {
      y[TID] = y[TID] - 0.00000000000001;
    }
    y[TID] *= (ng - 3);

    f1 = (uint32_t)floor(y[TID]);
    d = y[TID] - (coord)f1;
    v1[0] = g2(1 + d);
    v1[1] = g1(d);
    v1[2] = g1(1 - d);
    v1[3] = g2(2 - d);

    for (int j = 0; j < nVec; j++) {
      coord qv = q[nPts * j + TID];
      for (int idx1 = 0; idx1 < 4; idx1++) {
        atomicAdd(&V[f1 + idx1 + j * ng], qv * v1[idx1]);
        // V[f1 + idx1 + j * ng] += qv * v1[idx1]
      }
    }
  }
}
__global__ void s2g1drb(coord *V, coord *y, coord *q, uint32_t *ib, uint32_t *cb,
                uint32_t ng, uint32_t np, uint32_t nPts, uint32_t nDim,
                uint32_t nVec, coord maxy){
  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {
    y[TID] /= maxy;
    if (y[TID] == 1) {
      y[TID] = y[TID] - 0.00000000000001;
    }
    y[TID] *= (ng - 3);
  }
  for (uint32_t s = 0; s < 2; s++) { // red-black sync

    for (uint32_t idual = 6*blockIdx.x; idual < (ng - 3); idual += 6*gridDim.x) { // coarse-grid

      for (uint32_t ifine = 0; ifine < 3; ifine++) { // fine-grid

        // get index of current grid box
        uint32_t i = 3 * s + idual + ifine;

        // if above boundaries, break
        if (i > ng - 4)
          break;

        // loop through all points inside box
        for (uint32_t TID = 0; TID < cb[i]; TID+=blockDim.x) {

          uint32_t f1;
          coord d;
          coord v1[4];

          f1 = (uint32_t)floor(y[ib[i]+TID]);
          d = y[ib[i]+TID] - (coord)f1;

          v1[0] = g2(1 + d);
          v1[1] = g1(d);
          v1[2] = g1(1 - d);
          v1[3] = g2(2 - d);

          for (uint32_t j = 0; j < nVec; j++) {

            coord qv = q[nPts * j +ib[i]+ TID];

            for (uint32_t idx1 = 0; idx1 < 4; idx1++) {
              atomicAdd(&V[f1 + idx1 + j * ng], qv * v1[idx1]);


            } // (idx1)

          } // (j)

        } // (k)

      } // (ifine)

    } // (idual)

  } // (s)
}


__global__ void g2s1d(coord *Phi, coord *V, coord *y, uint32_t ng,
                      uint32_t nPts, uint32_t nDim, uint32_t nVec) {
  coord v1[4];
  uint32_t f1;
  coord d;

  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {
    f1 = (uint32_t)floor(y[TID]);
    d = y[TID] - (coord)f1;
    v1[0] = g2(1 + d);
    v1[1] = g1(d);
    v1[2] = g1(1 - d);
    v1[3] = g2(2 - d);

    for (uint32_t j = 0; j < nVec; j++) {
      coord accum = 0;
      for (uint32_t idx1 = 0; idx1 < 4; idx1++) {
        // printf("CUDA, V[%d]=%lf\n",f1 + idx1 + j * ng,V[f1 + idx1 + j * ng]
        // );
        accum += V[f1 + idx1 + j * ng] * v1[idx1];
      }
      Phi[TID + j * nPts] = accum;
    }
  }
}

__global__ void s2g2d(coord *V, coord *y, coord *q, uint32_t ng, uint32_t nPts,
                      uint32_t nDim, uint32_t nVec, coord maxy) {
  coord v1[4];
  coord v2[4];
  uint32_t f1;
  uint32_t f2;
  coord d;

  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {

    y[TID] /= maxy;
    if (y[TID] == 1) {
      y[TID] = y[TID] - 0.00000000000001;
    }
    y[TID] *= (ng - 3);

    y[TID + nPts] /= maxy;
    if (y[TID + nPts] == 1) {
      y[TID + nPts] = y[TID + nPts] - 0.00000000000001;
    }
    y[TID + nPts] *= (ng - 3);

    f1 = (uint32_t)floor(y[TID]);
    d = y[TID] - (coord)f1;
    v1[0] = g2(1 + d);
    v1[1] = g1(d);
    v1[2] = g1(1 - d);
    v1[3] = g2(2 - d);

    f2 = (uint32_t)floor(y[TID + nPts]);
    d = y[TID + nPts] - (coord)f2;
    v2[0] = g2(1 + d);
    v2[1] = g1(d);
    v2[2] = g1(1 - d);
    v2[3] = g2(2 - d);

    for (int j = 0; j < nVec; j++) {

      for (int idx2 = 0; idx2 < 4; idx2++) {
        coord qv = q[nPts * j + TID] * v2[idx2];

        for (int idx1 = 0; idx1 < 4; idx1++) {

          atomicAdd(&V[f1 + idx1 + (f2 + idx2) * ng + j * ng * ng],
                    qv * v1[idx1]);
        }
      }
    }
  }
}

__global__ void s2g3d(coord *V, coord *y, coord *q, uint32_t ng, uint32_t nPts,
                      uint32_t nDim, uint32_t nVec, coord maxy) {
  coord v1[4];
  coord v2[4];
  coord v3[4];
  uint32_t f1;
  uint32_t f2;
  uint32_t f3;
  coord d;

  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {
    y[TID] /= maxy;
    if (y[TID] == 1) {
      y[TID] = y[TID] - 0.00000000000001;
    }
    y[TID] *= (ng - 3);

    y[TID + nPts] /= maxy;
    if (y[TID + nPts] == 1) {
      y[TID + nPts] = y[TID + nPts] - 0.00000000000001;
    }
    y[TID + nPts] *= (ng - 3);

    f1 = (uint32_t)floor(y[TID]);
    d = y[TID] - (coord)f1;
    v1[0] = g2(1 + d);
    v1[1] = g1(d);
    v1[2] = g1(1 - d);
    v1[3] = g2(2 - d);

    f2 = (uint32_t)floor(y[TID + nPts]);
    d = y[TID + nPts] - (coord)f2;
    v2[0] = g2(1 + d);
    v2[1] = g1(d);
    v2[2] = g1(1 - d);
    v2[3] = g2(2 - d);

    f3 = (uint32_t)floor(y[TID + 2 * nPts]);
    d = y[TID + 2 * nPts] - (coord)f3;
    v3[0] = g2(1 + d);
    v3[1] = g1(d);
    v3[2] = g1(1 - d);
    v3[3] = g2(2 - d);

    for (int j = 0; j < nVec; j++) {
      for (int idx3 = 0; idx3 < 4; idx3++) {

        for (int idx2 = 0; idx2 < 4; idx2++) {
          coord qv = q[nPts * j + TID] * v2[idx2] * v3[idx3];

          for (int idx1 = 0; idx1 < 4; idx1++) {
            atomicAdd(&V[idx4(f1 + idx1, f2 + idx2, f3 + idx3, j, ng, ng, ng)],
                      qv * v1[idx1]);
          }
        }
      }
    }
  }
}

__global__ void g2s2d(coord *Phi, coord *V, coord *y, uint32_t ng,
                      uint32_t nPts, uint32_t nDim, uint32_t nVec) {
  coord v1[4];
  coord v2[4];
  uint32_t f1;
  uint32_t f2;
  coord d;

  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {

    f1 = (uint32_t)floor(y[TID]);
    d = y[TID] - (coord)f1;
    v1[0] = g2(1 + d);
    v1[1] = g1(d);
    v1[2] = g1(1 - d);
    v1[3] = g2(2 - d);

    f2 = (uint32_t)floor(y[TID + nPts]);
    d = y[TID + nPts] - (coord)f2;
    v2[0] = g2(1 + d);
    v2[1] = g1(d);
    v2[2] = g1(1 - d);
    v2[3] = g2(2 - d);

    for (int j = 0; j < nVec; j++) {
      coord accum = 0;
      for (int idx2 = 0; idx2 < 4; idx2++) {
        coord qv = v2[idx2];

        for (int idx1 = 0; idx1 < 4; idx1++) {

          accum +=
              V[f1 + idx1 + (f2 + idx2) * ng + j * ng * ng] * qv * v1[idx1];
        }
      }
      Phi[TID + j * nPts] = accum;
    }
  }
}
__global__ void g2s3d(coord *Phi, coord *V, coord *y, uint32_t ng,
                      uint32_t nPts, uint32_t nDim, uint32_t nVec) {
  coord v1[4];
  coord v2[4];
  coord v3[4];
  uint32_t f1;
  uint32_t f2;
  uint32_t f3;
  coord d;

  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {

    f1 = (uint32_t)floor(y[TID]);
    d = y[TID] - (coord)f1;
    v1[0] = g2(1 + d);
    v1[1] = g1(d);
    v1[2] = g1(1 - d);
    v1[3] = g2(2 - d);

    f2 = (uint32_t)floor(y[TID + nPts]);
    d = y[TID + nPts] - (coord)f2;
    v2[0] = g2(1 + d);
    v2[1] = g1(d);
    v2[2] = g1(1 - d);
    v2[3] = g2(2 - d);

    f3 = (uint32_t)floor(y[TID + 2 * nPts]);
    d = y[TID + 2 * nPts] - (coord)f3;
    v3[0] = g2(1 + d);
    v3[1] = g1(d);
    v3[2] = g1(1 - d);
    v3[3] = g2(2 - d);

    for (int j = 0; j < nVec; j++) {
      coord accum = 0;
      for (int idx3 = 0; idx3 < 4; idx3++) {

        for (int idx2 = 0; idx2 < 4; idx2++) {
          coord qv = v2[idx2] * v3[idx3];

          for (int idx1 = 0; idx1 < 4; idx1++) {

            accum += V[idx4(f1 + idx1, f2 + idx2, f3 + idx3, j, ng, ng, ng)] *
                     qv * v1[idx1];
          }
        }
        Phi[TID + j * nPts] = accum;
      }
    }
  }
}
