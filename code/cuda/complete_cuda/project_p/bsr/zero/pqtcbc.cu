#include "hip/hip_runtime.h"
template <typename data_type, typename index_type>
void spmv(const index_type* d_columns,const index_type* d_row_ptr,const data_type* d_values,index_type n_rows,index_type bs ,const data_type* Y, const data_type* Fattr, index_type n, index_type d ) {
  dim3 block_size = 32;
  dim3 grid_size{};

  grid_size.x = (n_rows * 32 + block_size.x - 1) / block_size.x;
  switch (bs) {
  case 1:
    bcsr_spmv_kernel_column_by_column_template<data_type, index_type, 1>
        <<<grid_size, block_size, block_size.x * sizeof(data_type)>>>(
            d_columns, d_row_ptr, d_values, Y, Fattr,n,d);
    break;
  case 2:
    bcsr_spmv_kernel_column_by_column_template<data_type, index_type, 2>
        <<<grid_size, block_size, block_size.x * sizeof(data_type)>>>(
            d_columns, d_row_ptr, d_values, Y, Fattr,n,d);
    break;
  case 3:
    bcsr_spmv_kernel_column_by_column_template<data_type, index_type, 3>
        <<<grid_size, block_size, block_size.x * sizeof(data_type)>>>(
            d_columns, d_row_ptr, d_values, Y, Fattr,n,d);
    break;
  case 4:
    bcsr_spmv_kernel_column_by_column_template<data_type, index_type, 4>
        <<<grid_size, block_size, block_size.x * sizeof(data_type)>>>(
            d_columns, d_row_ptr, d_values, Y, Fattr,n,d);
    break;
  case 8:
    bcsr_spmv_kernel_column_by_column_template<data_type, index_type, 8>
        <<<grid_size, block_size, block_size.x * sizeof(data_type)>>>(
            d_columns, d_row_ptr, d_values, Y, Fattr,n,d);
    break;
  case 16:
    bcsr_spmv_kernel_column_by_column_template<data_type, index_type, 16>
        <<<grid_size, block_size, block_size.x * sizeof(data_type)>>>(
            d_columns, d_row_ptr, d_values, Y, Fattr,n,d);
    break;
  case 32:
    bcsr_spmv_kernel_column_by_column_template<data_type, index_type, 32>
        <<<grid_size, block_size, block_size.x * sizeof(data_type)>>>(
            d_columns, d_row_ptr, d_values, Y, Fattr,n,d);
    break;
  }
}

template <typename data_type, typename index_type, index_type bs>
__global__ void bcsr_spmv_kernel_column_by_column_template(
    const index_type *__restrict__ col_ids,
    const index_type *__restrict__ row_ptr, const data_type *__restrict__ data,
    const data_type *__restrict__ Y, data_type *__restrict__ Fattr,index_type n,index_type d) {
  const index_type idx = blockIdx.x * blockDim.x + threadIdx.x;
  const index_type lane = idx % 32;
  const index_type block_row = idx / 32; ///< Warp per block row
  const index_type first_block = row_ptr[block_row];
  const index_type last_block = row_ptr[block_row + 1];

  index_type col = first_block * bs + lane / bs;
  index_type r = lane % bs;

  data_type *partial_sums =
      shared_memory<data_type>(); ///< Size is equal to blockDim.x *
                                  ///< sizeof(data_type)

                                  data_type sum1=0;
                                  data_type sum2=0;
                                  data_type sum3=0;
  for (; col < last_block * bs; col += 32 / bs) {
    const index_type block = col / bs;
    const index_type c = col % bs;
    int column=col_ids[block] * bs + c;
    int row=block_row*bs+r;
    coord dist=0;
    for(int dim=0;dim<d;dim++){
      dist+=(Y[row+dim*n]-Y[column+dim*n])*(Y[row+dim*n]-Y[column+dim*n]);
    }
    const coord value = data[block * bs * bs + c * bs + r]/(1+dist);
    switch (d) {
      case 1:
        sum1+=value*(Y[row]- Y[column]);
        break;
      case 2:
        sum1+=value*(Y[row]-Y[column]);
        sum2+=value*(Y[row+n]-Y[column+n]);
        break;
      case 3:
        sum1+=value*(Y[row]-Y[column]);
        sum2+=value*(Y[row+n]-Y[column+n]);
        sum3+=value*(Y[row+2*n]-Y[column+2*n]);
        break;
    }
  }

  switch (d) {
    case 1:
    partial_sums[threadIdx.x] = sum1;
    break;
    case 2:
    partial_sums[threadIdx.x] = sum1;
    partial_sums[threadIdx.x+blockDim.x]=sum2;
    break;
    case 3:
    partial_sums[threadIdx.x] = sum1;
    partial_sums[threadIdx.x+blockDim.x]=sum2;
    partial_sums[threadIdx.x+2*blockDim.x]=sum3;
    break;

  }
  for (index_type stride = round_up_to_power_of_two((32 / bs) / 2); stride > 0;
       stride /= 2) {
    __syncthreads();
    if ((lane < stride * bs) && ((threadIdx.x + stride * bs) < 32))
    for(int dim=0;dim<d;dim++){
      partial_sums[threadIdx.x+blockDim.x*dim] += partial_sums[threadIdx.x + stride * bs+blockDim.x*dim];

    }
  }

  if (lane < bs)
  for(int dim=0;dim<d;dim++){
    Fattr[block_row * bs + lane+dim*n] = partial_sums[threadIdx.x+blockDim.x*dim];
  }
}
