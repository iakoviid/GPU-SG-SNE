#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2020, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/


template <int num_thread_per_worker, bool atomic, typename ValueType,
          typename IndexType, typename Closure>
__device__ void spmv_kernel(
    const size_type num_rows, const int num_worker_per_row,
    const ValueType *__restrict__ val, const IndexType *__restrict__ col,
    const size_type stride, const size_type num_stored_elements_per_row,
    const ValueType *__restrict__ b, const size_type b_stride,
    ValueType *__restrict__ c, const size_type c_stride, Closure op)
{
    const auto tidx = thread::get_thread_id_flat();
    const auto column_id = blockIdx.y;
    if (num_thread_per_worker == 1) {
        // Specialize the num_thread_per_worker = 1. It doesn't need the shared
        // memory, __syncthreads, and atomic_add
        if (tidx < num_rows) {
            ValueType temp = zero<ValueType>();
            for (size_type idx = 0; idx < num_stored_elements_per_row; idx++) {
                const auto ind = tidx + idx * stride;
                const auto col_idx = col[ind];
                if (col_idx < idx) {
                    break;
                } else {
                    temp += val[ind] * b[col_idx * b_stride + column_id];
                }
            }
            const auto c_ind = tidx * c_stride + column_id;
            c[c_ind] = op(temp, c[c_ind]);
        }
    } else {
        if (tidx < num_worker_per_row * num_rows) {
            const auto idx_in_worker = threadIdx.y;
            const auto x = tidx % num_rows;
            const auto worker_id = tidx / num_rows;
            const auto step_size = num_worker_per_row * num_thread_per_worker;
            __shared__ UninitializedArray<ValueType, default_block_size /
                                                         num_thread_per_worker>
                storage;
            if (idx_in_worker == 0) {
                storage[threadIdx.x] = 0;
            }
            __syncthreads();
            ValueType temp = zero<ValueType>();
            for (size_type idx =
                     worker_id * num_thread_per_worker + idx_in_worker;
                 idx < num_stored_elements_per_row; idx += step_size) {
                const auto ind = x + idx * stride;
                const auto col_idx = col[ind];
                if (col_idx < idx) {
                    break;
                } else {
                    temp += val[ind] * b[col_idx * b_stride + column_id];
                }
            }
            atomic_add(&storage[threadIdx.x], temp);
            __syncthreads();
            if (idx_in_worker == 0) {
                const auto c_ind = x * c_stride + column_id;
                if (atomic) {
                    atomic_add(&(c[c_ind]), op(storage[threadIdx.x], c[c_ind]));
                } else {
                    c[c_ind] = op(storage[threadIdx.x], c[c_ind]);
                }
            }
        }
    }
}


template <int num_thread_per_worker, bool atomic = false, typename ValueType,
          typename IndexType>
__global__ __launch_bounds__(default_block_size) void spmv(
    const size_type num_rows, const int num_worker_per_row,
    const ValueType *__restrict__ val, const IndexType *__restrict__ col,
    const size_type stride, const size_type num_stored_elements_per_row,
    const ValueType *__restrict__ b, const size_type b_stride,
    ValueType *__restrict__ c, const size_type c_stride)
{
    spmv_kernel<num_thread_per_worker, atomic>(
        num_rows, num_worker_per_row, val, col, stride,
        num_stored_elements_per_row, b, b_stride, c, c_stride,
        [](const ValueType &x, const ValueType &y) { return x; });
}


template <int num_thread_per_worker, bool atomic = false, typename ValueType,
          typename IndexType>
__global__ __launch_bounds__(default_block_size) void spmv(
    const size_type num_rows, const int num_worker_per_row,
    const ValueType *__restrict__ alpha, const ValueType *__restrict__ val,
    const IndexType *__restrict__ col, const size_type stride,
    const size_type num_stored_elements_per_row,
    const ValueType *__restrict__ b, const size_type b_stride,
    const ValueType *__restrict__ beta, ValueType *__restrict__ c,
    const size_type c_stride)
{
    const ValueType alpha_val = alpha[0];
    const ValueType beta_val = beta[0];
    // Because the atomic operation changes the values of c during computation,
    // it can not do the right alpha * a * b + beta * c operation.
    // Thus, the cuda kernel only computes alpha * a * b when it uses atomic
    // operation.
    if (atomic) {
        spmv_kernel<num_thread_per_worker, atomic>(
            num_rows, num_worker_per_row, val, col, stride,
            num_stored_elements_per_row, b, b_stride, c, c_stride,
            [&alpha_val](const ValueType &x, const ValueType &y) {
                return alpha_val * x;
            });
    } else {
        spmv_kernel<num_thread_per_worker, atomic>(
            num_rows, num_worker_per_row, val, col, stride,
            num_stored_elements_per_row, b, b_stride, c, c_stride,
            [&alpha_val, &beta_val](const ValueType &x, const ValueType &y) {
                return alpha_val * x + beta_val * y;
            });
    }
}
