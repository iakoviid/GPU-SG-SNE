void cusparse_bsrmv (
  hipsparseHandle_t  &handle,
  hipsparseMatDescr_t  &descr_A,
  hipsparseDirection_t direction,

  int n_rows,
  int n_cols,
  int nnzb,
  int bs,

  const float *A,
  const int *row_ptr,
  const int *columns,
  const float *x,
  float *y
  )
{
  const float alpha = 1.0;
  const float beta = 0.0;

  hipsparseSbsrmv (
    handle,
    direction,
    HIPSPARSE_OPERATION_NON_TRANSPOSE,
    n_rows, n_cols, nnzb,
    &alpha, descr_A, A,
    row_ptr, columns, bs,
    x, &beta, y);
}

void cusparse_bsrmv (
  hipsparseHandle_t  &handle,
  hipsparseMatDescr_t  &descr_A,
  hipsparseDirection_t direction,

  int n_rows,
  int n_cols,
  int nnzb,
  int bs,

  const double *A,
  const int *row_ptr,
  const int *columns,
  const double *x,
  double *y
)
{
  const double alpha = 1.0;
  const double beta = 0.0;

  hipsparseDbsrmv (
    handle,
    direction,
    HIPSPARSE_OPERATION_NON_TRANSPOSE,
    n_rows, n_cols, nnzb,
    &alpha, descr_A, A,
    row_ptr, columns, bs,
    x, &beta, y);
}

/// cuSPARSE Column major
{
  hipsparseHandle_t handle;
  hipsparseCreate (&handle);

  hipsparseMatDescr_t descr_A;
  hipsparseCreateMatDescr (&descr_A);
  hipsparseSetMatType (descr_A, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase (descr_A, HIPSPARSE_INDEX_BASE_ZERO);

  hipEvent_t start, stop;
  hipEventCreate (&start);
  hipEventCreate (&stop);

  hipDeviceSynchronize ();
  hipEventRecord (start);

  cusparse_bsrmv (handle, descr_A, HIPSPARSE_DIRECTION_COLUMN, matrix.n_rows, matrix.n_cols, matrix.nnzb, matrix.bs, d_values, d_row_ptr, d_columns, d_x, d_y);

  hipEventRecord (stop);
  hipEventSynchronize (stop);

  float milliseconds = 0;
  hipEventElapsedTime (&milliseconds, start, stop);
  const double elapsed = milliseconds / 1000;

  hipEventDestroy (start);
  hipEventDestroy (stop);

  hipsparseDestroyMatDescr (descr_A);
  hipsparseDestroy (handle);

  results.emplace_back ("GPU BSR (cuSPARSE, column major)", elapsed, 0, 0);

  hipMemcpy (cpu_y.get (), d_y, y_size * sizeof (data_type), hipMemcpyDeviceToHost);
  compare_results (y_size, reference_y, cpu_y.get ());
}
