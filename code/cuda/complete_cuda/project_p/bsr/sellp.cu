#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2020, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/



template <typename ValueType, typename IndexType>
__global__ __launch_bounds__(matrix::default_slice_size) void spmv_kernel(
    size_type num_rows, size_type num_right_hand_sides, size_type b_stride,
    size_type c_stride, const size_type *__restrict__ slice_lengths,
    const size_type *__restrict__ slice_sets, const ValueType *__restrict__ a,
    const IndexType *__restrict__ col, const ValueType *__restrict__ b,
    ValueType *__restrict__ c)
{
    const auto slice_id = blockIdx.x;
    const auto slice_size = blockDim.x;
    const auto row_in_slice = threadIdx.x;
    const auto global_row =
        static_cast<size_type>(slice_size) * slice_id + row_in_slice;
    const auto column_id = blockIdx.y;
    ValueType val = 0;
    IndexType ind = 0;
    if (global_row < num_rows && column_id < num_right_hand_sides) {
        for (size_type i = 0; i < slice_lengths[slice_id]; i++) {
            ind = row_in_slice + (slice_sets[slice_id] + i) * slice_size;
            val += a[ind] * b[col[ind] * b_stride + column_id];
        }
        c[global_row * c_stride + column_id] = val;
    }
}


template <typename ValueType, typename IndexType>
__global__
    __launch_bounds__(matrix::default_slice_size) void advanced_spmv_kernel(
        size_type num_rows, size_type num_right_hand_sides, size_type b_stride,
        size_type c_stride, const size_type *__restrict__ slice_lengths,
        const size_type *__restrict__ slice_sets,
        const ValueType *__restrict__ alpha, const ValueType *__restrict__ a,
        const IndexType *__restrict__ col, const ValueType *__restrict__ b,
        const ValueType *__restrict__ beta, ValueType *__restrict__ c)
{
    const auto slice_id = blockIdx.x;
    const auto slice_size = blockDim.x;
    const auto row_in_slice = threadIdx.x;
    const auto global_row =
        static_cast<size_type>(slice_size) * slice_id + row_in_slice;
    const auto column_id = blockIdx.y;
    ValueType val = 0;
    IndexType ind = 0;
    if (global_row < num_rows && column_id < num_right_hand_sides) {
        for (size_type i = 0; i < slice_lengths[slice_id]; i++) {
            ind = row_in_slice + (slice_sets[slice_id] + i) * slice_size;
            val += alpha[0] * a[ind] * b[col[ind] * b_stride + column_id];
        }
        c[global_row * c_stride + column_id] =
            beta[0] * c[global_row * c_stride + column_id] + val;
    }
}
