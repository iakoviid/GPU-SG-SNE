
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define WSIZE 32
#define LOOPS 100000
#define UPPER_BIT 31
#define LOWER_BIT 0

__device__ unsigned int ddata[WSIZE];

// naive warp-level bitwise radix sort

__global__ void mykernel(){
  __shared__ volatile unsigned int sdata[WSIZE*2];
  // load from global into shared variable
  sdata[threadIdx.x] = ddata[threadIdx.x];
  unsigned int bitmask = 1<<LOWER_BIT;
  unsigned int offset  = 0;
  unsigned int thrmask = 0xFFFFFFFFU << threadIdx.x;
  unsigned int mypos;
  //  for each LSB to MSB
  for (int i = LOWER_BIT; i <= UPPER_BIT; i++){
    unsigned int mydata = sdata[((WSIZE-1)-threadIdx.x)+offset];
    unsigned int mybit  = mydata&bitmask;
    // get population of ones and zeroes (cc 2.0 ballot)
    unsigned int ones = __ballot(mybit); // cc 2.0
    unsigned int zeroes = ~ones;
    offset ^= WSIZE; // switch ping-pong buffers
    // do zeroes, then ones
    if (!mybit) // threads with a zero bit
      // get my position in ping-pong buffer
      mypos = __popc(zeroes&thrmask);
    else        // threads with a one bit
      // get my position in ping-pong buffer
      mypos = __popc(zeroes)+__popc(ones&thrmask);
    // move to buffer  (or use shfl for cc 3.0)
    sdata[mypos-1+offset] = mydata;
    // repeat for next bit
    bitmask <<= 1;
    }
  // save results to global
  ddata[threadIdx.x] = sdata[threadIdx.x+offset];
  }

int main(){

  unsigned int hdata[WSIZE];
  for (int lcount = 0; lcount < LOOPS; lcount++){
    unsigned int range = 1U<<UPPER_BIT;
    for (int i = 0; i < WSIZE; i++) hdata[i] = rand()%range;
    hipMemcpyToSymbol(HIP_SYMBOL(ddata), hdata, WSIZE*sizeof(unsigned int));
    mykernel<<<1, WSIZE>>>();
    hipMemcpyFromSymbol(hdata, HIP_SYMBOL(ddata), WSIZE*sizeof(unsigned int));
    for (int i = 0; i < WSIZE-1; i++) if (hdata[i] > hdata[i+1]) {printf("sort error at loop %d, hdata[%d] = %d, hdata[%d] = %d\n", lcount,i, hdata[i],i+1, hdata[i+1]); return 1;}
    // printf("sorted data:\n");
    //for (int i = 0; i < WSIZE; i++) printf("%u\n", hdata[i]);
    }
  printf("Success!\n");
  return 0;
}
