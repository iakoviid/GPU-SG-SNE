#include "hip/hip_runtime.h"

#include "nuconv.cuh"
//#include "gridding.cpp"
//#include "non_periodic_conv.cpp"

void nuconv(coord *PhiScat, coord *y, coord *VScat, uint32_t *ib, uint32_t *cb,
            int n, int d, int m, int np, int nGridDim) {

  // ~~~~~~~~~~ normalize coordinates (inside bins)
  thrust::device_ptr<double> yVec_ptr(y_d);
  thrust::device_vector<double> yVec_d(yVec_ptr, yVec_ptr + n * d);
  thrust::device_vector<double>::iterator iter =
      thrust::max_element(yVec_d.begin(), yVec_d.end());
  unsigned int position = iter - yVec_d.begin();
  maxy = yVec_d[position];



    coord h = maxy / (nGridDim - 1 - std::numeric_limits<coord>::epsilon());

    // ~~~~~~~~~~ scat2grid
    int szV = pow(nGridDim + 2, d) * m;
    coord *VGrid;
    CUDA_CALL(hipMallocManaged(&VGrid, szV * sizeof(coord)));


      switch (d) {

      case 1:
        if (nGridDim <= GRID_SIZE_THRESHOLD)
          s2g1d( VGrid, y, VScat, nGridDim+2, np, n, d, m );
        else
          printf("papa\n" );

          //s2g1drb( VGrid, y, VScat, ib, cb, nGridDim+2, np, n, d, m );
        break;

      case 2:
        if (nGridDim <= GRID_SIZE_THRESHOLD)
          printf("papa\n" );

          //s2g2d( VGrid, y, VScat, nGridDim+2, np, n, d, m );
        else
          printf("papa\n" );

          //s2g2drb( VGrid, y, VScat, ib, cb, nGridDim+2, np, n, d, m );
        break;

      case 3:
        if (nGridDim <= GRID_SIZE_THRESHOLD)
          printf("papa\n" );

          //s2g3d( VGrid, y, VScat, nGridDim+2, np, n, d, m );
        else
          printf("papa\n" );

          //s2g3drb( VGrid, y, VScat, ib, cb, nGridDim+2, np, n, d, m );
        break;

      }



    // ~~~~~~~~~~ grid2grid
    coord *PhiGrid;
    CUDA_CALL(hipMallocManaged(&PhiGrid, szV * sizeof(coord)));


    /*
      switch (d) {

      case 1:
        conv1dnopad( PhiGrid, VGrid, h, nGridDims, m, d, np );
        break;

      case 2:
        conv2dnopad( PhiGrid, VGrid, h, nGridDims, m, d, np );
        break;

      case 3:
        conv3dnopad( PhiGrid, VGrid, h, nGridDims, m, d, np );
        break;

      }
    */

    // ~~~~~~~~~~ grid2scat

    /*
      switch (d) {

      case 1:
        g2s1d( PhiScat, PhiGrid, y, nGridDim+2, n, d, m );
        break;

      case 2:
        g2s2d( PhiScat, PhiGrid, y, nGridDim+2, n, d, m );
        break;

      case 3:
        g2s3d( PhiScat, PhiGrid, y, nGridDim+2, n, d, m );
        break;

      }
    */

    // ~~~~~~~~~~ deallocate memory

  }
