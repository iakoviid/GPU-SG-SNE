#include "hip/hip_runtime.h"

#include "nuconv.cuh"
#define E_LIMIT 0.00000000000001
__global__ void Scale(coord *y, uint32_t nPts, uint32_t ng, uint32_t d,
                          coord maxy) {
  for (register int TID = threadIdx.x + blockIdx.x * blockDim.x; TID < nPts;
       TID += gridDim.x * blockDim.x) {
    for (int dim = 0; dim < d; dim++) {
      y[TID + dim * nPts] /= maxy;
      if (y[TID + dim * nPts] == 1) {
        y[TID + dim * nPts] = y[TID + dim * nPts] - E_LIMIT;
      }
      y[TID + dim * nPts] *= (ng - 3);
    }
  }
}

void nuconv(coord *PhiScat, coord *y, coord *VScat, uint32_t *ib, uint32_t *cb,
            int n, int d, int m, int nGridDim) {

  // ~~~~~~~~~~ Scale coordinates (inside bins)
  thrust::device_ptr<double> yVec_ptr(y);
  thrust::device_vector<double> yVec_d(yVec_ptr, yVec_ptr + n * d);
  thrust::device_vector<double>::iterator iter =
      thrust::max_element(yVec_d.begin(), yVec_d.end());
  unsigned int position = iter - yVec_d.begin();
  coord maxy = yVec_d[position];

  coord h = maxy / (nGridDim - 1 - std::numeric_limits<coord>::epsilon());

  // ~~~~~~~~~~ scat2grid
  int szV = pow(nGridDim + 2, d) * m;
  coord *VGrid;
  CUDA_CALL(hipMallocManaged(&VGrid, szV * sizeof(coord)));
  Scale<<<64,512>>>(y, n, nGridDim, d, maxy);
  hipDeviceSynchronize();
  switch (d) {

  case 1:
    if (nGridDim <= GRID_SIZE_THRESHOLD) {

      s2g1d<<<32, 512>>>(VGrid, y, VScat, nGridDim + 2, n, d, d + 1);
    } else {
      s2g1drb<<<64, 32>>>(VGrid, y, VScat, ib, cb, nGridDim + 2, n, d, m);
    }

    // s2g1drb( VGrid, y, VScat, ib, cb, nGridDim+2, np, n, d, m );
    break;

  case 2:
    if (nGridDim <= GRID_SIZE_THRESHOLD)

      s2g2d<<<32, 512>>>(VGrid, y, VScat, nGridDim + 2, n, d, m);

    else

      s2g2drb<<<64,32>>>(VGrid, y, VScat, ib, cb, nGridDim + 2, n, d, m);
    break;

  case 3:
    if (nGridDim <= GRID_SIZE_THRESHOLD)
       s2g3d<<<64,256>>>( VGrid, y, VScat, nGridDim+2, n, d, m );
    else
        s2g3drb<<<64,32>>>(VGrid, y, VScat, ib, cb, nGridDim + 2, n, d, m);

    // s2g3drb( VGrid, y, VScat, ib, cb, nGridDim+2, np, n, d, m );
    break;
  }

  // ~~~~~~~~~~ grid2grid
  coord *PhiGrid;
  CUDA_CALL(hipMallocManaged(&PhiGrid, szV * sizeof(coord)));

  uint32_t *const nGridDims = new uint32_t[d]();
  for (int i = 0; i < d; i++) {
    nGridDims[i] = nGridDim + 2;
  }

  switch (d) {

  case 1:
    conv1dnopadcuda(PhiGrid, VGrid, h, nGridDim + 2, m, d);

    break;

  case 2:
    conv2dnopadcuda(PhiGrid, VGrid, h, nGridDims, m, d);

    break;

  case 3:
    conv3dnopadcuda(PhiGrid,VGrid,h,nGridDims,m,d);
    break;
  }

  // ~~~~~~~~~~ grid2scat

  switch (d) {

  case 1:
    g2s1d<<<32, 256>>>(PhiScat, PhiGrid, y, nGridDim + 2, n, d, m);

    break;

  case 2:
    g2s2d<<<32, 256>>>(PhiScat, PhiGrid, y, nGridDim + 2, n, d, m);
    break;

  case 3:
    g2s3d<<<32, 256>>>( PhiScat, PhiGrid, y, nGridDim+2, n, d, m );
    break;
  }

  // ~~~~~~~~~~ deallocate memory
}
